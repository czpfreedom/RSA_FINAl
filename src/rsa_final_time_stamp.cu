#include "stdio.h"
#include "string.h"
#include "rsa_final_time_stamp.h"
#include <ctime>
#include <sys/time.h>

namespace namespace_rsa_final{

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

Time_Stamp:: Time_Stamp(){

}


Time_Stamp:: Time_Stamp(struct timeval tv){
    struct tm* pTime;
    pTime = localtime(&tv.tv_sec);
    m_data=(unsigned long)((pTime->tm_year+900)%1000)*(unsigned long)(10000000000000000)+(unsigned long)(pTime->tm_mon+1)*(unsigned long)(100000000000000)+(unsigned long)(pTime->tm_mday)*(unsigned long)(1000000000000)+ (unsigned long)(pTime->tm_hour)*(unsigned long)(10000000000)+ (unsigned long)(pTime->tm_min)*(unsigned long)(100000000)+(unsigned long)(pTime->tm_sec)*(unsigned long)(1000000)+(unsigned long)(tv.tv_usec/1000)*(unsigned long)(1000)+(unsigned long)tv.tv_usec%1000;

    sprintf(m_abbr,"%d-%d-%d-%d-%d-%d-%d-%d",pTime->tm_year+1900,pTime->tm_mon+1,pTime->tm_mday,pTime->tm_hour,pTime->tm_min,pTime->tm_sec,tv.tv_usec/1000,tv.tv_usec%1000);
    sprintf(m_full,"Year:%d Month:%d Day:%d Hour:%d Min:%d Sec:%d ", pTime->tm_year+1900,pTime->tm_mon+1,pTime->tm_mday,pTime->tm_hour,pTime->tm_min,pTime->tm_sec);

}

Time_Stamp:: Time_Stamp(Time_Stamp& time_stamp){
    m_data=time_stamp.m_data;
    memcpy(m_abbr,time_stamp.m_abbr,TIME_STAMP_ABBR_LENGTH);
    memcpy(m_full,time_stamp.m_full,TIME_STAMP_FULL_LENGTH);
}

Time_Stamp& Time_Stamp:: operator=(Time_Stamp& time_stamp){
    m_data=time_stamp.m_data;
    memcpy(m_abbr,time_stamp.m_abbr,TIME_STAMP_ABBR_LENGTH);
    memcpy(m_full,time_stamp.m_full,TIME_STAMP_FULL_LENGTH);
    return * this;
}


int Time_Stamp:: refresh(){
    return 0;
}

Time_System :: Time_System(){
    hipEventCreate(&m_cuda_create_time);	
    hipEventCreate(&m_cuda_start_time);	
    hipEventCreate(&m_cuda_pre_time);	
    hipEventCreate(&m_cuda_impl_time);	
    hipEventCreate(&m_cuda_quit_time);	
}

Time_System :: ~Time_System(){
    hipEventDestroy(m_cuda_create_time);
    hipEventDestroy(m_cuda_start_time);
    hipEventDestroy(m_cuda_pre_time);
    hipEventDestroy(m_cuda_impl_time);
    hipEventDestroy(m_cuda_quit_time);
}

int Time_System :: refresh(Time_System_Node time_system_node){
    if(time_system_node==Time_Create_NODE){
	m_cpu_create_time=cpuSecond();
	hipEventRecord(m_cuda_create_time, 0);
        return 0;
    }
    if(time_system_node==Time_Start_NODE){
	m_cpu_start_time=cpuSecond();
	hipEventRecord(m_cuda_start_time, 0);
	return 0;
    }
    if(time_system_node==Time_Pre_NODE){
	m_cpu_pre_time=cpuSecond();
	hipEventRecord(m_cuda_pre_time, 0);
	return 0;
    }
    if(time_system_node==Time_Impl_NODE){
	m_cpu_impl_time=cpuSecond();
	hipEventRecord(m_cuda_impl_time, 0);
	return 0;
    }
    if(time_system_node==Time_Quit_NODE){
	m_cpu_impl_time=cpuSecond();
	hipEventRecord(m_cuda_impl_time, 0);
	return 0;
    }
    return -1;
}

double Time_System :: CPU_TIME(double start_time, double end_time){
    return end_time-start_time; 
}

float Time_System :: CUDA_TIME(hipEvent_t &start_time, hipEvent_t &end_time){
     hipEventSynchronize(start_time);
     hipEventSynchronize(end_time);
     float time;
     hipEventElapsedTime(&time, start_time, end_time);
     return time;
}

}
