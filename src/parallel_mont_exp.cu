#include "hip/hip_runtime.h"
#include "bn_word_operation.h"
#include "pseudo.h"
#include "parallel_mont_exp.h"
#include "stdio.h"
#include "time.h"

#define CUDA_TIMING

__host__ __device__ int int_mod(const int a,const int b){
    int c=a;
    if(c<0){
        c=c+b;
    }
    return c%b;
}

__host__ int BN_WORD_inverse(const BN_WORD *n, BN_WORD *n_inverse){
    int dmax=n->dmax;
    BN_WORD *zero,*one,*R1, *R2, *s1, *s2, *t1, *t2,*temp,*div_temp,*temp_mul,*q;
    zero=BN_WORD_new(dmax);
    one=BN_WORD_new(dmax);
    R1=BN_WORD_new(dmax);
    R2=BN_WORD_new(dmax);
    s1=BN_WORD_new(dmax);
    s2=BN_WORD_new(dmax);
    t1=BN_WORD_new(dmax);
    t2=BN_WORD_new(dmax);
    temp=BN_WORD_new(dmax);
    div_temp=BN_WORD_new(dmax);
    q=BN_WORD_new(dmax);
    temp_mul=BN_WORD_new(dmax);
    BN_WORD_setzero(zero);
    BN_WORD_setone(one);
    BN_WORD_copy(n,R1);
    BN_WORD_sub(zero,R1,R2);
    BN_WORD_div(R2,R1,q,R2);
    BN_WORD_add(q,one,q);
    BN_WORD_setzero(s1);
    BN_WORD_setone(t1);
    BN_WORD_sub(zero,q,t2);
    while(BN_WORD_cmp(R2,one)!=0){
        BN_WORD_copy(R2,temp);
	BN_WORD_div(R1,R2,q,div_temp);
	BN_WORD_copy(div_temp,R2);
	BN_WORD_copy(temp,R1);
	BN_WORD_copy(s2,temp);
	BN_WORD_mul(q,s2,temp_mul);
	BN_WORD_sub(s1,temp_mul,s2);
	BN_WORD_copy(temp,s1);
	BN_WORD_copy(t2,temp);
	BN_WORD_mul(q,t2,temp_mul);
	BN_WORD_sub(t1,temp_mul,t2);
        BN_WORD_copy(temp,t1);
    }
    BN_WORD_copy(t2,n_inverse);
    BN_WORD_free(zero);
    BN_WORD_free(one);
    BN_WORD_free(R1);
    BN_WORD_free(R2);
    BN_WORD_free(s1);
    BN_WORD_free(s2);
    BN_WORD_free(t1);
    BN_WORD_free(t2);
    BN_WORD_free(temp);
    BN_WORD_free(q);
    BN_WORD_free(temp_mul);
    return 0;
}


__host__ int BN_NUM_inverse(const BN_NUM *n, const int d, const int l, BN_NUM *n_inverse){
    BN_NUM *zero,*one,*R1, *R2, *s1, *s2, *t1, *t2,*temp,*temp_mul,*q;
    zero=BN_NUM_new(d,l);
    one=BN_NUM_new(d,l);
    R1=BN_NUM_new(d,l);
    R2=BN_NUM_new(d,l);
    s1=BN_NUM_new(d,l);
    s2=BN_NUM_new(d,l);
    t1=BN_NUM_new(d,l);
    t2=BN_NUM_new(d,l);
    temp=BN_NUM_new(d,l);
    q=BN_NUM_new(d,l);
    temp_mul=BN_NUM_new(d,l);
    BN_NUM_setzero(zero);
    BN_NUM_setone(one);
    BN_NUM_copy(n,R1);
    BN_NUM_sub(zero,R1,R2);
    BN_NUM_div(R2,R1,q,R2);
    BN_NUM_add(q,one,q);
    BN_NUM_setzero(s1);
    BN_NUM_setone(t1);
    BN_NUM_sub(zero,q,t2);
    while(BN_NUM_cmp(R2,one)!=0){
        BN_NUM_copy(R2,temp);
	BN_NUM_div(R1,R2,q,R2);
	BN_NUM_copy(temp,R1);
	BN_NUM_copy(s2,temp);
	BN_NUM_mul(q,s2,temp_mul);
	BN_NUM_sub(s1,temp_mul,s2);
	BN_NUM_copy(temp,s1);
	BN_NUM_copy(t2,temp);
	BN_NUM_mul(q,t2,temp_mul);
	BN_NUM_sub(t1,temp_mul,t2);
        BN_NUM_copy(temp,t1);
    }
    BN_NUM_copy(t2,n_inverse);
    BN_NUM_free(zero);
    BN_NUM_free(one);
    BN_NUM_free(R1);
    BN_NUM_free(R2);
    BN_NUM_free(s1);
    BN_NUM_free(s2);
    BN_NUM_free(t1);
    BN_NUM_free(t2);
    BN_NUM_free(temp);
    BN_NUM_free(q);
    BN_NUM_free(temp_mul);
    return 0;
}

__host__ int BN_NUM_parallel_mod_mul(const BN_NUM *a, const BN_NUM *b, const BN_NUM *n, BN_NUM *result){    	
#ifdef CLOCKING
    clock_t start, stop, sumtime;
#endif

#ifdef CUDA_TIMING
    hipEvent_t start, stop;
    float sum_time = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

#ifdef CUDA_TIMING
    hipEventRecord(start, 0);
#endif

    int wmax=a->wmax;
    int dmax=a->word[0]->dmax;
    BN_NUM *u, *u_temp, *v, *v_temp,*m, *c ,*temp_result,*a_sub,*b_sub;
    int *any_value;
    BN_WORD *n0_inverse, *temp_word_result, *zero;
    hipMallocManaged((void **)&(any_value),wmax*sizeof(int));
    temp_result=BN_NUM_new(wmax,dmax);
    u=BN_NUM_new(wmax,dmax);
    u_temp=BN_NUM_new(wmax,dmax);
    v_temp=BN_NUM_new(wmax,dmax);
    v=BN_NUM_new(wmax,dmax);
    m=BN_NUM_new(wmax,dmax);
    c=BN_NUM_new(wmax,dmax);
    a_sub=BN_NUM_new(wmax,dmax);
    b_sub=BN_NUM_new(wmax,dmax);
    n0_inverse=BN_WORD_new(dmax);
    zero=BN_WORD_new(dmax);
    temp_word_result=BN_WORD_new(dmax);
    BN_NUM_copy(a,a_sub);
    BN_NUM_copy(b,b_sub);
    BN_WORD_setzero(zero);
    while(BN_NUM_cmp(a_sub,n)==1){
        BN_NUM_sub(a_sub,n,temp_result);
	BN_NUM_copy(temp_result,a_sub);
    }
    if(BN_NUM_cmp(a_sub,n)==0){
        BN_NUM_setzero(result);
	return 0;
    }
    while(BN_NUM_cmp(b_sub,n)==1){
        BN_NUM_sub(b_sub,n,temp_result);
        BN_NUM_copy(temp_result,b_sub);
    }
    if(BN_NUM_cmp(b_sub,n)==0){
        BN_NUM_setzero(result);
        return 0;
    }
    BN_WORD_inverse(n->word[0],n0_inverse);
    BN_WORD_sub(zero,n0_inverse,temp_word_result);
    BN_WORD_copy(temp_word_result,n0_inverse);

#ifdef CUDA_TIMING
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&sum_time, start, stop);
    printf("pro_time:%f\n",sum_time);
#endif

#ifdef CLOCKING
    start=clock();
#endif

#ifdef CUDA_TIMING
    hipEventRecord(start, 0);
#endif
    parallel_mont_mul<<<1,wmax>>>(a_sub,b_sub,n,wmax,dmax,n0_inverse,result,u,u_temp,v,m,c,v_temp,any_value);
    hipDeviceSynchronize();

#ifdef CLOCKING
    stop=clock();
    sumtime=stop-start;
    printf("device_time:%ld\n",sumtime);
#endif

#ifdef CUDA_TIMING
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&sum_time, start, stop);
    printf("device_time:%f\n",sum_time);
#endif
    hipFree(any_value);
    BN_NUM_free(temp_result);
    BN_NUM_free(u);
    BN_NUM_free(u_temp);
    BN_NUM_free(v_temp);
    BN_NUM_free(v);
    BN_NUM_free(m);
    BN_NUM_free(c);
    BN_NUM_free(a_sub);
    BN_NUM_free(b_sub);
    BN_WORD_free(zero);
    BN_WORD_free(n0_inverse);
    BN_WORD_free(temp_word_result);
    return 0;
}


__global__ void parallel_mont_mul(const BN_NUM *a,const BN_NUM *b,const BN_NUM *n,const int wmax,const int dmax,const BN_WORD *n0_inverse,
		BN_NUM *result, BN_NUM *u, BN_NUM *u_temp,BN_NUM *v, BN_NUM *m, BN_NUM *c,BN_NUM *v_temp, int *any_value){
    int j=threadIdx.x+blockIdx.x*blockDim.x;
#ifdef TIMING
    time_t start, stop , sumtime;
    start=time(NULL);
#endif

#ifdef CLOCKING
    clock_t start, stop , sumtime;
    if(j==0){
	    start=clock();
    }
#endif
    
    BN_WORD_setzero(u->word[j]);
    BN_WORD_setzero(v->word[j]);
    __syncthreads();
    BN_WORD *temp_result, *zero,* one;
    BN_NUM  *bn_temp_result, *bn_temp_result2, *zero_num;
    temp_result=BN_WORD_new_device(dmax);
    zero=BN_WORD_new_device(dmax);
    one=BN_WORD_new_device(dmax);
    BN_WORD_setzero(zero);
    BN_WORD_setone(one);
    bn_temp_result=BN_NUM_new_device(wmax,dmax);
    bn_temp_result2=BN_NUM_new_device(wmax,dmax);
    zero_num=BN_NUM_new_device(wmax,dmax);
    BN_NUM_setzero(zero_num);
//need error_check
    
#ifdef CLOCKING
    if(j==0){
       stop=clock();
       sumtime=stop-start;
       printf("pro_time:%ld\n",sumtime);
    }
#endif

#ifdef CLOCKING
    if(j==0){
        start=clock();
    }
#endif
    for(int i=0;i<wmax;i++){
        mad_lo(a->word[j],b->word[i],v->word[j],u_temp->word[j],temp_result);
	BN_WORD_copy(temp_result,v->word[j]);
	BN_WORD_add(u->word[j],u_temp->word[j],temp_result);
	BN_WORD_copy(temp_result,u->word[j]);
	mul_lo(v->word[j],n0_inverse,m->word[j]);
	__syncthreads();
	//need synchronization
	BN_WORD_copy(m->word[0],m->word[j]);
	mad_lo(n->word[j],m->word[j],v->word[j],u_temp->word[j],temp_result);
	BN_WORD_copy(temp_result,v->word[j]);
	BN_WORD_add(u->word[j],u_temp->word[j],temp_result);
	BN_WORD_copy(temp_result,u->word[j]);
	BN_WORD_copy(v->word[j],v_temp->word[j]);
	__syncthreads();
	//need synchronization
	BN_WORD_copy(v_temp->word[int_mod(j+1,wmax)],v->word[j]);
	BN_WORD_add(u->word[j],v->word[j],temp_result);
	BN_WORD_copy(temp_result,v->word[j]);
	if(BN_WORD_cmp(u->word[j],v->word[j])==1){
	    BN_WORD_setone(u->word[j]);
	    v->word[j]->carry=0;
	}
	else {
	    BN_WORD_setzero(u->word[j]);
	    v->word[j]->carry=0;
	}
	mad_hi(a->word[j],b->word[i],v->word[j],u_temp->word[j],temp_result);
	BN_WORD_copy(temp_result,v->word[j]);
	BN_WORD_add(u->word[j],u_temp->word[j],temp_result);
	BN_WORD_copy(temp_result,u->word[j]);
	mad_hi(n->word[j],m->word[j],v->word[j],u_temp->word[j],temp_result);
	BN_WORD_copy(temp_result,v->word[j]);
	BN_WORD_add(u->word[j],u_temp->word[j],temp_result);
	BN_WORD_copy(temp_result,u->word[j]);
    }
#ifdef CLOCKING
    if(j==0){
        stop=clock();
	sumtime=stop-start;
 	printf("loop_time:%ld\n",sumtime);
    }
#endif

#ifdef CLOCKING
    if(j==0){
        start=clock();
    }
#endif

    BN_WORD_copy(u->word[j],c->word[j]);
    __syncthreads();
	__syncthreads();
    any_value[j]=any(u);
    __syncthreads();
    while(any_value[j]==0){
        BN_WORD_copy(u->word[int_mod(j-1,wmax)],u_temp->word[j]);
	__syncthreads();
	BN_WORD_copy(u_temp->word[j],u->word[j]);
	if(j==0){
	    BN_WORD_setzero(u->word[j]);
	}
	BN_WORD_add(u->word[j],v->word[j],temp_result);
	BN_WORD_copy(temp_result,v->word[j]);
        if((BN_WORD_cmp(u->word[j],v->word[j])==2)||(BN_WORD_cmp(u->word[j],v->word[j])==0)){
            BN_WORD_setzero(u->word[j]);
            v->word[j]->carry=0;
        }
        else {
            BN_WORD_setone(u->word[j]);
            v->word[j]->carry=0;
        }
	BN_WORD_add(c->word[j],u->word[j],temp_result);
	BN_WORD_copy(temp_result,c->word[j]);
	__syncthreads();
	any_value[j]=any(u);
    }
    
    BN_WORD_copy(v->word[j],result->word[j]);
    __syncthreads();

#ifdef CLOCKING
    if(j==0){
        stop=clock();
	sumtime=stop-start;
	printf("carry_time:%ld\n",sumtime);
    }
#endif

#ifdef CLOCKING
    if(j==0){
        start=clock();
    }
#endif
    //need sy
    BN_WORD_copy(c->word[wmax-1],c->word[j]);
    while(BN_WORD_cmp(c->word[j],zero)==1){
        while((BN_NUM_cmp(result,n)==1)||(BN_NUM_cmp(result,n)==0)){
	    BN_NUM_sub_device(result,n,bn_temp_result);
	    BN_NUM_copy(bn_temp_result,result);
	}
	BN_WORD_sub(c->word[j],one,temp_result);
	BN_WORD_copy(temp_result,c->word[j]);
	BN_NUM_sub_device(zero_num,n,bn_temp_result);
	BN_NUM_add_device(bn_temp_result,result,bn_temp_result2);
	BN_NUM_copy(bn_temp_result2,result);
    }
    while((BN_NUM_cmp(result,n)==1)||(BN_NUM_cmp(result,n)==0)){
        BN_NUM_sub_device(result,n,bn_temp_result);
        BN_NUM_copy(bn_temp_result,result);
    }
#ifdef CLOCKING
    if(j==0){
        stop=clock();
	sumtime=stop-start;
        printf("end_time:%ld\n",sumtime);
    }
#endif
    BN_WORD_free_device(temp_result);
    BN_WORD_free_device(zero);
    BN_WORD_free_device(one);
    BN_NUM_free_device(bn_temp_result);
    BN_NUM_free_device(bn_temp_result2);
    BN_NUM_free_device(zero_num);
    
}
/*
__host__ int BN_NUM_R_inverse(const BN_NUM *n, BN_NUM *result){
    int wmax=n->wmax;
    int dmax=n->word[0]->dmax;
    int neg1=0, neg2=0, neg_temp=0;
    BN_NUM *R1, *R2, *temp_result, *mul_result,*zero, *one, *t1, *t2, *q;
    R1=BN_NUM_new(wmax,dmax);
    R2=BN_NUM_new(wmax,dmax);
    temp_result=BN_NUM_new(wmax,dmax);
    mul_result=BN_NUM_new(wmax,dmax);
    zero=BN_NUM_new(wmax,dmax);
    one=BN_NUM_new(wmax,dmax);
    t1=BN_NUM_new(wmax,dmax);
    t2=BN_NUM_new(wmax,dmax);
    q=BN_NUM_new(wmax,dmax);
    BN_NUM_copy(n,R1);
    BN_NUM_setzero(zero);
    BN_NUM_setone(one);
    BN_NUM_sub(zero,n,R2);
    while((BN_NUM_cmp(R2,n)==1)||BN_NUM_cmp(R2,n)==0){
        BN_NUM_sub(R2,n,temp_result);
        BN_NUM_copy(temp_result,R2);
    }
    printf("R1:\n");
    BN_NUM_print(R1);
    printf("R2:\n");
    BN_NUM_print(R2);
    BN_NUM_setzero(s2);
    BN_NUM_setone(t2);
    while(BN_NUM_cmp(R2,one)!=0){
        BN_NUM_div(R1,R2,q,temp_result);
        BN_NUM_copy(R2,R1);
        BN_NUM_copy(temp_result,R2);
        BN_NUM_mul(t2,q,mul_result);
        BN_NUM_sub(t1,mul_result,temp_result);
	if(BN_NUM_cmp(temp_result,t1)==1){
	    neg=0;
	}
        BN_NUM_copy(t2,t1);
        BN_NUM_copy(temp_result,t2);
    }
    BN_NUM_copy(t2,result);
    BN_NUM_sub(zero,n,R1);
    BN_NUM_mul_mod_host(R1,result,n,R2);
    BN_NUM_print(R2);
    return 0;
}
*/
__host__ int BN_NUM_mul_mod_host(const BN_NUM *a, const BN_NUM *b, const BN_NUM *n, BN_NUM *result){
    int wmax=a->wmax;
    int dmax=a->word[0]->dmax;
    int bit;
    BN_NUM *a_sub, *b_sub, *temp_result;
    a_sub=BN_NUM_new(wmax,dmax);
    b_sub=BN_NUM_new(wmax,dmax);
    temp_result=BN_NUM_new(wmax,dmax);
    BN_NUM_copy(a,a_sub);
    BN_NUM_copy(b,b_sub);
    while(BN_NUM_cmp(a_sub,n)==1){
        BN_NUM_sub(a_sub,n,temp_result);
        BN_NUM_copy(temp_result,a_sub);
    }
    if(BN_NUM_cmp(a_sub,n)==0){
        BN_NUM_setzero(result);
        return 0;
    }
    while(BN_NUM_cmp(b_sub,n)==1){
        BN_NUM_sub(b_sub,n,temp_result);
        BN_NUM_copy(temp_result,b_sub);
    }
    if(BN_NUM_cmp(b_sub,n)==0){
        BN_NUM_setzero(result);
        return 0;
    }
    BN_NUM_setzero(result);
    for(int i=wmax-1;i>=0;i--){
        for(int j=dmax-1;j>=0;j--){
            for(int k=sizeof(BN_ULONG)*8-1;k>=0;k--){
                bit=get_bit(b_sub->word[i]->d[j],k);
                BN_NUM_add(result,result,temp_result);
                if((BN_NUM_cmp(temp_result, result)==2)||(BN_NUM_cmp(temp_result,n)==1)||(BN_NUM_cmp(temp_result,n)==0)){
                    BN_NUM_sub(temp_result,n,result);
                }
                else {
                    BN_NUM_copy(temp_result,result);
                }
                if(bit==1){
                    BN_NUM_add(result,a_sub,temp_result);
                    if((BN_NUM_cmp(temp_result, result)==2)||(BN_NUM_cmp(temp_result,n)==1)
                                    ||(BN_NUM_cmp(temp_result,n)==0)){
                            BN_NUM_sub(temp_result,n,result);
                    }
                    else {
                            BN_NUM_copy(temp_result,result);
                    }
                }
            }
        }
    }
    return 0;
}


__host__ int BN_NUM_parallel_mont_exp(const BN_NUM *a, const BN_NUM *e, const BN_NUM *n,const int d,const int l,
                BN_NUM *result){
#ifdef TIMING
    time_t start, stop, sum_time;
#endif

#ifdef TIMING
    start = time(NULL);
#endif

    int *any_value;
    BN_WORD *n0_inverse, *zero_word, *bn_word_temp;
    BN_NUM *a_pro, *bn_temp, *R_pro, *zero, *one, *u, *u_temp,*v,*m, *c,*v_temp;
    hipMallocManaged((void **)&(any_value),d*sizeof(int));
    n0_inverse=BN_WORD_new(l);
    zero_word=BN_WORD_new(l);
    bn_word_temp=BN_WORD_new(l);
    a_pro=BN_NUM_new(d,l);
    bn_temp=BN_NUM_new(d,l);
    R_pro=BN_NUM_new(d,l);
    zero=BN_NUM_new(d,l);
    one=BN_NUM_new(d,l);
    u=BN_NUM_new(d,l);
    v=BN_NUM_new(d,l);
    m=BN_NUM_new(d,l);
    c=BN_NUM_new(d,l);
    u_temp=BN_NUM_new(d,l);
    v_temp=BN_NUM_new(d,l);
   // R_inverse=BN_NUM_new(d,l);
    BN_NUM_copy(a,a_pro);
    BN_NUM_setzero(zero);
    BN_NUM_setone(one);
    BN_WORD_setzero(zero_word);
    while(BN_NUM_cmp(a_pro,n)==1){
        BN_NUM_sub(a_pro,n,bn_temp);
        BN_NUM_copy(bn_temp,a_pro);
    }
    if(BN_NUM_cmp(a_pro,n)==0){
        BN_NUM_setzero(result);
        return 0;
    }
    BN_NUM_sub(zero,n,R_pro);

#ifdef TIMING
    stop= time(NULL);
    sum_time= difftime(stop,start);
    printf("pro_calculate:%d s\n", sum_time);
#endif

#ifdef TIMING
    start= time(NULL);
#endif

    BN_NUM_mul_mod_host(a_pro,R_pro,n,bn_temp);

#ifdef TIMING
    stop= time(NULL);
    sum_time= difftime(stop,start);
    printf("mul_mod_host:%d s\n", sum_time);
#endif

    BN_NUM_copy(bn_temp,a_pro);

#ifdef TIMING
    start= time(NULL);
#endif

    BN_WORD_inverse(n->word[0],n0_inverse);
printf("1\n");
#ifdef TIMING
    stop= time(NULL);
    sum_time= difftime(stop,start);
    printf("inverse:%d s\n", sum_time);
#endif

    BN_WORD_sub(zero_word,n0_inverse,bn_word_temp);
    BN_WORD_copy(bn_word_temp,n0_inverse);
    BN_NUM_copy(R_pro,result);

#ifdef TIMING
    start= time(NULL);
#endif

    for(int i=d-1;i>=0;i--){
        for(int j=l-1;j>=0;j--){
            for(int k=sizeof(BN_ULONG)*8-1;k>=0;k--){
                parallel_mont_mul<<<1,d>>>(result,result,n,d,l,n0_inverse,bn_temp,u,u_temp,v,m,c,v_temp, any_value);
                hipDeviceSynchronize();
                BN_NUM_copy(bn_temp,result);
                if(get_bit(e->word[i]->d[j],k)==(BN_ULONG)1){
                    parallel_mont_mul<<<1,d>>>(result,a_pro,n,d,l,n0_inverse,bn_temp,u,u_temp,v,m,c,v_temp, any_value);
                    hipDeviceSynchronize();
                    BN_NUM_copy(bn_temp,result);
//		    printf("result:\n");
//		    BN_NUM_print(result);
                }
            }
        }
    }

#ifdef TIMING
    stop= time(NULL);
    sum_time= difftime(stop,start);
    printf("loop:%d s\n", sum_time);
#endif

    parallel_mont_mul<<<1,d>>>(result,one,n,d,l,n0_inverse,bn_temp,u,u_temp,v,m,c,v_temp, any_value);
    hipDeviceSynchronize();
    BN_NUM_copy(bn_temp,result);
    return 0;
}

