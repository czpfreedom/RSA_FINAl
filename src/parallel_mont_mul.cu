#include "hip/hip_runtime.h"
#include "bn_word_operation.h"
#include "pseudo.h"
#include "parallel_mont_mul.h"

__host__ __device__ int_mod(const int a,const int b){
    return a%b;
}


__global__ void parallel_mont_mul(const BN_NUM *a,const BN_NUM *b,const BN_WORD_NUM *n,const int wmax,const int dmax,const BN_WORD *n0_inverse,
		BN_NUM *result, BN_NUM *u, BN_NUM *u_temp,BN_NUM *v, BN_NUM *m, BN_NUM *c, BN_NUM *t){
    int j=threadIdx.x+blockIdx.x*blockDim.x;
    BN_WORD_setzero(u->word[j]);
    BN_WORD_setzero(v->word[j]);
//need error_check
    for(int i=0;i<wmax;i++){
        mad_lo(a->word[j],b->word[i],v->word[j],u_temp->word[j],v->word[j]);
	BN_WORD_add(u->word[j],u_temp->word[j],u->word[j]);
	mul_lo(u->word[j],n0_inverse,n->word[j]);
	//need synchronization
	BN_WORD_copy(m->word[0],m->word[j]);
	mad_lo(n->word[i],m->word[j],v->word[j],u_temp->word[j],v->word[j]);
	BN_WORD_add(u->word[j],u_temp->word[j],u->word[j]);
	//need synchronization
	BN_WORD_copy(v->word[j],v->word[int_mod(j+1,wmax)]);
	BN_WORD_add(u->word[j],v->word[j],v->word[j]);
	if(v->word[j]->carry==0){
	    BN_WORD_setzero(u->word[j]);
	    v->word[j]->carry=0;
	}
	else {
	    BN_WORD_setone(u->word[j]);
	    v->word[j]->carry=0;
	}
	mad_hi(a->word[j],b->word[i],v->word[j],u_temp->word[j],v->word[j]);
	BN_WORD_add(u->word[j],u_temp->word[j],u->word[j]);
	mad_hi(n->word[i],m->word[j],v->word[j],u_temp->word[j],v->word[j]);
	BN_WORD_add(u->word[j],u_temp->word[j],u->word[j]);
    }
    BN_WORD_copy(u->word[j],c->word[j]);
    while(any(u->word[j])==0){
        BN_WORD_copy(u->word[int_mod(j-1)],u->word[j]);
	if(j==0){
	    BN_WORD_setzero(u->word[j]);
	}
	BN_WORD_add(u->word[j],v->word[j],v->word[j]);
        if(v->word[j]->carry==0){
            BN_WORD_setzero(u->word[j]);
            v->word[j]->carry=0;
        }
        else {
            BN_WORD_setone(u->word[j]);
            v->word[j]->carry=0;
        }
	BN_WORD_add(c->word[j],u->word[j],c->word[j]);
    }
    BN_WORD_copy(v->word[j],t->word[j]);
    //need sy
    BN_WORD_copy(c->word[wmax-1],c->word[j]);
    if(any(c->word[j])){
        BN_NUM_sub(t,n,t);
    }
}
