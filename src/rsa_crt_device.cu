#include "hip/hip_runtime.h"
#include "rsa_crt.h"
#include "rsa_final.h"
#include "stdlib.h"
#include "string.h"

namespace namespace_rsa_final{

__device__ int  GPU_WORD_parallel_Mon(BN_PART *A, BN_PART *B, BN_PART *N, BN_PART n0_inverse, BN_PART *M, BN_PART *U, BN_PART *V, BN_PART *C, BN_PART *result, int thread_id){
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;
    p_a=A[thread_id];
    p_n=N[thread_id];
    p_u=0;
    p_v=0;
    M[thread_id]=0;
    C[thread_id]=0;

    for(int i=0;i<WARP_SIZE;i++){
        p_b=B[i];
        BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mul_lo(p_v,n0_inverse,M[thread_id]);
        p_m=M[0];
        BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        V[thread_id]=p_v;
        p_v=V[int_mod(thread_id+1,WARP_SIZE)];
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
    }
    C[thread_id]=p_u;
    U[thread_id]=p_u;
    while(BN_PART_any(U,WARP_SIZE)==0){
        p_u=U[int_mod(thread_id-1,WARP_SIZE)];
        if(thread_id==0){
            p_u=0;
	}
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        C[thread_id]=C[thread_id]+p_u;
        U[thread_id]=p_u;
    }
    result[thread_id]=p_v;

    return 1;
};

__device__ int GPU_WORD_delete_carry(BN_PART *result, BN_PART *N, BN_PART c){
    GPU_WORD gw_result, gw_n;
    gw_result.setzero();
    gw_n.setzero();
    for(int i=0;i<WARP_SIZE;i++){
        gw_result.m_data[i]=result[i];
        gw_n.m_data[i]=N[i];
    }
    gw_result.m_data[WARP_SIZE]=c;
    gw_result.m_top=WARP_SIZE+1;
    gw_n.m_top=WARP_SIZE;
    gw_result.check_top();
    gw_result=gw_result%gw_n;
    for(int i=0;i<WARP_SIZE;i++){
        result[i]=gw_result.m_data[i];
    }
    return 1;
}	

__global__ void  GPU_WORD_mod_mul(BN_PART *A, BN_PART *B, BN_PART *N, BN_PART n0_inverse, BN_PART *result){
   int j=threadIdx.x;

   __shared__ BN_PART M[WARP_SIZE];
   __shared__ BN_PART U[WARP_SIZE];
   __shared__ BN_PART V[WARP_SIZE];
   __shared__ BN_PART C[WARP_SIZE];

   __shared__ BN_PART ONE[WARP_SIZE];
   ONE[j]=0;
   ONE[0]=1;

   GPU_WORD_parallel_Mon(A,B,N,n0_inverse,M,U,V,C,result,j);

   if(j==WARP_SIZE-1){
       GPU_WORD_delete_carry(result, N, C[j]);
   }
   __syncthreads();

   GPU_WORD_parallel_Mon(result,ONE,N,n0_inverse,M,U,V,C,result,j);

   if(j==WARP_SIZE-1){
       GPU_WORD_delete_carry(result, N, C[j]);
   }
   __syncthreads();

}

__global__ void GPU_WORD_mod_exp( BN_PART *A, BN_PART *E , int E_bits, BN_PART *mR, BN_PART *N , BN_PART n0_inverse, BN_PART *result){

    int i=threadIdx.x/WARP_SIZE;
    int j=threadIdx.x%WARP_SIZE;

    __shared__ BN_PART M1[WARP_SIZE];
    __shared__ BN_PART U1[WARP_SIZE];
    __shared__ BN_PART V1[WARP_SIZE];
    __shared__ BN_PART C1[WARP_SIZE];

    __shared__ BN_PART M2[WARP_SIZE];
    __shared__ BN_PART U2[WARP_SIZE];
    __shared__ BN_PART V2[WARP_SIZE];
    __shared__ BN_PART C2[WARP_SIZE];

    __shared__ BN_PART R[WARP_SIZE];
    __shared__ BN_PART ONE[WARP_SIZE];
    __shared__ BN_PART S1[WARP_SIZE];
    __shared__ BN_PART S2[WARP_SIZE];
    __shared__ BN_PART S3[WARP_SIZE];

    __shared__ BN_PART R2[WARP_SIZE];

    int k,k_i,k_j;
   
    if(i==0){
        ONE[j]=0;
 	ONE[0]=1;   
    }
    else{
        R[j]=mR[j];
    }

    __syncthreads();

    if(i==0){
        S1[j]=A[j];
    }
    else{
        S2[j]=A[j];
    }

    __syncthreads();

   
    for(k=0;k<E_bits; k++){
        k_i=k/(sizeof(BN_PART)*8);
        k_j=k%(sizeof(BN_PART)*8);
        if(i==0){
	    GPU_WORD_parallel_Mon(R,S1,N,n0_inverse,M1,U1,V1,C1,R2,j);
 	}
 	else{
 	    GPU_WORD_parallel_Mon(S2,S2,N,n0_inverse,M2,U2,V2,C2,S3,j);
 	}
 	__syncthreads();   
 	if(i==0){
 	    if(j==WARP_SIZE-1){
 	        GPU_WORD_delete_carry(R2, N, C1[j]);
	    }
 	}
 	else{
 	    if(j==WARP_SIZE-1){
 	        GPU_WORD_delete_carry(S3, N, C2[j]);
	    }       
 	}
 	__syncthreads();
 	if(i==0){
            if(BN_PART_get_bit(E[k_i],k_j)==1){
     	        R[j]=R2[j];
 	    }
 	}
 	else{
            S2[j]=S3[j];
 	}
 	__syncthreads();   
	S1[j]=S2[j];
	__syncthreads();   
    }

    if(i==0){
        GPU_WORD_parallel_Mon(R,ONE,N,n0_inverse,M1,U1,V1,C1,R2,j);
        if(j==WARP_SIZE-1){
	    GPU_WORD_delete_carry(R2, N, C1[j]);
 	}
	result[j]=R2[j];
    }
}

__global__ void GPU_WORD_ARRAY_mod_exp( BN_PART *A, BN_PART *E , int E_bits, BN_PART *mR, BN_PART *N , BN_PART n0_inverse, BN_PART *result){

    int bid=blockIdx.x;
    int i=threadIdx.x/WARP_SIZE;
    int j=threadIdx.x%WARP_SIZE;

    __shared__ BN_PART M1[WARP_SIZE];
    __shared__ BN_PART U1[WARP_SIZE];
    __shared__ BN_PART V1[WARP_SIZE];
    __shared__ BN_PART C1[WARP_SIZE];

    __shared__ BN_PART M2[WARP_SIZE];
    __shared__ BN_PART U2[WARP_SIZE];
    __shared__ BN_PART V2[WARP_SIZE];
    __shared__ BN_PART C2[WARP_SIZE];

    __shared__ BN_PART R[WARP_SIZE];
    __shared__ BN_PART ONE[WARP_SIZE];
    __shared__ BN_PART S1[WARP_SIZE];
    __shared__ BN_PART S2[WARP_SIZE];
    __shared__ BN_PART S3[WARP_SIZE];

    __shared__ BN_PART R2[WARP_SIZE];

    int k,k_i,k_j;
   
    if(i==0){
        ONE[j]=0;
 	ONE[0]=1;   
    }
    else{
        R[j]=mR[j];
    }

    __syncthreads();

    if(i==0){
        S1[j]=A[bid*WARP_SIZE+j];
    }
    else{
        S2[j]=A[bid*WARP_SIZE+j];
    }

    __syncthreads();

   
    for(k=0;k<E_bits; k++){
        k_i=k/(sizeof(BN_PART)*8);
        k_j=k%(sizeof(BN_PART)*8);
        if(i==0){
	    GPU_WORD_parallel_Mon(R,S1,N,n0_inverse,M1,U1,V1,C1,R2,j);
 	}
 	else{
 	    GPU_WORD_parallel_Mon(S2,S2,N,n0_inverse,M2,U2,V2,C2,S3,j);
 	}
 	__syncthreads();   
 	if(i==0){
 	    if(j==WARP_SIZE-1){
 	        GPU_WORD_delete_carry(R2, N, C1[j]);
	    }
 	}
 	else{
 	    if(j==WARP_SIZE-1){
 	        GPU_WORD_delete_carry(S3, N, C2[j]);
	    }       
 	}
 	__syncthreads();
 	if(i==0){
            if(BN_PART_get_bit(E[k_i],k_j)==1){
     	        R[j]=R2[j];
 	    }
 	}
 	else{
            S2[j]=S3[j];
 	}
 	__syncthreads();   
	S1[j]=S2[j];
	__syncthreads();   
    }

    if(i==0){
        GPU_WORD_parallel_Mon(R,ONE,N,n0_inverse,M1,U1,V1,C1,R2,j);
        if(j==WARP_SIZE-1){
	    GPU_WORD_delete_carry(R2, N, C1[j]);
 	}
        result[bid*WARP_SIZE+j]=R2[j];
    }

}



}
