#include "hip/hip_runtime.h"
#include "pseudo.h"
#include "stdio.h"


__device__ int mul_lo (const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    int dmax=a->dmax;
    if ((a->carry!=0)||(b->carry!=0)){
	return -2;
    }
    if((b->dmax!=dmax)||(result->dmax!=dmax)){
	return -1;
    }
    result->carry=0;
    BN_WORD_setzero(result);
    BN_WORD *result_temp;
    result_temp=BN_WORD_new_device(dmax*2);
    BN_WORD_setzero(result_temp);
    BN_WORD_mul(a,b,result_temp);
    for(int i=0;i<dmax;i++){
        result->d[i]=result_temp->d[i];
    }
    return 0;
}

 __device__ int mad_lo (const BN_WORD *a, const BN_WORD *b, const BN_WORD *c, BN_WORD *result_u, BN_WORD *result_v){
    int dmax=a->dmax;
    if ((a->carry!=0)||(b->carry!=0)){
	return -2;
    }
    if((b->dmax!=dmax)||(c->dmax!=dmax)||(result_u->dmax!=dmax)||(result_v->dmax!=dmax)){
        return -1;
    }
    BN_WORD *temp_result;
    temp_result=BN_WORD_new_device(dmax);
    BN_WORD_setzero(result_u);
    BN_WORD_setzero(result_v);
    mul_lo(a,b,result_v);
    BN_WORD_add(result_v, c, temp_result);
    BN_WORD_copy(temp_result,result_v);
    if(BN_WORD_cmp(c,result_v)==1){
        result_u->carry=0;
        result_u->d[0]=1;
        for(int i=1;i<dmax;i++){
            result_u->d[i]=0;
        }
    }
    else{
        result_u->carry=0;
	BN_WORD_setzero(result_u);
    }
    result_v->carry=0;
    return 0;
}

__device__ int mad_hi(const BN_WORD *a, const BN_WORD *b, const BN_WORD *c, BN_WORD *result_u, BN_WORD *result_v){
    int dmax=a->dmax;
    if ((a->carry!=0)||(b->carry!=0)){
        return -2;
    }
    if((b->dmax!=dmax)||(c->dmax!=dmax)||(result_u->dmax!=dmax)||(result_v->dmax!=dmax)){
        return -1;
    }
    BN_WORD *result, *temp_result,*temp_2_result;
    result=BN_WORD_new_device(dmax*2);
    temp_result=BN_WORD_new_device(dmax);
    temp_2_result=BN_WORD_new_device(dmax*2);
    BN_WORD_setzero(result);
    BN_WORD_mul(a,b,result);
    BN_WORD_right_shift(result,temp_2_result,dmax);
    BN_WORD_copy(temp_2_result,result);
    BN_WORD_setzero(result_u);
    BN_WORD_setzero(result_v);
    for(int i=0;i<dmax;i++){
        result_v->d[i]=result->d[i];
    }
    BN_WORD_add(result_v,c,temp_result);
    BN_WORD_copy(temp_result,result_v);
    if(BN_WORD_cmp(c,result_v)==1){
        BN_WORD_setone(result_u);
    }
    BN_WORD_free_device(result);
    return 0;
}

__device__ int any(BN_NUM *a){
    int cmp;
    BN_NUM *zero;
    zero=BN_NUM_new_device(a->wmax,a->word[0]->dmax);
    cmp=BN_NUM_cmp(a,zero);
    if(cmp==0)
	    return 1;
    else return 0;
}

__global__ void mul_lo_global(const BN_WORD *a, const BN_WORD *b, BN_WORD*result){
    mul_lo(a,b,result);
}

__global__ void mad_lo_global(const BN_WORD *a, const BN_WORD *b, const BN_WORD *c, BN_WORD *result_u, BN_WORD *result_v){
    mad_lo(a,b,c,result_u,result_v);
}

__global__ void mad_hi_global(const BN_WORD *a, const BN_WORD *b, const BN_WORD *c, BN_WORD *result_u, BN_WORD *result_v){
    mad_hi(a,b,c,result_u,result_v);
}

/*
__host__ __device__ void mad_hi(BN_WORD *a, BN_WORD *b, BN_WORD *c, BN_WORD *result_u, BN_WORD *result_v,BN_WORD *mul_word_result,
		BN_WORD *mid_value1, BN_WORD *mid_value2, BN_WORD *mid_value3, BN_WORD *mid_value4, BN_WORD *mid_value5,int *mul_return_value,
		int *add_return_value, int *mid_return_value, int * return_value){
    BN_WORD_mul(a,b,result_u,result_v,mul_word_result, mid_value1, mid_value2, mid_value3, mid_value4, mid_value5,mul_return_value,add_return_value,
		    mid_return_value);
    BN_WORD_left_shift(result_u,mid_value1,DMAX-1,mid_return_value);
    BN_WORD_right_shift(result_v,mid_value3,1,mid_return_value);
    BN_WORD_right_shift(result_u,mid_value4,1,mid_return_value);
    BN_WORD_add(mid_value1,result_v,mid_value2,mid_return_value);
    BN_WORD_add(mid_value2,c,result_v,mid_return_value);
    BN_WORD_setzero(mid_value2);
    if(*(result_v->carry)!=0){
        *(result_v->carry)=0;
	BN_WORD_setone(mid_value2);
    }
    BN_WORD_add(mid_value4,mid_value2,result_u,mid_return_value);
    *return_value=0;
    return;
}
*/
