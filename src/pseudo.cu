#include "hip/hip_runtime.h"
#include "pseudo.h"
#include "stdio.h"

#ifndef DMAX
#define DMAX 32
#endif

__device__ int mul_lo (BN_WORD *a, BN_WORD *b, BN_WORD *result){
    int dmax=a->dmax;
    if ((a->carry!=0)||(b->carry!=0)){
	return -2;
    }
    if((b->dmax!=dmax)||(result->dmax!=dmax)){
	return -1;
    }
    result->carry=0;
    BN_WORD_setzero(result);
    result->d[0]=a->d[0]*b->d[0];
    for(int i=1;i<dmax;i++){
        result->d[i]=0;
    }
    return 0;
}

 __device__ int mad_lo (BN_WORD *a, BN_WORD *b, BN_WORD *c, BN_WORD *result_u, BN_WORD *result_v){
    int dmax=a->dmax;
    BN_WORD *temp_result;
    temp_result=BN_WORD_new_device(dmax);
    if ((a->carry!=0)||(b->carry!=0)){
	return -2;
    }
    if((b->dmax!=dmax)||(c->dmax!=dmax)||(result_u->dmax!=dmax)||(result_v->dmax!=dmax)){
        return -1;
    }
    BN_WORD_setzero(temp_result);
    BN_WORD_setzero(result_u);
    BN_WORD_setzero(result_v);
    mul_lo(a,b,result_v);
    BN_WORD_add(result_v, c, temp_result);
    BN_WORD_copy(temp_result,result_v);
    if(result_v->carry==1){
        result_u->carry=0;
        result_u->d[0]=1;
        for(int i=1;i<dmax;i++){
            result_u->d[i]=0;
        }
    }
    else{
        result_u->carry=0;
	BN_WORD_setzero(result_u);
    }
    result_v->carry=0;
    BN_WORD_free_device(temp_result);
    return 0;
}

__device__ int mad_hi(BN_WORD *a, BN_WORD *b, BN_WORD *c, BN_WORD *result_u, BN_WORD *result_v){
    int dmax=a->dmax;
    if ((a->carry!=0)||(b->carry!=0)){
        return -2;
    }
    if((b->dmax!=dmax)||(c->dmax!=dmax)||(result_u->dmax!=dmax)||(result_v->dmax!=dmax)){
        return -1;
    }
    BN_WORD *result, *temp_result,*c_2dmax;
    result=BN_WORD_new_device(dmax*2);
    temp_result=BN_WORD_new_device(dmax*2);
    c_2dmax=BN_WORD_new_device(dmax*2);
    BN_WORD_setzero(result);
    BN_WORD_setzero(temp_result);
    BN_WORD_setzero(c_2dmax);
    BN_WORD_mul(a,b,result);
    BN_WORD_right_shift(result,temp_result,1);
    BN_WORD_copy(temp_result,result);
    for(int i=0;i<dmax;i++){
        c_2dmax->d[i]=c->d[i];
    }
    for(int i=dmax;i<dmax*2;i++){
        c_2dmax->d[i]=0;
    }
    BN_WORD_setzero(temp_result);
    BN_WORD_setzero(result_u);
    BN_WORD_setzero(result_v);
    BN_WORD_add(result,c_2dmax,temp_result);
    for(int i=0;i<dmax;i++){
        result_v->d[i]=temp_result->d[i];
    }
    for(int i=0;i<dmax;i++){
        result_u->d[i]=temp_result->d[i+dmax];
    }
    BN_WORD_free_device(result);
    BN_WORD_free_device(temp_result);
    BN_WORD_free_device(c_2dmax);
    return 0;
}

__global__ void mad_lo_global(BN_WORD *a, BN_WORD *b, BN_WORD *c, BN_WORD *result_u, BN_WORD *result_v){
    mad_lo(a,b,c,result_u,result_v);
}

__global__ void mad_hi_global(BN_WORD *a, BN_WORD *b, BN_WORD *c, BN_WORD *result_u, BN_WORD *result_v){
    printf("start\n");
    mad_hi(a,b,c,result_u,result_v);
}

/*
__host__ __device__ void mad_hi(BN_WORD *a, BN_WORD *b, BN_WORD *c, BN_WORD *result_u, BN_WORD *result_v,BN_WORD *mul_word_result,
		BN_WORD *mid_value1, BN_WORD *mid_value2, BN_WORD *mid_value3, BN_WORD *mid_value4, BN_WORD *mid_value5,int *mul_return_value,
		int *add_return_value, int *mid_return_value, int * return_value){
    BN_WORD_mul(a,b,result_u,result_v,mul_word_result, mid_value1, mid_value2, mid_value3, mid_value4, mid_value5,mul_return_value,add_return_value,
		    mid_return_value);
    BN_WORD_left_shift(result_u,mid_value1,DMAX-1,mid_return_value);
    BN_WORD_right_shift(result_v,mid_value3,1,mid_return_value);
    BN_WORD_right_shift(result_u,mid_value4,1,mid_return_value);
    BN_WORD_add(mid_value1,result_v,mid_value2,mid_return_value);
    BN_WORD_add(mid_value2,c,result_v,mid_return_value);
    BN_WORD_setzero(mid_value2);
    if(*(result_v->carry)!=0){
        *(result_v->carry)=0;
	BN_WORD_setone(mid_value2);
    }
    BN_WORD_add(mid_value4,mid_value2,result_u,mid_return_value);
    *return_value=0;
    return;
}
*/
