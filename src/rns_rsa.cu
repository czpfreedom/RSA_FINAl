#include "hip/hip_runtime.h"
#include "rns_rsa.h"
#include "iostream"
#include "bn_word_parallel_mont_exp.h"


using namespace std;

__host__ RSA_N *RSA_N_new(int dmax){
    RSA_N *rsa_n;
    rsa_n=(RSA_N*)malloc(sizeof(RSA_N));
    rsa_n->n=BN_WORD_new(dmax);
    rsa_n->p=BN_WORD_new(dmax);
    rsa_n->q=BN_WORD_new(dmax);
    return rsa_n;
}

__host__ int RSA_N_free(RSA_N *rsa_n){
    BN_WORD_free(rsa_n->n);
    BN_WORD_free(rsa_n->p);
    BN_WORD_free(rsa_n->q);
    free(rsa_n);
    return 0;
}

__host__ int RSA_N_print(RSA_N *rsa_n){
   printf("p:\n");
   BN_WORD_print(rsa_n->p);
   printf("q:\n");
   BN_WORD_print(rsa_n->q);
   printf("n:\n");
   BN_WORD_print(rsa_n->n);
   return 0;
}

__host__ __device__ int RNS_WORD_BN_WORD_transform(RNS_WORD a, int bits, BN_WORD *result){
    int shift_num=bits/(sizeof(BN_PART)*8);
    int shift_bits=bits%(sizeof(BN_PART)*8);
    BN_WORD_setzero(result);
    if(shift_bits==0){
        result->d[shift_num-1]=a;
    }
    else{
        result->d[shift_num-1]=a<<(shift_bits);
	result->d[shift_num]=a>>(sizeof(RNS_WORD)*8-shift_bits);
    }
    return 0;
}

__host__ int BN_WORD_RNS_WORD_mod (BN_WORD *a, RNS_WORD b, RNS_WORD &c){
    BN_WORD *a_temp;
    BN_WORD *b_temp;
    a_temp=BN_WORD_new(a->dmax);
    b_temp=BN_WORD_new(a->dmax);
    BN_WORD_copy(a,a_temp);
    for(int i=(sizeof(BN_PART)*8*(a->dmax));i>=(sizeof(RNS_WORD)*8);i--){
        RNS_WORD_BN_WORD_transform(b,i,b_temp);
	while((BN_WORD_cmp(a_temp,b_temp)==1)||(BN_WORD_cmp(a_temp,b_temp)==0)){
	    BN_WORD_sub(a_temp,b_temp,a_temp);
	}
    }
    c=a_temp->d[0];
    BN_WORD_free(a_temp);
    BN_WORD_free(b_temp);
    return 0;
}

__device__ int BN_WORD_RNS_WORD_mod_device (BN_WORD *a, RNS_WORD b, RNS_WORD &c){
    BN_WORD *a_temp;
    BN_WORD *b_temp;
    a_temp=BN_WORD_new_device(a->dmax);
    b_temp=BN_WORD_new_device(a->dmax);
    BN_WORD_copy(a,a_temp);
    for(int i=(sizeof(BN_PART)*8*(a->dmax));i>=(sizeof(RNS_WORD)*8);i--){
        RNS_WORD_BN_WORD_transform(b,i,b_temp);
	while((BN_WORD_cmp(a_temp,b_temp)==1)||(BN_WORD_cmp(a_temp,b_temp)==0)){
	    BN_WORD_sub(a_temp,b_temp,a_temp);
	}
    }
    c=a_temp->d[0];
    BN_WORD_free_device(a_temp);
    BN_WORD_free_device(b_temp);
    return 0;
}

__host__ int RNS_WORD_mod_inverse (RNS_WORD a, RNS_WORD n, RNS_WORD &a_inverse){
    long s1, s2, q, r1, r2,temp;
    r1=(long)n;
    r2=(long)a;
    s1=0;
    s2=1;
    while(r2!=1){
       q=r1/r2;
       temp=r2;
       r2=r1%r2;
       r1=temp;
       temp=s2;
       s2=s1-q*s2;
       s1=temp;
    }
    while(s2<0){
       s2=s2+(long)n;
    }
    s2=s2%n;
    a_inverse=(RNS_WORD)s2;
    return 0;
}

__host__ RNS_N:: RNS_N(RSA_N *rsa_n){
    m_rsa_n=rsa_n;
    m_base_num=m_rsa_n->n->dmax;
    hipMallocManaged((void**)&(m_m1),BASE_MAX*sizeof(RNS_WORD));
    hipMallocManaged((void**)&(m_m2),BASE_MAX*sizeof(RNS_WORD));
    m_M1=BN_WORD_new(m_base_num);
    m_M2=BN_WORD_new(m_base_num);
    m_M1_n=BN_WORD_new(m_base_num);
    m_M2_n=BN_WORD_new(m_base_num);
    m_M1_i=(BN_WORD**)malloc(m_base_num*sizeof(BN_WORD*));
    m_M2_i=(BN_WORD**)malloc(m_base_num*sizeof(BN_WORD*));
    m_M1_red_i=(BN_WORD**)malloc(m_base_num*sizeof(BN_WORD*));
    m_M2_red_i=(BN_WORD**)malloc(m_base_num*sizeof(BN_WORD*));
    for(int i=0;i<m_base_num;i++){
	m_M1_i[i]=BN_WORD_new(m_base_num);
    }
    for(int i=0;i<m_base_num;i++){
	m_M2_i[i]=BN_WORD_new(m_base_num);
    }
    for(int i=0;i<m_base_num;i++){
        m_M1_red_i[i]=BN_WORD_new(m_base_num);
    }
    for(int i=0;i<m_base_num;i++){
	m_M2_red_i[i]=BN_WORD_new(m_base_num);
    }
    hipMallocManaged((void**)&(m_d),m_base_num*sizeof(RNS_WORD));
    hipMallocManaged((void**)&(m_e),m_base_num*sizeof(RNS_WORD));
    hipMallocManaged((void**)&(m_a),m_base_num*m_base_num*sizeof(RNS_WORD));
    hipMallocManaged((void**)&(m_a_2),m_base_num*sizeof(RNS_WORD));
    hipMallocManaged((void**)&(m_b),m_base_num*m_base_num*sizeof(RNS_WORD));
    hipMallocManaged((void**)&(m_b_2),m_base_num*sizeof(RNS_WORD));
    hipMallocManaged((void**)&(m_c),m_base_num*sizeof(RNS_WORD));
    RNS_WORD M, M_inverse, p_temp, M1_i_inverse,M1_i_m1_i, M2_i_inverse,M2_i_m2_i; 
    BN_WORD *temp_result, *bn_m, *q, *r, *bn_M1_i_inverse, *bn_M2_i_inverse;
    temp_result=BN_WORD_new(m_base_num);
    bn_m=BN_WORD_new(m_base_num);
    q=BN_WORD_new(m_base_num);
    r=BN_WORD_new(m_base_num);
    bn_M1_i_inverse=BN_WORD_new(m_base_num);
    bn_M2_i_inverse=BN_WORD_new(m_base_num);

    m_m1[0]=0xffffffef;
    m_m1[1]=0xffffffbf;
    m_m1[2]=0xffffff9d;
    m_m1[3]=0xffffff95;
    m_m1[4]=0xffffff79;
    m_m1[5]=0xffffff67;
    m_m1[6]=0xffffff47;
    m_m1[7]=0xffffff2f;
    m_m1[8]=0xfffffef5;
    m_m1[9]=0xfffffed5;
    m_m1[10]=0xfffffec5;
    m_m1[11]=0xfffffe9f;
    m_m1[12]=0xfffffe8f;
    m_m1[13]=0xfffffe7d;
    m_m1[14]=0xfffffe5d;
    m_m1[15]=0xfffffe2d;
    m_m1[16]=0xfffffe1d;
    m_m1[17]=0xfffffdf1;
    m_m1[18]=0xfffffd8b;
    m_m1[19]=0xfffffd85;
    m_m1[20]=0xfffffd81;
    m_m1[21]=0xfffffd7b;
    m_m1[22]=0xfffffd6f;
    m_m1[23]=0xfffffd5b;
    m_m1[24]=0xfffffd3f;
    m_m1[25]=0xfffffd37;
    m_m1[26]=0xfffffd19;
    m_m1[27]=0xfffffccd;
    m_m1[28]=0xfffffcaf;
    m_m1[29]=0xfffffca9;
    m_m1[30]=0xfffffc9b;
    m_m1[31]=0xfffffc65;
    m_m1[32]=0xfffff9fd;
    m_m1[33]=0xfffff9e9;
    m_m1[34]=0xfffff9e5;
    m_m1[35]=0xfffff9d9;
    m_m1[36]=0xfffff9bb;
    m_m1[37]=0xfffff9b3;
    m_m1[38]=0xfffff9af;
    m_m1[39]=0xfffff9a9;
    m_m1[40]=0xfffff9a7;
    m_m1[41]=0xfffff99b;
    m_m1[42]=0xfffff989;
    m_m1[43]=0xfffff971;
    m_m1[44]=0xfffff96d;
    m_m1[45]=0xfffff961;
    m_m1[46]=0xfffff94d;
    m_m1[47]=0xfffff919;
    m_m1[48]=0xfffff8ef;
    m_m1[49]=0xfffff8d5;
    m_m1[50]=0xfffff8d1;
    m_m1[51]=0xfffff8a5;
    m_m1[52]=0xfffff887;
    m_m1[53]=0xfffff871;
    m_m1[54]=0xfffff863;
    m_m1[55]=0xfffff853;
    m_m1[56]=0xfffff841;
    m_m1[57]=0xfffff83b;
    m_m1[58]=0xfffff80f;
    m_m1[59]=0xfffff803;
    m_m1[60]=0xfffff7ed;
    m_m1[61]=0xfffff7d3;
    m_m1[62]=0xfffff7c9;
    m_m1[63]=0xfffff7a9;


    m_m2[0]=0xfffffc5f;
    m_m2[1]=0xfffffc41;
    m_m2[2]=0xfffffc19;
    m_m2[3]=0xfffffbe3;
    m_m2[4]=0xfffffbdd;
    m_m2[5]=0xfffffbd7;
    m_m2[6]=0xfffffbc9;
    m_m2[7]=0xfffffbab;
    m_m2[8]=0xfffffba1;
    m_m2[9]=0xfffffb93;
    m_m2[10]=0xfffffb89;
    m_m2[11]=0xfffffb71;
    m_m2[12]=0xfffffb69;
    m_m2[13]=0xfffffb53;
    m_m2[14]=0xfffffb47;
    m_m2[15]=0xfffffb39;
    m_m2[16]=0xfffffb1b;
    m_m2[17]=0xfffffaf7;
    m_m2[18]=0xfffffaf1;
    m_m2[19]=0xfffffad9;
    m_m2[20]=0xfffffad3;
    m_m2[21]=0xfffffacf;
    m_m2[22]=0xfffffabd;
    m_m2[23]=0xfffffab1;
    m_m2[24]=0xfffffa97;
    m_m2[25]=0xfffffa7f;
    m_m2[26]=0xfffffa57;
    m_m2[27]=0xfffffa51;
    m_m2[28]=0xfffffa4f;
    m_m2[29]=0xfffffa3d;
    m_m2[30]=0xfffffa21;
    m_m2[31]=0xfffffa07;
    m_m2[32]=0xfffff79f;
    m_m2[33]=0xfffff791;
    m_m2[34]=0xfffff78b;
    m_m2[35]=0xfffff71b;
    m_m2[36]=0xfffff6f5;
    m_m2[37]=0xfffff6f1;
    m_m2[38]=0xfffff6e9;
    m_m2[39]=0xfffff6df;
    m_m2[40]=0xfffff6cb;
    m_m2[41]=0xfffff6c1;
    m_m2[42]=0xfffff6bb;
    m_m2[43]=0xfffff6a7;
    m_m2[44]=0xfffff6a3;
    m_m2[45]=0xfffff6a1;
    m_m2[46]=0xfffff69d;
    m_m2[47]=0xfffff697;
    m_m2[48]=0xfffff68f;
    m_m2[49]=0xfffff661;
    m_m2[50]=0xfffff65b;
    m_m2[51]=0xfffff649;
    m_m2[52]=0xfffff635;
    m_m2[53]=0xfffff623;
    m_m2[54]=0xfffff60d;
    m_m2[55]=0xfffff5d1;
    m_m2[56]=0xfffff5cb;
    m_m2[57]=0xfffff599;
    m_m2[58]=0xfffff58d;
    m_m2[59]=0xfffff577;
    m_m2[60]=0xfffff563;
    m_m2[61]=0xfffff551;
    m_m2[62]=0xfffff53f;
    m_m2[63]=0xfffff539;

    BN_WORD_setone(m_M1);
    BN_WORD_setone(m_M2);
    for(int i=0;i<m_base_num;i++){
        RNS_WORD_BN_WORD_transform(m_m1[i],sizeof(RNS_WORD)*8, bn_m);
	BN_WORD_mul(bn_m,m_M1,temp_result);
	BN_WORD_copy(temp_result,m_M1);
    }
    for(int i=0;i<m_base_num;i++){
	RNS_WORD_BN_WORD_transform(m_m2[i],sizeof(RNS_WORD)*8, bn_m);
	BN_WORD_mul(bn_m,m_M2,temp_result);
	BN_WORD_copy(temp_result,m_M2);
    }					    
    BN_WORD_div(m_M1,m_rsa_n->n,q,m_M1_n);
    BN_WORD_div(m_M2,m_rsa_n->n,q,m_M2_n);
    for(int i=0;i<m_base_num;i++){
        RNS_WORD_BN_WORD_transform(m_m1[i], sizeof(RNS_WORD)*8, bn_m);
	BN_WORD_div(m_M1,bn_m,m_M1_i[i],r);
        RNS_WORD_BN_WORD_transform(m_m2[i], sizeof(RNS_WORD)*8, bn_m);
	BN_WORD_div(m_M2,bn_m,m_M2_i[i],r);
	BN_WORD_RNS_WORD_mod(m_M1_i[i],m_m1[i],M1_i_m1_i);
	RNS_WORD_mod_inverse(M1_i_m1_i,m_m1[i],M1_i_inverse);
	RNS_WORD_BN_WORD_transform(M1_i_inverse,sizeof(RNS_WORD)*8, bn_M1_i_inverse);
	BN_WORD_mul_mod_host(bn_M1_i_inverse,m_M1_i[i],m_M1,m_M1_red_i[i]);
	BN_WORD_RNS_WORD_mod(m_M2_i[i],m_m2[i],M2_i_m2_i);
	RNS_WORD_mod_inverse(M2_i_m2_i,m_m2[i],M2_i_inverse);
	RNS_WORD_BN_WORD_transform(M2_i_inverse,sizeof(RNS_WORD)*8, bn_M2_i_inverse);
	BN_WORD_mul_mod_host(bn_M2_i_inverse,m_M2_i[i],m_M2,m_M2_red_i[i]);
    }
    for(int i=0;i<m_base_num;i++){
        BN_WORD_RNS_WORD_mod(m_M1_i[i],m_m1[i],M);
	BN_WORD_RNS_WORD_mod(m_rsa_n->n,m_m1[i],p_temp);
    	M=rns_word_mul_mod(M,p_temp,m_m1[i]);
    	RNS_WORD_mod_inverse(M,m_m1[i],M_inverse);
    	m_d[i]=m_m1[i]-M_inverse;
    }
    for(int i=0;i<m_base_num;i++){
        BN_WORD_RNS_WORD_mod(m_M2_i[i],m_m2[i],M);
	BN_WORD_RNS_WORD_mod(m_M1,m_m2[i],M_inverse);
	M=rns_word_mul_mod(M,M_inverse,m_m2[i]);
	RNS_WORD_mod_inverse(M,m_m2[i],m_e[i]);
    }
    for(int i=0;i<m_base_num;i++){
	for(int j=0;j<m_base_num;j++){
	    BN_WORD_RNS_WORD_mod(m_M2_i[i],m_m2[i],M);
	    M=rns_word_mul_mod(M,m_m1[j],m_m2[i]);
	    RNS_WORD_mod_inverse(M,m_m2[i],M_inverse);
	    BN_WORD_RNS_WORD_mod(m_rsa_n->n,m_m2[i],p_temp);
	    m_a[i*m_base_num+j]=rns_word_mul_mod(M_inverse,p_temp,m_m2[i]);
	}		
    }
    for(int i=0;i<m_base_num;i++){
        BN_WORD_RNS_WORD_mod(m_M2_i[i],m_m2[i],M);
	M=m_m2[i]-M;
	RNS_WORD_mod_inverse(M,m_m2[i],M_inverse);
	BN_WORD_RNS_WORD_mod(m_rsa_n->n,m_m2[i],p_temp);
	m_a_2[i]=rns_word_mul_mod(M_inverse,p_temp,m_m2[i]);
    }
    for(int i=0;i<m_base_num;i++){
        for(int j=0;j<m_base_num;j++){
	    BN_WORD_RNS_WORD_mod(m_M2_i[j],m_m1[i],M);
	    m_b[i*m_base_num+j]=M;
	}
    }
    for(int i=0;i<m_base_num;i++){
	BN_WORD_RNS_WORD_mod(m_M2,m_m1[i],M);
	M=m_m1[i]-M;
	m_b_2[i]=M;
    }
    for(int i=0;i<m_base_num;i++){
	BN_WORD_RNS_WORD_mod(m_M2_i[i],m_m2[i],M);
	m_c[i]=M;
    }
    BN_WORD_free(temp_result);
    BN_WORD_free(q);
    BN_WORD_free(bn_m);
    BN_WORD_free(r); 
    BN_WORD_free(bn_M1_i_inverse);
}

__host__ RNS_N:: ~RNS_N(){
    hipFree(m_m1);
    hipFree(m_m2);
    BN_WORD_free(m_M1);
    BN_WORD_free(m_M2);
    BN_WORD_free(m_M1_n);
    BN_WORD_free(m_M2_n);
    for(int i=0;i<m_base_num;i++){
        BN_WORD_free(m_M1_i[i]);
	BN_WORD_free(m_M1_red_i[i]);
    }
    free(m_M1_i);
    free(m_M2_i);
    free(m_M1_red_i);
    free(m_M2_red_i);
    hipFree(m_d);
    hipFree(m_e);
    hipFree(m_a);
    hipFree(m_a_2);
    hipFree(m_b);
    hipFree(m_b_2);
    hipFree(m_c);
}

__host__  int RNS_N:: RNS_print(){
    printf("rsa_n:\n");
    RSA_N_print(m_rsa_n);
    printf("base_num:%x\n",m_base_num);
    for(int i=0;i<m_base_num;i++){
        printf("m1[%x]:%x\n",i,m_m1[i]);
    }
    for(int i=0;i<m_base_num;i++){ 
	printf("m2[%x]:%x\n",i,m_m2[i]);
    }
    printf("M1:\n");
    BN_WORD_print(m_M1);
    printf("M2:\n");
    BN_WORD_print(m_M2);
    for(int i=0;i<m_base_num;i++){
	printf("M1_%x:",i);
    	BN_WORD_print(m_M1_i[i]);
    }
    for(int i=0;i<m_base_num;i++){
        printf("M2_%x:",i);
	BN_WORD_print(m_M2_i[i]);
    }
    for(int i=0;i<m_base_num;i++){
	printf("M1_red_%x:\n",i);
    	BN_WORD_print(m_M1_red_i[i]);
    }
    for(int i=0;i<m_base_num;i++){
	printf("M2_red_%x:\n",i);
	BN_WORD_print(m_M2_red_i[i]);
    }		
    printf("M1_n:\n");
    BN_WORD_print(m_M1_n);
    printf("M2_n:\n");
    BN_WORD_print(m_M2_n);
    for(int i=0;i<m_base_num;i++){
        printf("d%x:%x\n",i,m_d[i]);
    }
    for(int i=0;i<m_base_num;i++){
        printf("e%x:%x\n",i,m_e[i]);
    }
    for(int i=0;i<m_base_num*m_base_num;i++){
        printf("a%x:%x\n",i,m_a[i]);
    }
    for(int i=0;i<m_base_num;i++){
        printf("a_2_%x:%u\n",i,m_a_2[i]);
    }
    for(int i=0;i<m_base_num*m_base_num;i++){
        printf("b%x:%x\n",i,m_b[i]);
    }
    for(int i=0;i<m_base_num;i++){
        printf("b_2_%x:%u\n",i,m_b_2[i]);
    }
    for(int i=0;i<m_base_num;i++){
	printf("c%x:%x\n",i,m_c[i]);
    }
    return 0;
}

__host__ int RNS_N:: RNS_mul_mod (BN_WORD *a, BN_WORD *b, BN_WORD *result){
    int dmax=a->dmax;
    BN_WORD *a_temp, *b_temp,*one;
    RNS_WORD *x_result;
    a_temp=BN_WORD_new(dmax);
    b_temp=BN_WORD_new(dmax);
    one=BN_WORD_new(dmax);
    hipMallocManaged((void**)&(x_result),m_base_num*sizeof(RNS_WORD));
    BN_WORD_mul_mod_host(a, m_M1, m_rsa_n->n, a_temp); //a=a*M mod n
    BN_WORD_mul_mod_host(b, m_M1, m_rsa_n->n, b_temp);
    BN_WORD_setone(one);
    if(dmax==32){
	RNS_mul_mod_kernel_32<<<1,dmax>>>(a_temp,b_temp,m_base_num, m_m1,m_m2,m_d,m_e,m_a,m_a_2,m_b,m_b_2,m_c,x_result);
    	hipDeviceSynchronize();
    }
    if(dmax==64){
	RNS_mul_mod_kernel_64<<<1,dmax>>>(a_temp,b_temp,m_base_num, m_m1,m_m2,m_d,m_e,m_a,m_a_2,m_b,m_b_2,m_c,x_result);
	hipDeviceSynchronize();
    }
    if((dmax!=32)&&(dmax!=64)){
	print("base_num is not 32 or 64, error\n");
	return -1;
    }
    RSA_RNS_reduction1(x_result,result);
    printf("mod_mul: result1:\n");
    BN_WORD_print(result);
    RNS_mul_mod_kernel<<<1,dmax>>>(result,one,m_base_num, m_m1,m_m2,m_d,m_e,m_a,m_a_2,m_b,m_b_2,m_c,x_result);
    hipDeviceSynchronize();
    RSA_RNS_reduction1(x_result,result);  
    BN_WORD_free(a_temp);
    hipFree(x_result);
    return 0;
}

__host__ int RNS_N:: RSA (BN_WORD *a, BN_WORD *e, BN_WORD *result){
    int dmax=a->dmax;
    BN_WORD *a_temp;
    RNS_WORD *x_result;
    a_temp=BN_WORD_new(dmax);
    hipMallocManaged((void**)&(x_result),m_base_num*sizeof(RNS_WORD));
    BN_WORD_mul_mod_host(a, m_M1, m_rsa_n->n, a_temp); //a=a*M mod n
    printf("bn_a_M:\n");
    BN_WORD_print(a_temp);
    if(dmax==32){
        RSA_RNS_kernel_32<<<1,dmax*2,m_base_num*2>>>(a_temp,e,m_base_num, m_m1,m_m2,m_d,m_e,m_a,m_a_2,m_b,m_b_2,m_c,x_result);    
    	hipDeviceSynchronize();
    }
    if(dmax==64){
        RSA_RNS_kernel_64<<<1,dmax*2,m_base_num*2>>>(a_temp,e,m_base_num, m_m1,m_m2,m_d,m_e,m_a,m_a_2,m_b,m_b_2,m_c,x_result);
	hipDeviceSynchronize();
    }
    if((dmax!=32)&&(dmax!=64)){
        print("base_num is not 32 or 64, error\n");
	return -1;
    }
    RSA_RNS_reduction1(x_result,result);
    BN_WORD_free(a_temp);
    hipFree(x_result);
    return 0;
}


__global__ void RNS_mul_mod_kernel(BN_WORD *bn_a,BN_WORD *bn_b,int base_num,RNS_WORD *m1, RNS_WORD *m2,RNS_WORD *d,RNS_WORD *e,RNS_WORD *a, RNS_WORD *a_2,RNS_WORD *b,RNS_WORD *b_2,RNS_WORD *c,RNS_WORD *x_result){
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x;
    unsigned int mask=0xffffffff;
    RNS_WORD x_1, x_2, y_1, y_2;
    RNS_WORD theta,xi,theta_k,sigma,sigma_add,sigma_k,x_result_1,x_result_add,L1,L2;
    float L1_float,L2_float;
    BN_WORD_RNS_WORD_mod_device(bn_a, m1[thread_id], x_1);
    BN_WORD_RNS_WORD_mod_device(bn_a, m2[thread_id], x_2);
    BN_WORD_RNS_WORD_mod_device(bn_b, m1[thread_id], y_1);
    BN_WORD_RNS_WORD_mod_device(bn_b, m2[thread_id], y_2);
    x_1=rns_word_mul_mod(x_1,y_1,m1[thread_id]);
    x_2=rns_word_mul_mod(x_2,y_2,m2[thread_id]);
    if((bn_b->d[0]!=1)||(bn_b->d[1]!=0)){
        printf("mul_mod;x_1[%x]:%x\n",thread_id,x_1);
    	printf("mul_mod;x_2[%x]:%x\n",thread_id,x_2);
    }
    theta=rns_word_mul_mod(x_1,d[thread_id],m1[thread_id]);
    xi=rns_word_mul_mod(x_2,e[thread_id],m2[thread_id]);
    L1_float=0;
    sigma=0;
    for(int k=0;k<base_num;k++){
        theta_k=__shfl_sync(mask,theta,k);
	L1_float+=(float)theta_k/m1[k];
	sigma_add=rns_word_mul_mod(a[thread_id*base_num+k],theta_k,m2[thread_id]);
	sigma=rns_word_add_mod(sigma,sigma_add,m2[thread_id]);
    }
    L1=(RNS_WORD)L1_float;
    sigma=rns_word_add_mod(sigma,xi,m2[thread_id]);
    sigma_add=rns_word_mul_mod(L1,a_2[thread_id],m2[thread_id]);
    sigma=rns_word_add_mod(sigma,sigma_add,m2[thread_id]);
    L2_float=0;
    x_result_1=0;
    for(int k=0;k<base_num;k++){
        sigma_k=__shfl_sync(mask,sigma,k,32);
	L2_float+=(float)sigma_k/m2[k];
	x_result_add=rns_word_mul_mod(b[thread_id*base_num+k],sigma_k,m1[thread_id]);
	x_result_1=rns_word_add_mod(x_result_1,x_result_add,m1[thread_id]);
    }
    L2=(RNS_WORD)L2_float;
    x_result_add=rns_word_mul_mod(L2,b_2[thread_id],m1[thread_id]);
    x_result[thread_id]=rns_word_add_mod(x_result_1,x_result_add,m1[thread_id]);
    if((bn_b->d[0]!=1)||(bn_b->d[1]!=0)){
	printf("mul_mod;x_result[%x]:%x\n",thread_id,x_result[thread_id]);
    }
}

__global__ void RSA_RNS_kernel(BN_WORD *bn_a,BN_WORD *bn_e,int base_num,RNS_WORD *m1,RNS_WORD *m2, RNS_WORD *d,RNS_WORD *e,RNS_WORD *a, RNS_WORD *a_2,RNS_WORD *b,RNS_WORD *b_2,RNS_WORD *c,RNS_WORD *x_result){
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x;
    int thread_j=thread_id%base_num;
    RNS_WORD x_square_1,x_square_2, x_result_1, x_result_2,L1,L2,theta, xi ,sigma,theta_k,sigma_k,sigma_add, s_1,s_1_add;
    float  L1_float, L2_float;
    int mark=0;
    extern __shared__ RNS_WORD x_shared[];
    BN_WORD_RNS_WORD_mod_device(bn_a, m1[thread_j], x_square_1);
    BN_WORD_RNS_WORD_mod_device(bn_a, m2[thread_j], x_square_2);
    printf("bn_a_M_mod_m1[%x]:%x\n",thread_id,x_square_1);
    printf("bn_a_M_mod_m2[%x]:%x\n",thread_id,x_square_2);
    unsigned int mask=0xffffffff;    
    for(int i=0;i<base_num;i++){
        for(int j=0;j<base_num;j++){
	    __syncthreads();
	    if(thread_id<base_num){
	        x_shared[thread_j]=x_square_1;
		x_shared[thread_j+base_num]=x_square_2;
	    }
	    __syncthreads();
	    if(thread_id>=base_num){
	        x_square_1=x_shared[thread_j];
		x_square_2=x_shared[thread_j+base_num];
	    }
	    __syncthreads();
	    //square*square
	    if(thread_id<base_num){
	        x_square_1=rns_word_mul_mod(x_square_1,x_square_1,m1[thread_j]);
    		x_square_2=rns_word_mul_mod(x_square_2,x_square_2,m2[thread_j]);
    		theta=rns_word_mul_mod(x_square_1,d[thread_j],m1[thread_j]);
		xi=rns_word_mul_mod(x_square_2,e[thread_j],m2[thread_j]);
		L1_float=0;
		sigma=0;
		for(int k=0;k<base_num;k++){
		    theta_k=__shfl_sync(mask,theta,k,32);
		    L1_float+=(float)theta_k/m1[k];
		    sigma_add=rns_word_mul_mod(a[thread_j*base_num+k],theta_k,m2[thread_j]);
		    sigma=rns_word_add_mod(sigma,sigma_add,m2[thread_j]);
		}
		L1=(RNS_WORD)L1_float;
		sigma=rns_word_add_mod(sigma,xi,m2[thread_j]);
		sigma_add=rns_word_mul_mod(L1,a_2[thread_j],m2[thread_j]);
		sigma=rns_word_add_mod(sigma,sigma_add,m2[thread_j]);
		L2_float=0;
		s_1=0;
		for(int k=0;k<base_num;k++){
		    sigma_k=__shfl_sync(mask,sigma,k,32);
		    L2_float+=(float)sigma_k/m2[k];
		    s_1_add=rns_word_mul_mod(b[thread_j*base_num+k],sigma_k,m1[thread_j]);
		    s_1=rns_word_add_mod(s_1,s_1_add,m1[thread_j]);
		}
		L2=(RNS_WORD)L2_float;
		s_1_add=rns_word_mul_mod(L2,b_2[thread_j],m1[thread_j]);
		x_square_1=rns_word_add_mod(s_1,s_1_add,m1[thread_j]);
		x_square_2=rns_word_mul_mod(sigma,c[thread_j],m2[thread_j]);
#ifdef I_BIT_0
		if((i==0)&&(j==0)){
		    printf("bn_a_exp_2_M_mod_m1[%x]:%x\n",thread_id,x_square_1);
		    printf("bn_a_exp_2_M_mod_m2[%x]:%x\n",thread_id,x_square_2);
		}
#endif

#ifdef I_BIT_1
                if((i==0)&&(j==1)){
                    printf("bn_a_exp_4_M_mod_m1[%x]:%x\n",thread_id,x_square_1);
                    printf("bn_a_exp_4_M_mod_m2[%x]:%x\n",thread_id,x_square_2);
                }
#endif
#ifdef I_BIT_2
                if((i==0)&&(j==2)){
                    printf("bn_a_exp_8_M_mod_m1[%x]:%x\n",thread_id,x_square_1);
                    printf("bn_a_exp_8_M_mod_m2[%x]:%x\n",thread_id,x_square_2);
                }
#endif
	    }
	    //result=square*result
	    else{
	        if(get_bit(bn_e->d[i],j)==(BN_PART)1){
		//need shared memory
		    if(mark==0){
		        x_result_1=x_square_1;
			x_result_2=x_square_2;
			mark=1;
		    }
		    else{
	 		x_result_1=rns_word_mul_mod(x_result_1,x_square_1,m1[thread_j]);
     			x_result_2=rns_word_mul_mod(x_result_2,x_square_2,m2[thread_j]);
			theta=rns_word_mul_mod(x_result_1,d[thread_j],m1[thread_j]);
     			xi=rns_word_mul_mod(x_result_2,e[thread_j],m2[thread_j]);
     			L1_float=0;
     			sigma=0;
     			for(int k=0;k<base_num;k++){
	 			theta_k=__shfl_sync(mask,theta,k,32);
	 			L1_float+=(float)theta_k/m1[k];
	 			sigma_add=rns_word_mul_mod(a[thread_j*base_num+k],theta_k,m2[thread_j]);
	 			sigma=rns_word_add_mod(sigma,sigma_add,m2[thread_j]);
     			}
			L1=(RNS_WORD)L1_float;
     			sigma=rns_word_add_mod(sigma,xi,m2[thread_j]);
			sigma_add=rns_word_mul_mod(L1,a_2[thread_j],m2[thread_j]);
     			sigma=rns_word_add_mod(sigma,sigma_add,m2[thread_j]);
     			L2_float=0;
     			s_1=0; 
     			for(int k=0;k<base_num;k++){
	 			sigma_k=__shfl_sync(mask,sigma,k,32);
		 		L2_float+=(float)sigma_k/m2[k];
	 			s_1_add=rns_word_mul_mod(b[thread_j*base_num+k],sigma_k,m1[thread_j]);
	 			s_1=rns_word_add_mod(s_1,s_1_add,m1[thread_j]);
	     		}		   
     			L2=(RNS_WORD)L2_float;
     			s_1_add=rns_word_mul_mod(L2,b_2[thread_j],m1[thread_j]);
     			x_result_1=rns_word_add_mod(s_1,s_1_add,m1[thread_j]);
     			x_result_2=rns_word_mul_mod(sigma,c[thread_j],m2[thread_j]);
		    }
		}
#ifdef I_BIT_0
		if((i==0)&&(j==0)){
		}
#endif
#ifdef I_BIT_1
		if((i==0)&&(j==1)){
		}
#endif
#ifdef I_BIT_2
		if((i==0)&&(j==2)){
		}
#endif
	    }
	
        }
    }
    __syncthreads();
    if(thread_id>=base_num){
	x_shared[thread_j]=x_result_1;
	x_shared[thread_j+base_num]=x_result_2;
    }
    if(thread_id<base_num){
	x_result_1=x_shared[thread_j];
	x_result_2=x_shared[thread_j+base_num];
    	theta=rns_word_mul_mod(x_result_1,d[thread_j],m1[thread_j]);
    	xi=rns_word_mul_mod(x_result_2,e[thread_j],m2[thread_j]);
    	L1_float=0;
    	sigma=0;
    	for(int k=0;k<base_num;k++){
		theta_k=__shfl_sync(mask,theta,k,32);
		L1_float+=(float)theta_k/m1[k];
		sigma_add=rns_word_mul_mod(a[thread_j*base_num+k],theta_k,m2[thread_j]);
		sigma=rns_word_add_mod(sigma,sigma_add,m2[thread_j]);
    	}
    	L1=(RNS_WORD)L1_float;
    	sigma=rns_word_add_mod(sigma,xi,m2[thread_j]);
    	sigma_add=rns_word_mul_mod(L1,a_2[thread_j],m2[thread_j]);
    	sigma=rns_word_add_mod(sigma,sigma_add,m2[thread_j]);
    	L2_float=0;
    	s_1=0; 
    	for(int k=0;k<base_num;k++){
		sigma_k=__shfl_sync(mask,sigma,k,32);
		L2_float+=(float)sigma_k/m2[k];
		s_1_add=rns_word_mul_mod(b[thread_j*base_num+k],sigma_k,m1[thread_j]);
		s_1=rns_word_add_mod(s_1,s_1_add,m1[thread_j]);
    	}		   
    	L2=(RNS_WORD)L2_float;
    	s_1_add=rns_word_mul_mod(L2,b_2[thread_j],m1[thread_j]);
    	x_result[thread_j]=rns_word_add_mod(s_1,s_1_add,m1[thread_j]);
    }
}

__host__ int RNS_N:: RSA_RNS_reduction1(RNS_WORD *x_result, BN_WORD *result){
    BN_WORD *result_add, *bn_x;
    bn_x=BN_WORD_new(m_base_num);
    result_add=BN_WORD_new(m_base_num);
    BN_WORD_setzero(result);
    for(int i=0;i<m_base_num;i++){
        RNS_WORD_BN_WORD_transform(x_result[i],sizeof(RNS_WORD)*8,bn_x);
	BN_WORD_mul_mod_host(bn_x,m_M1_red_i[i],m_M1,result_add);
	BN_WORD_add_mod_host(result,result_add,m_M1,result);
    }
    return 0;
}

__host__ int RNS_N:: RSA_RNS_reduction2(RNS_WORD *x_result, BN_WORD *result){
    BN_WORD *result_add, *bn_x;
    bn_x=BN_WORD_new(m_base_num);
    result_add=BN_WORD_new(m_base_num);
    BN_WORD_setzero(result);
    for(int i=0;i<m_base_num;i++){
        RNS_WORD_BN_WORD_transform(x_result[i],sizeof(RNS_WORD)*8,bn_x);
        BN_WORD_mul_mod_host(bn_x,m_M2_red_i[i],m_M2,result_add);
        BN_WORD_add_mod_host(result,result_add,m_M2,result);
    }
    return 0;
}

/*
    m_M1->d[0]=0x77985e5f;
    m_M1->d[1]=0x3b24aba7;
    m_M1->d[2]=0x1860cdd4;
    m_M1->d[3]=0x4b54b64d;
    m_M1->d[4]=0x587fd8b2;
    m_M1->d[5]=0xf0ea0ad5;
    m_M1->d[6]=0x8fcd9f4d;
    m_M1->d[7]=0x2da1a3f4;
    m_M1->d[8]=0xb68892ed;
    m_M1->d[9]=0x0239d494;
    m_M1->d[10]=0x1b911305;
    m_M1->d[11]=0xf4afddef;
    m_M1->d[12]=0x576cf656;
    m_M1->d[13]=0xaee50801;
    m_M1->d[14]=0xce2511a5;
    m_M1->d[15]=0xef50a335;
    m_M1->d[16]=0xee9a7715;
    m_M1->d[17]=0xe0276ec7;
    m_M1->d[18]=0xdf431b50;
    m_M1->d[19]=0x8fcd69ff;
    m_M1->d[20]=0x5675a100;
    m_M1->d[21]=0x6e41d06c;
    m_M1->d[22]=0x62bd6520;
    m_M1->d[23]=0x43a789e4;
    m_M1->d[24]=0x2da442e2;
    m_M1->d[25]=0x600dc5b9;
    m_M1->d[26]=0xdb8fe947;
    m_M1->d[27]=0x920141be;
    m_M1->d[28]=0x0e417b4b;
    m_M1->d[29]=0xb134f2fe;
    m_M1->d[30]=0x069d4df9;
    m_M1->d[31]=0xffffc492;

    m_M2->d[0]=0xf92db8dd;
    m_M2->d[1]=0xeb720815;
    m_M2->d[2]=0x31eeb1bc;
    m_M2->d[3]=0x283022fa;
    m_M2->d[4]=0xeace80df;
    m_M2->d[5]=0xbc3d8630;
    m_M2->d[6]=0x395bb69b;
    m_M2->d[7]=0xe0f69eb0;
    m_M2->d[8]=0xfec07a5f;
    m_M2->d[9]=0xcde9f980;
    m_M2->d[10]=0x8dbebedd;
    m_M2->d[11]=0x23201df2;
    m_M2->d[12]=0xf25e6242;
    m_M2->d[13]=0xa8231c1b;
    m_M2->d[14]=0x0219f9c9;
    m_M2->d[15]=0x1de97696;
    m_M2->d[16]=0x914aa0d6;
    m_M2->d[17]=0xf5a99d64;
    m_M2->d[18]=0xe9a9a8cc;
    m_M2->d[19]=0xbd420dc5;
    m_M2->d[20]=0x593ebeb4;
    m_M2->d[21]=0xe0fccc6d;
    m_M2->d[22]=0xe469c091;
    m_M2->d[23]=0xf1c229a8;
    m_M2->d[24]=0x43979531;
    m_M2->d[25]=0xee1d2206;
    m_M2->d[26]=0xc89784e8;
    m_M2->d[27]=0x70638d2b;
    m_M2->d[28]=0x795198af;
    m_M2->d[29]=0x923f12c4;
    m_M2->d[30]=0x2db293aa;
    m_M2->d[31]=0xffff648c;
*/
