#include "hip/hip_runtime.h"
#include "bn_word_parallel_mont_exp.h"
#include "stdio.h"
#include "iostream"
#include "time.h"
#include "sys/time.h"

using namespace std;

#define LOOP_NUM 1

#define HZ_PER_US (double)1

#ifdef NOTEMP

__global__ void BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, const BN_PART n0_inverse, BN_WORD *result, 
		BN_WORD *u, BN_WORD *v, BN_WORD *m, BN_WORD *c){
    int j=threadIdx.x+blockIdx.x*blockDim.x;
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;
    int dmax=a->dmax;
    //branch need cal

#ifdef CUDA_TIMING
    clock_t start_t, start_loop_t,start_deletecarry_t, start_madlo_t,start_deleteu_t, end_setzero_t, end_total_t,end_loop_t, end_deletecarry_t, end_deleteu_t,end_madlo_t;
    double total_t;
#endif

#ifdef CUDA_TIMING
    start_t=clock();
#endif
    p_a=a->d[j];
    p_n=n->d[j];
    p_u=0;
    p_v=0;
    m->d[j]=0;
    c->d[j]=0;
#ifdef CUDA_TIMING
    end_setzero_t=clock();
    total_t=(end_setzero_t-start_t)/HZ_PER_US;
    if(j==0){
        printf("setzero_time:%f\n",total_t);
    }
#endif

#ifdef CUDA_TIMING
    start_loop_t=clock();
#endif
    
    for(int i=0;i<dmax;i++){
	p_b=b->d[i];
#ifdef CUDA_TIMING
	if(j==0){
	    start_madlo_t=clock();
	}
#endif
        BN_WORD_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
#ifdef CUDA_TIMING
	if((j==0)&&(i==0)){
	    total_t=0;
	}
	if(j==0){
	    end_madlo_t=clock();
    	    total_t=total_t+(end_madlo_t-start_madlo_t)/HZ_PER_US;
	}
    	if((j==0)&&(i==dmax-1)){
	    printf("madlo_time:%f\n",total_t);
    	}
#endif
	p_u=ptemp_u+p_u;
        BN_WORD_mul_lo(p_v,n0_inverse,m->d[j]);
        __syncthreads();
	p_m=m->d[0];
        BN_WORD_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	v->d[j]=p_v;
        __syncthreads();
	p_v=v->d[int_mod(j+1,dmax)];
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
        BN_WORD_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
        BN_WORD_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
    }

#ifdef CUDA_TIMING
    end_loop_t=clock();
    total_t=(end_loop_t-start_loop_t)/HZ_PER_US;
    if(j==0){
        printf("loop_time:%f\n",total_t);
    }
#endif
#ifdef CUDA_TIMING
    start_deleteu_t=clock();
#endif

    c->d[j]=p_u;
    u->d[j]=p_u;
    while(BN_WORD_any(&(u->d[0]),dmax)==0){
        __syncthreads();
	p_u=u->d[int_mod(j-1,dmax)];
        if(j==0){
            p_u=0;
        }
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
	c->d[j]=c->d[j]+p_u;
	u->d[j]=p_u;
    }
    result->d[j]=p_v;
    c->d[j]=c->d[dmax-1];
#ifdef CUDA_TIMING
    end_deleteu_t=clock();
    total_t=(end_deleteu_t-start_deleteu_t)/HZ_PER_US;
    if(j==0){
        printf("deleteu_time:%f\n",total_t);
    }
#endif
#ifdef CUDA_TIMING
    start_deletecarry_t=clock();
#endif

    while(c->d[j]!=0){
        while((BN_WORD_cmp(result,n)==1)||(BN_WORD_cmp(result,n)==0)){
            BN_WORD_sub(result,n,result);
        }
	c->d[j]=c->d[j]-1;
        BN_WORD_sub(result,n,result);
    }
    while((BN_WORD_cmp(result,n)==1)||(BN_WORD_cmp(result,n)==0)){
        BN_WORD_sub(result,n,result);
    }
#ifdef CUDA_TIMING
    end_deletecarry_t=clock();
    total_t=(end_deletecarry_t-start_deletecarry_t)/HZ_PER_US;
    if(j==0){
        printf("deletecarry_time:%f\n",total_t);
    }
#endif    
#ifdef CUDA_TIMING
    end_total_t=clock();
    total_t=(end_total_t-start_t)/HZ_PER_US;
    if(j==0){
        printf("total_time:%f\n",total_t);
    }
#endif

}

#endif


#ifdef SHARED

#define U 0
#define V dmax
#define C dmax*2
#define M dmax*3

__global__ void BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, const BN_PART n0_inverse, BN_WORD *result){
    int j=threadIdx.x+blockIdx.x*blockDim.x;
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;
    int dmax=a->dmax;
    extern __shared__ BN_PART temp[];
    //branch need cal

#ifdef CUDA_TIMING
    clock_t start_t, start_loop_t,start_deletecarry_t, start_deleteu_t,start_madlo_t, end_setzero_t, end_total_t,end_loop_t, end_deletecarry_t, end_deleteu_t,end_madlo_t;
    double total_t;
#endif

#ifdef CUDA_TIMING
    start_t=clock();
#endif
    p_a=a->d[j];
    p_n=n->d[j];
    p_u=0;
    p_v=0;
    temp[M+j]=0;
    temp[C+j]=0;
#ifdef CUDA_TIMING
    end_setzero_t=clock();
    total_t=(end_setzero_t-start_t)/HZ_PER_US;
    if(j==0){
        printf("setzero_time:%f\n",total_t);
    }
#endif

#ifdef CUDA_TIMING
    start_loop_t=clock();
#endif
    for(int i=0;i<dmax;i++){
	p_b=b->d[i];
        BN_WORD_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
        BN_WORD_mul_lo(p_v,n0_inverse,temp[M+j]);
        __syncthreads();
	p_m=temp[M+0];
        BN_WORD_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	temp[V+j]=p_v;
	__syncthreads();
	p_v=temp[V+int_mod(j+1,dmax)];
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
        BN_WORD_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
        BN_WORD_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
    }

#ifdef CUDA_TIMING
    end_loop_t=clock();
    total_t=(end_loop_t-start_loop_t)/HZ_PER_US;
    if(j==0){
        printf("loop_time:%f\n",total_t);
    }
#endif
#ifdef CUDA_TIMING
    start_deleteu_t=clock();
#endif

    temp[C+j]=p_u;
    temp[U+j]=p_u;
    __syncthreads();//
    while(BN_WORD_any(temp+U,dmax)==0){
	p_u=temp[U+int_mod(j-1,dmax)];
        if(j==0){
            p_u=0;
        }
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
	temp[C+j]=temp[C+j]+p_u;
	temp[U+j]=p_u;
	__syncthreads();
    }
    result->d[j]=p_v;
    temp[C+j]=temp[C+dmax-1];
#ifdef CUDA_TIMING
    end_deleteu_t=clock();
    total_t=(end_deleteu_t-start_deleteu_t)/HZ_PER_US;
    if(j==0){
        printf("deleteu_time:%f\n",total_t);
    }
#endif
#ifdef CUDA_TIMING
    start_deletecarry_t=clock();
#endif
    while(temp[C+j]!=0){
        while((BN_WORD_cmp(result,n)==1)||(BN_WORD_cmp(result,n)==0)){
            BN_WORD_sub(result,n,result);
        }
	temp[C+j]=temp[C+j]-1;
        BN_WORD_sub(result,n,result);
    }
    while((BN_WORD_cmp(result,n)==1)||(BN_WORD_cmp(result,n)==0)){
        BN_WORD_sub(result,n,result);
    }
#ifdef CUDA_TIMING
    end_deletecarry_t=clock();
    total_t=(end_deletecarry_t-start_deletecarry_t)/HZ_PER_US;
    if(j==0){
        printf("deletecarry_time:%f\n",total_t);
    }
#endif    
#ifdef CUDA_TIMING
    end_total_t=clock();
    total_t=(end_total_t-start_t)/HZ_PER_US;
    if(j==0){
        printf("total_time:%f\n",total_t);
    }
#endif

}

#endif

#ifdef notemp

__host__ int BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, BN_WORD *result){

//#ifdef CUDA_TIMING
    timeval start, stop;
    double sum_time;
  //  clock_t start_t, end_t;
  //  double total_t;

//#endif

    int dmax=a->dmax;
    BN_PART n0_inverse;
    BN_WORD *a_pro, *b_pro, *temp_result, *u,*v,*m,*c,*one, *zero,*R_pro;
    a_pro=BN_WORD_new(dmax);
    b_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    u=BN_WORD_new(dmax);
    v=BN_WORD_new(dmax);
    m=BN_WORD_new(dmax);
    c=BN_WORD_new(dmax);
    one=BN_WORD_new(dmax);
    zero=BN_WORD_new(dmax);
    R_pro=BN_WORD_new(dmax);
    BN_WORD_setone(one);
    BN_WORD_setzero(zero);
    BN_WORD_copy(a,a_pro);
    BN_WORD_copy(b,b_pro);
    while(BN_WORD_cmp(a_pro,n)==1){
        BN_WORD_sub(a_pro,n,a_pro);
    }
    if(BN_WORD_cmp(a_pro,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    while(BN_WORD_cmp(b_pro,n)==1){
        BN_WORD_sub(b_pro,n,b_pro);
    }
    if(BN_WORD_cmp(b_pro,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    BN_WORD_sub(zero,n,R_pro);
    while(BN_WORD_cmp(R_pro,n)==1){
        BN_WORD_sub(R_pro,n,R_pro);
    }
    BN_WORD_mul_mod_host(a_pro,R_pro,n,temp_result);//
    BN_WORD_copy(temp_result,a_pro);
    BN_WORD_mul_mod_host(b_pro,R_pro,n,temp_result);
    BN_WORD_copy(temp_result,b_pro);
    BN_PART_inverse(n->d[0],n0_inverse);//

#ifdef CUDA_TIMING
//    gettimeofday(&start,0);
#endif

    BN_WORD_parallel_mont_mul<<<1,dmax>>>(a_pro,b_pro,n,n0_inverse,temp_result, u, v, m, c);
    hipDeviceSynchronize();

#ifdef CUDA_TIMING
//    gettimeofday(&stop,0);
//    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
//    cout<<"parallel_time: "<<sum_time<<endl;
#endif
    
    BN_WORD_copy(temp_result,result);
    BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,one,n,n0_inverse,temp_result, u, v, m, c);
    hipDeviceSynchronize();
    BN_WORD_copy(temp_result,result);
    return 0;

}

#endif


#ifdef SHARED

__host__ int BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, BN_WORD *result){

//#ifdef CUDA_TIMING
    timeval start, stop;
    double sum_time;
  //  clock_t start_t, end_t;
  //  double total_t;

//#endif

    int dmax=a->dmax;
    BN_PART n0_inverse;
    BN_WORD *a_pro, *b_pro, *temp_result, *zero, *one, *R_pro;
    a_pro=BN_WORD_new(dmax);
    b_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    one=BN_WORD_new(dmax);
    zero=BN_WORD_new(dmax);
    R_pro=BN_WORD_new(dmax);
    BN_WORD_setone(one);
    BN_WORD_setzero(zero);
    BN_WORD_copy(a,a_pro);
    BN_WORD_copy(b,b_pro);
    while(BN_WORD_cmp(a_pro,n)==1){
        BN_WORD_sub(a_pro,n,a_pro);
    }
    if(BN_WORD_cmp(a_pro,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    while(BN_WORD_cmp(b_pro,n)==1){
        BN_WORD_sub(b_pro,n,b_pro);
    }
    if(BN_WORD_cmp(b_pro,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    BN_WORD_sub(zero,n,R_pro);
    while(BN_WORD_cmp(R_pro,n)==1){
        BN_WORD_sub(R_pro,n,R_pro);
    }
    BN_WORD_mul_mod_host(a_pro,R_pro,n,temp_result);//
    BN_WORD_copy(temp_result,a_pro);
    BN_WORD_mul_mod_host(b_pro,R_pro,n,temp_result);
    BN_WORD_copy(temp_result,b_pro);
    BN_PART_inverse(n->d[0],n0_inverse);//

#ifdef CUDA_TIMING
    gettimeofday(&start,0);

#endif

    BN_WORD_parallel_mont_mul<<<1,dmax,4*dmax*sizeof(BN_PART)>>>(a_pro,b_pro,n,n0_inverse,temp_result);
    hipDeviceSynchronize();

#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"parallel_time: "<<sum_time<<endl;
#endif
    
    BN_WORD_copy(temp_result,result);
    BN_WORD_parallel_mont_mul<<<1,dmax,4*dmax*sizeof(BN_PART)>>>(result,one,n,n0_inverse,temp_result);
    hipDeviceSynchronize();
    BN_WORD_copy(temp_result,result);
    return 0;

}

#endif

#ifdef NOTEMP
__host__ int BN_WORD_parallel_mont_exp(const BN_WORD *a, const BN_WORD *e, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;
//time
    timeval start, stop;
    double sum_time;

    BN_PART n0_inverse;
    BN_WORD *a_pro, *temp_result,*u,*v,*m,*c,*one, *zero,*R_pro;
    a_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    u=BN_WORD_new(dmax);
    v=BN_WORD_new(dmax);
    m=BN_WORD_new(dmax);
    c=BN_WORD_new(dmax);
    one=BN_WORD_new(dmax);
    zero=BN_WORD_new(dmax);
    R_pro=BN_WORD_new(dmax);
    BN_WORD_setone(one);
    BN_WORD_setzero(zero);
    BN_WORD_copy(a,a_pro);
    while(BN_WORD_cmp(a_pro,n)==1){
        BN_WORD_sub(a_pro,n,a_pro);
    }
    if(BN_WORD_cmp(a_pro,n)==0){
        BN_WORD_setzero(result);
	return 0;
    }
    BN_WORD_sub(zero,n,R_pro);
    while(BN_WORD_cmp(R_pro,n)==1){
        BN_WORD_sub(R_pro,n,R_pro);
    }
    BN_WORD_mul_mod_host(a_pro,R_pro,n,temp_result);//
    BN_WORD_copy(temp_result,a_pro);
    BN_PART_inverse(n->d[0],n0_inverse);//
    BN_WORD_copy(R_pro,result);
//time
//    gettimeofday(&start,0);

    for(int i=dmax-1;i>=0;i--){
        for(int j=sizeof(BN_PART)*8-1;j>=0;j--){
	     BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,result,n,n0_inverse,temp_result, u, v, m, c);
	     hipDeviceSynchronize();
	     BN_WORD_copy(temp_result,result);
	     if(get_bit(e->d[i],j)==(BN_PART)1){
		 BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,a_pro,n,n0_inverse,temp_result, u, v, m, c);
		 hipDeviceSynchronize();       
		 BN_WORD_copy(temp_result,result);
	     }
	}
    }
    BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,one,n,n0_inverse,temp_result, u, v, m, c);
    hipDeviceSynchronize();
    BN_WORD_copy(temp_result,result);
//time
/*    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"parallel_time: "<<sum_time<<endl;
*/
    return 0;
}
#endif

#ifdef SHARED
__host__ int BN_WORD_parallel_mont_exp(const BN_WORD *a, const BN_WORD *e, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;
//time
//    timeval start, stop;
//    double sum_time;

    BN_PART n0_inverse;
    BN_WORD *a_pro, *temp_result,*one, *zero,*R_pro;
    a_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    one=BN_WORD_new(dmax);
    zero=BN_WORD_new(dmax);
    R_pro=BN_WORD_new(dmax);
    BN_WORD_setone(one);
    BN_WORD_setzero(zero);
    BN_WORD_copy(a,a_pro);
    while(BN_WORD_cmp(a_pro,n)==1){
        BN_WORD_sub(a_pro,n,a_pro);
    }
    if(BN_WORD_cmp(a_pro,n)==0){
        BN_WORD_setzero(result);
	return 0;
    }
    BN_WORD_sub(zero,n,R_pro);
    while(BN_WORD_cmp(R_pro,n)==1){
        BN_WORD_sub(R_pro,n,R_pro);
    }
    BN_WORD_mul_mod_host(a_pro,R_pro,n,temp_result);//
    BN_WORD_copy(temp_result,a_pro);
    BN_PART_inverse(n->d[0],n0_inverse);//
    BN_WORD_copy(R_pro,result);
//time
//    gettimeofday(&start,0);

    for(int i=dmax-1;i>=0;i--){
        for(int j=sizeof(BN_PART)*8-1;j>=0;j--){
	     BN_WORD_parallel_mont_mul<<<1,dmax,4*dmax*sizeof(BN_PART)>>>(result,result,n,n0_inverse,temp_result);
	     hipDeviceSynchronize();
	     BN_WORD_copy(temp_result,result);
	     if(get_bit(e->d[i],j)==(BN_PART)1){
		 BN_WORD_parallel_mont_mul<<<1,dmax,4*dmax*sizeof(BN_PART)>>>(result,a_pro,n,n0_inverse,temp_result);
		 hipDeviceSynchronize();       
		 BN_WORD_copy(temp_result,result);
	     }
	}
    }
    BN_WORD_parallel_mont_mul<<<1,dmax,4*dmax*sizeof(BN_PART)>>>(result,one,n,n0_inverse,temp_result);
    hipDeviceSynchronize();
    BN_WORD_copy(temp_result,result);
//time
/*    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"parallel_time: "<<sum_time<<endl;
*/
    return 0;
}
#endif
