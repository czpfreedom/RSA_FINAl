#include "hip/hip_runtime.h"
#include "bn_word_parallel_mont_exp.h"
#include "stdio.h"
#include "iostream"
#include "time.h"
#include "sys/time.h"

using namespace std;

#define LOOP_NUM 1

//#define notemp

#define SHARE

#define CUDA_TIMING

#define HZ_PER_US (double)1

#ifdef SHARE
#define U 0
#define V dmax
#define M dmax*2
#define C dmax*3
#endif

#ifdef notemp

__global__ void BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, const BN_ULONG n0_inverse, BN_WORD *result, 
		BN_WORD *u, BN_WORD *v, BN_WORD *m, BN_WORD *c){
    int j=threadIdx.x+blockIdx.x*blockDim.x;
    BN_ULONG p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;
    int dmax=a->dmax;
    //branch need cal

#ifdef CUDA_TIMING
    clock_t start_t, start_loop_t,start_deletecarry_t, start_deleteu_t, end_setzero_t, end_total_t,end_loop_t, end_deletecarry_t, end_deleteu_t;
    double total_t;
#endif

#ifdef CUDA_TIMING
    start_t=clock();
#endif
    p_a=a->d[j];
    p_n=n->d[j];
    p_u=0;
    p_v=0;
    m->d[j]=0;
    c->d[j]=0;

    __syncthreads();


#ifdef CUDA_TIMING
    end_setzero_t=clock();
    total_t=(end_setzero_t-start_t)/HZ_PER_US;
    if(j==0){
        printf("setzero_time:%f\n",total_t);
    }
#endif

#ifdef CUDA_TIMING
    start_loop_t=clock();
#endif
    
    for(int i=0;i<dmax;i++){
	p_b=b->d[i];
        BN_WORD_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
        BN_WORD_mul_lo(p_v,n0_inverse,m->d[j]);
        __syncthreads();
	p_m=m->d[0];
        BN_WORD_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	v->d[j]=p_v;
        __syncthreads();
	p_v=v->d[int_mod(j+1,dmax)];
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
        BN_WORD_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
        BN_WORD_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	__syncthreads();
    }

#ifdef CUDA_TIMING
    end_loop_t=clock();
    total_t=(end_loop_t-start_loop_t)/HZ_PER_US;
    if(j==0){
        printf("loop_time:%f\n",total_t);
    }
#endif
#ifdef CUDA_TIMING
    start_deleteu_t=clock();
#endif

    c->d[j]=p_u;
    u->d[j]=p_u;
    __syncthreads();
    while(BN_WORD_any(u,dmax)==0){
	u->d[j]=p_u;
        __syncthreads();
	p_u=u->d[int_mod(j-1,dmax)];
        if(j==0){
            p_u=0;
        }
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
	c->d[j]=c->d[j]+p_u;
        __syncthreads();
	u->d[j]=p_u;
    }
    result->d[j]=p_v;
    c->d[j]=c->d[dmax-1];
#ifdef CUDA_TIMING
    end_deleteu_t=clock();
    total_t=(end_deleteu_t-start_deleteu_t)/HZ_PER_US;
    if(j==0){
        printf("deleteu_time:%f\n",total_t);
    }
#endif
#ifdef CUDA_TIMING
    start_deletecarry_t=clock();
#endif

    while(c->d[j]!=0){
        while((BN_WORD_cmp(result,n)==1)||(BN_WORD_cmp(result,n)==0)){
            BN_WORD_sub(result,n,result);
        }
	c->d[j]=c->d[j]-1;
        BN_WORD_sub(result,n,result);
    }
    while((BN_WORD_cmp(result,n)==1)||(BN_WORD_cmp(result,n)==0)){
        BN_WORD_sub(result,n,result);
    }
#ifdef CUDA_TIMING
    end_deletecarry_t=clock();
    total_t=(end_deletecarry_t-start_deletecarry_t)/HZ_PER_US;
    if(j==0){
        printf("deletecarry_time:%f\n",total_t);
    }
#endif    
#ifdef CUDA_TIMING
    end_total_t=clock();
    total_t=(end_total_t-start_t)/HZ_PER_US;
    if(j==0){
        printf("total_time:%f\n",total_t);
    }
#endif

}

#endif


#ifdef SHARE

__global__ void BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, const BN_ULONG n0_inverse, BN_WORD *result){
    int j=threadIdx.x+blockIdx.x*blockDim.x;
    BN_ULONG p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;
    int dmax=a->dmax;
    extern __shared__ BN_ULONG host_temp[];
    BN_ULONG* temp = (BN_ULONG*)host_temp;
    //branch need cal

#ifdef CUDA_TIMING
    clock_t start_t, start_loop_t,start_deletecarry_t, start_deleteu_t, end_setzero_t, end_total_t,end_loop_t, end_deletecarry_t, end_deleteu_t;
    double total_t;
#endif

#ifdef CUDA_TIMING
    start_t=clock();
#endif
    p_a=a->d[j];
    p_n=n->d[j];
    p_u=0;
    p_v=0;
    temp[M+j]=0;
    temp[C+j]=0;

    __syncthreads();


#ifdef CUDA_TIMING
    end_setzero_t=clock();
    total_t=(end_setzero_t-start_t)/HZ_PER_US;
    if(j==0){
        printf("setzero_time:%f\n",total_t);
    }
#endif

#ifdef CUDA_TIMING
    start_loop_t=clock();
#endif
    
    for(int i=0;i<dmax;i++){
	p_b=b->d[i];
        BN_WORD_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
        BN_WORD_mul_lo(p_v,n0_inverse,temp[M+j]);
        __syncthreads();
	p_m=temp[M+0];
        BN_WORD_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	temp[V+j]=p_v;
        __syncthreads();
	p_v=temp[V+int_mod(j+1,dmax)];
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
        BN_WORD_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
        BN_WORD_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	__syncthreads();
    }

#ifdef CUDA_TIMING
    end_loop_t=clock();
    total_t=(end_loop_t-start_loop_t)/HZ_PER_US;
    if(j==0){
        printf("loop_time:%f\n",total_t);
    }
#endif
#ifdef CUDA_TIMING
    start_deleteu_t=clock();
#endif

    temp[C+j]=p_u;
    temp[U+j]=p_u;
    __syncthreads();//
    while(BN_WORD_any(temp+U,dmax)==0){
	p_u=temp[U+int_mod(j-1,dmax)];
        if(j==0){
            p_u=0;
        }
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
	temp[C+j]=temp[C+j]+p_u;
        __syncthreads();
	temp[U+j]=p_u;
	__syncthreads();
    }
    result->d[j]=p_v;
    temp[C+j]=temp[C+dmax-1];
#ifdef CUDA_TIMING
    end_deleteu_t=clock();
    total_t=(end_deleteu_t-start_deleteu_t)/HZ_PER_US;
    if(j==0){
        printf("deleteu_time:%f\n",total_t);
    }
#endif
#ifdef CUDA_TIMING
    start_deletecarry_t=clock();
#endif

    while(temp[C+j]!=0){
        while((BN_WORD_cmp(result,n)==1)||(BN_WORD_cmp(result,n)==0)){
            BN_WORD_sub(result,n,result);
        }
	temp[C+j]=temp[C+j]-1;
        BN_WORD_sub(result,n,result);
    }
    while((BN_WORD_cmp(result,n)==1)||(BN_WORD_cmp(result,n)==0)){
        BN_WORD_sub(result,n,result);
    }
#ifdef CUDA_TIMING
    end_deletecarry_t=clock();
    total_t=(end_deletecarry_t-start_deletecarry_t)/HZ_PER_US;
    if(j==0){
        printf("deletecarry_time:%f\n",total_t);
    }
#endif    
#ifdef CUDA_TIMING
    end_total_t=clock();
    total_t=(end_total_t-start_t)/HZ_PER_US;
    if(j==0){
        printf("total_time:%f\n",total_t);
    }
#endif

}

#endif

__host__ int BN_WORD_mul_mod_host(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;
    int bit;
    BN_WORD *a_sub, *b_sub, *temp_result;
    a_sub=BN_WORD_new(dmax);
    b_sub=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    BN_WORD_copy(a,a_sub);
    BN_WORD_copy(b,b_sub);
    while(BN_WORD_cmp(a_sub,n)==1){
        BN_WORD_sub(a_sub,n,a_sub);
    }
    if(BN_WORD_cmp(a_sub,n)==0){
        BN_WORD_setzero(result);
	return 0;
    }
    while(BN_WORD_cmp(b_sub,n)==1){
        BN_WORD_sub(b_sub,n,b_sub);
    }
    if(BN_WORD_cmp(b_sub,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    BN_WORD_setzero(result);
    for(int i=dmax-1;i>=0;i--){
        for(int j=sizeof(BN_ULONG)*8-1;j>=0;j--){
		bit=get_bit(b_sub->d[i],j);
                BN_WORD_add(result,result,temp_result);
                if((BN_WORD_cmp(temp_result, result)==2)||(BN_WORD_cmp(temp_result,n)==1)||(BN_WORD_cmp(temp_result,n)==0)){
                    BN_WORD_sub(temp_result,n,result);
                }
                else {
                    BN_WORD_copy(temp_result,result);
                }
                if(bit==1){
                    BN_WORD_add(result,a_sub,temp_result);
                    if((BN_WORD_cmp(temp_result, result)==2)||(BN_WORD_cmp(temp_result,n)==1)||(BN_WORD_cmp(temp_result,n)==0)){
                            BN_WORD_sub(temp_result,n,result);
                    }
                    else {
                            BN_WORD_copy(temp_result,result);
                    }
                }
	}
    }
    BN_WORD_free(a_sub);
    BN_WORD_free(b_sub);
    BN_WORD_free(temp_result);
    return 0;
}

__host__ int BN_ULONG_inverse(const BN_ULONG n, BN_ULONG &n_inverse){
    BN_ULONG temp;
    BN_ULONG R1=n;
    BN_ULONG R2=0-n;
    BN_ULONG t1=1;
    BN_ULONG q=1+R2/R1;
    R2=R2%R1;
    BN_ULONG t2=0-q;
    while(R2!=1){
        temp=R2;
	q=R1/R2;
	R2=R1%R2;
	R1=temp;
	temp=t2;
	t2=t1-t2*q;
	t1=temp;
    }
    n_inverse=0-t2;
    return 0;
}

#ifdef notemp

__host__ int BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, BN_WORD *result){

//#ifdef CUDA_TIMING
    timeval start, stop;
    double sum_time;
  //  clock_t start_t, end_t;
  //  double total_t;

//#endif

    int dmax=a->dmax;
    BN_ULONG n0_inverse;
    BN_WORD *a_pro, *b_pro, *temp_result, *u,*v,*m,*c,*one, *zero,*R_pro;
    a_pro=BN_WORD_new(dmax);
    b_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    u=BN_WORD_new(dmax);
    v=BN_WORD_new(dmax);
    m=BN_WORD_new(dmax);
    c=BN_WORD_new(dmax);
    one=BN_WORD_new(dmax);
    zero=BN_WORD_new(dmax);
    R_pro=BN_WORD_new(dmax);
    BN_WORD_setone(one);
    BN_WORD_setzero(zero);
    BN_WORD_copy(a,a_pro);
    BN_WORD_copy(b,b_pro);
    while(BN_WORD_cmp(a_pro,n)==1){
        BN_WORD_sub(a_pro,n,a_pro);
    }
    if(BN_WORD_cmp(a_pro,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    while(BN_WORD_cmp(b_pro,n)==1){
        BN_WORD_sub(b_pro,n,b_pro);
    }
    if(BN_WORD_cmp(b_pro,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    BN_WORD_sub(zero,n,R_pro);
    while(BN_WORD_cmp(R_pro,n)==1){
        BN_WORD_sub(R_pro,n,R_pro);
    }
    BN_WORD_mul_mod_host(a_pro,R_pro,n,temp_result);//
    BN_WORD_copy(temp_result,a_pro);
    BN_WORD_mul_mod_host(b_pro,R_pro,n,temp_result);
    BN_WORD_copy(temp_result,b_pro);
    BN_ULONG_inverse(n->d[0],n0_inverse);//

//#ifdef CUDA_TIMING
    gettimeofday(&start,0);
  //  start_t=clock();

//#endif

    BN_WORD_parallel_mont_mul<<<1,dmax>>>(a_pro,b_pro,n,n0_inverse,temp_result, u, v, m, c);
    hipDeviceSynchronize();

//#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
  //  end_t=clock();
//    total_t=(end_t-start_t);
 //   printf("clock_parallel_time:%f\n",total_t);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"parallel_time: "<<sum_time<<endl;
//#endif
    
    BN_WORD_copy(temp_result,result);
    BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,one,n,n0_inverse,temp_result, u, v, m, c);
    hipDeviceSynchronize();
    BN_WORD_copy(temp_result,result);
    return 0;

}

#endif


#ifdef SHARE

__host__ int BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, BN_WORD *result){

//#ifdef CUDA_TIMING
    timeval start, stop;
    double sum_time;
  //  clock_t start_t, end_t;
  //  double total_t;

//#endif

    int dmax=a->dmax;
    BN_ULONG n0_inverse;
    BN_WORD *a_pro, *b_pro, *temp_result, *zero, *one, *R_pro;
    a_pro=BN_WORD_new(dmax);
    b_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    one=BN_WORD_new(dmax);
    zero=BN_WORD_new(dmax);
    R_pro=BN_WORD_new(dmax);
    BN_WORD_setone(one);
    BN_WORD_setzero(zero);
    BN_WORD_copy(a,a_pro);
    BN_WORD_copy(b,b_pro);
    while(BN_WORD_cmp(a_pro,n)==1){
        BN_WORD_sub(a_pro,n,a_pro);
    }
    if(BN_WORD_cmp(a_pro,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    while(BN_WORD_cmp(b_pro,n)==1){
        BN_WORD_sub(b_pro,n,b_pro);
    }
    if(BN_WORD_cmp(b_pro,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    BN_WORD_sub(zero,n,R_pro);
    while(BN_WORD_cmp(R_pro,n)==1){
        BN_WORD_sub(R_pro,n,R_pro);
    }
    BN_WORD_mul_mod_host(a_pro,R_pro,n,temp_result);//
    BN_WORD_copy(temp_result,a_pro);
    BN_WORD_mul_mod_host(b_pro,R_pro,n,temp_result);
    BN_WORD_copy(temp_result,b_pro);
    BN_ULONG_inverse(n->d[0],n0_inverse);//

//#ifdef CUDA_TIMING
    gettimeofday(&start,0);
  //  start_t=clock();

//#endif

    BN_WORD_parallel_mont_mul<<<1,dmax,4*dmax*sizeof(BN_ULONG)>>>(a_pro,b_pro,n,n0_inverse,temp_result);
    hipDeviceSynchronize();

//#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
  //  end_t=clock();
//    total_t=(end_t-start_t);
 //   printf("clock_parallel_time:%f\n",total_t);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"parallel_time: "<<sum_time<<endl;
//#endif
    
    BN_WORD_copy(temp_result,result);
    BN_WORD_parallel_mont_mul<<<1,dmax,4*dmax*sizeof(BN_ULONG)>>>(result,one,n,n0_inverse,temp_result);
    hipDeviceSynchronize();
    BN_WORD_copy(temp_result,result);
    return 0;

}

#endif


/*
__host__ int BN_WORD_parallel_mont_exp(const BN_WORD *a, const BN_WORD *e, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;
//time
    timeval start, stop;
    double sum_time;

    BN_ULONG n0_inverse;
    BN_WORD *a_pro, *temp_result,*u,*u_temp,*v,*v_temp,*m,*c,*one, *zero,*R_pro;
    a_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    u=BN_WORD_new(dmax);
    u_temp=BN_WORD_new(dmax);
    v=BN_WORD_new(dmax);
    v_temp=BN_WORD_new(dmax);
    m=BN_WORD_new(dmax);
    c=BN_WORD_new(dmax);
    one=BN_WORD_new(dmax);
    zero=BN_WORD_new(dmax);
    R_pro=BN_WORD_new(dmax);
    BN_WORD_setone(one);
    BN_WORD_setzero(zero);
    BN_WORD_copy(a,a_pro);
    while(BN_WORD_cmp(a_pro,n)==1){
        BN_WORD_sub(a_pro,n,a_pro);
    }
    if(BN_WORD_cmp(a_pro,n)==0){
        BN_WORD_setzero(result);
	return 0;
    }
    BN_WORD_sub(zero,n,R_pro);
    while(BN_WORD_cmp(R_pro,n)==1){
        BN_WORD_sub(R_pro,n,R_pro);
    }
    BN_WORD_mul_mod_host(a_pro,R_pro,n,temp_result);//
    BN_WORD_copy(temp_result,a_pro);
    BN_ULONG_inverse(n->d[0],n0_inverse);//
    BN_WORD_copy(R_pro,result);
//time
    gettimeofday(&start,0);

    for(int i=dmax-1;i>=0;i--){
        for(int j=sizeof(BN_ULONG)*8-1;j>=0;j--){
	     BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,result,n,n0_inverse,temp_result, u, u_temp, v, v_temp, m, c);
	     hipDeviceSynchronize();
	     BN_WORD_copy(temp_result,result);
	     if(get_bit(e->d[i],j)==(BN_ULONG)1){
		 BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,a_pro,n,n0_inverse,temp_result, u, u_temp, v, v_temp, m, c);
		 hipDeviceSynchronize();       
		 BN_WORD_copy(temp_result,result);
	     }
	}
    }
    BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,one,n,n0_inverse,temp_result, u, u_temp, v, v_temp, m, c);
    hipDeviceSynchronize();
    BN_WORD_copy(temp_result,result);
//time
    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"parallel_time: "<<sum_time<<endl;

    return 0;
}

*/
