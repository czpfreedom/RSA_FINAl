#include "hip/hip_runtime.h"
#include "bn_word_parallel_mont_exp.h"
#include "stdio.h"
#include "iostream"

using namespace std;

#define CUDA_TIMING

#ifdef CUDA_TIMING
#include "sys/time.h"
#endif

__global__ void BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, const BN_ULONG n0_inverse, BN_WORD *result, 
		BN_WORD *u, BN_WORD *u_temp,BN_WORD *v, BN_WORD *m, BN_WORD *c,BN_WORD *v_temp, int *any_value){
    int j=threadIdx.x+blockIdx.x*blockDim.x;
    int dmax=a->dmax;
    //branch need cal

    if(j==0){
	    BN_WORD_setzero(u);
	    BN_WORD_setzero(v);
	    BN_WORD_setzero(u_temp);
	    BN_WORD_setzero(v_temp);
	    BN_WORD_setzero(m);
	    BN_WORD_setzero(c);
    }
    __syncthreads();


    for(int i=0;i<dmax;i++){
        BN_WORD_mad_lo(a->d[j],b->d[i],v->d[j],u_temp->d[j],v->d[j]);
	u->d[j]=u_temp->d[j]+u->d[j];
        BN_WORD_mul_lo(v->d[j],n0_inverse,m->d[j]);
        __syncthreads();
        m->d[j]=m->d[0];
        BN_WORD_mad_lo(n->d[j],m->d[j],v->d[j],u_temp->d[j],v->d[j]);
	u->d[j]=u_temp->d[j]+u->d[j];
	v_temp->d[j]=v->d[j];
        __syncthreads();
	v->d[j]=v_temp->d[int_mod(j+1,dmax)];
	v->d[j]=u->d[j]+v->d[j];
	if(v->d[j]<u->d[j]){
	    u->d[j]=1;
	}
	else{
	    u->d[j]=0;
	}
        BN_WORD_mad_hi(a->d[j],b->d[i],v->d[j],u_temp->d[j],v->d[j]);
	u->d[j]=u_temp->d[j]+u->d[j];
        BN_WORD_mad_hi(n->d[j],m->d[j],v->d[j],u_temp->d[j],v->d[j]);
	u->d[j]=u_temp->d[j]+u->d[j];
	__syncthreads();
    }
    c->d[j]=u->d[j];
    __syncthreads();
    any_value[j]=BN_WORD_any(u);
    __syncthreads();
    while(any_value[j]==0){
	u_temp->d[j]=u->d[int_mod(j-1,dmax)];
        __syncthreads();
	u->d[j]=u_temp->d[j];
        if(j==0){
            u->d[j]=0;
        }
	v->d[j]=u->d[j]+v->d[j];
	if(v->d[j]<u->d[j]){
	    u->d[j]=1;
	}
	else{
	    u->d[j]=0;
	}
	c->d[j]=c->d[j]+u->d[j];
        __syncthreads();
        any_value[j]=BN_WORD_any(u);
    }
    BN_WORD_copy(v,result);
    c->d[j]=c->d[dmax-1];
    while(c->d[j]!=0){
        while((BN_WORD_cmp(result,n)==1)||(BN_WORD_cmp(result,n)==0)){
            BN_WORD_sub(result,n,result);
        }
	c->d[j]=c->d[j]-1;
        BN_WORD_sub(result,n,result);
    }
    while((BN_WORD_cmp(result,n)==1)||(BN_WORD_cmp(result,n)==0)){
        BN_WORD_sub(result,n,result);
    }
}


__host__ int BN_WORD_mul_mod_host(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;
    int bit;
    BN_WORD *a_sub, *b_sub, *temp_result;
    a_sub=BN_WORD_new(dmax);
    b_sub=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    BN_WORD_copy(a,a_sub);
    BN_WORD_copy(b,b_sub);
    while(BN_WORD_cmp(a_sub,n)==1){
        BN_WORD_sub(a_sub,n,a_sub);
    }
    if(BN_WORD_cmp(a_sub,n)==0){
        BN_WORD_setzero(result);
	return 0;
    }
    while(BN_WORD_cmp(b_sub,n)==1){
        BN_WORD_sub(b_sub,n,b_sub);
    }
    if(BN_WORD_cmp(b_sub,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    BN_WORD_setzero(result);
    for(int i=dmax-1;i>=0;i--){
        for(int j=sizeof(BN_ULONG)*8-1;j>=0;j--){
		bit=get_bit(b_sub->d[i],j);
                BN_WORD_add(result,result,temp_result);
                if((BN_WORD_cmp(temp_result, result)==2)||(BN_WORD_cmp(temp_result,n)==1)||(BN_WORD_cmp(temp_result,n)==0)){
                    BN_WORD_sub(temp_result,n,result);
                }
                else {
                    BN_WORD_copy(temp_result,result);
                }
                if(bit==1){
                    BN_WORD_add(result,a_sub,temp_result);
                    if((BN_WORD_cmp(temp_result, result)==2)||(BN_WORD_cmp(temp_result,n)==1)||(BN_WORD_cmp(temp_result,n)==0)){
                            BN_WORD_sub(temp_result,n,result);
                    }
                    else {
                            BN_WORD_copy(temp_result,result);
                    }
                }
	}
    }
    BN_WORD_free(a_sub);
    BN_WORD_free(b_sub);
    BN_WORD_free(temp_result);
    return 0;
}

__host__ int BN_ULONG_inverse(const BN_ULONG n, BN_ULONG &n_inverse){
    BN_ULONG temp;
    BN_ULONG R1=n;
    BN_ULONG R2=0-n;
    BN_ULONG t1=1;
    BN_ULONG q=1+R2/R1;
    R2=R2%R1;
    BN_ULONG t2=0-q;
    while(R2!=1){
        temp=R2;
	q=R1/R2;
	R2=R1%R2;
	R1=temp;
	temp=t2;
	t2=t1-t2*q;
	t1=temp;
    }
    n_inverse=0-t2;
    return 0;
}

__host__ int BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, BN_WORD *result){

#ifdef CUDA_TIMING
    timeval start, stop;
    double sum_time;
#endif

    int dmax=a->dmax;
    int *any_value;
    BN_ULONG n0_inverse;
    BN_WORD *a_pro, *b_pro, *temp_result, *u,*u_temp,*v,*v_temp,*m,*c,*one, *zero,*R_pro;
    hipMallocManaged((void **)&(any_value),dmax*sizeof(int));
    a_pro=BN_WORD_new(dmax);
    b_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    u=BN_WORD_new(dmax);
    u_temp=BN_WORD_new(dmax);
    v=BN_WORD_new(dmax);
    v_temp=BN_WORD_new(dmax);
    m=BN_WORD_new(dmax);
    c=BN_WORD_new(dmax);
    one=BN_WORD_new(dmax);
    zero=BN_WORD_new(dmax);
    R_pro=BN_WORD_new(dmax);
    BN_WORD_setone(one);
    BN_WORD_setzero(zero);
    BN_WORD_copy(a,a_pro);
    BN_WORD_copy(b,b_pro);
    while(BN_WORD_cmp(a_pro,n)==1){
        BN_WORD_sub(a_pro,n,a_pro);
    }
    if(BN_WORD_cmp(a_pro,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    while(BN_WORD_cmp(b_pro,n)==1){
        BN_WORD_sub(b_pro,n,b_pro);
    }
    if(BN_WORD_cmp(b_pro,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    BN_WORD_sub(zero,n,R_pro);
    while(BN_WORD_cmp(R_pro,n)==1){
        BN_WORD_sub(R_pro,n,R_pro);
    }
    BN_WORD_mul_mod_host(a_pro,R_pro,n,temp_result);//
    BN_WORD_copy(temp_result,a_pro);
    BN_WORD_mul_mod_host(b_pro,R_pro,n,temp_result);
    BN_WORD_copy(temp_result,b_pro);
    BN_ULONG_inverse(n->d[0],n0_inverse);//

#ifdef CUDA_TIMING
    gettimeofday(&start,0);
#endif

    BN_WORD_parallel_mont_mul<<<1,dmax>>>(a_pro,b_pro,n,n0_inverse,temp_result, u, u_temp, v, v_temp, m, c, any_value);
    hipDeviceSynchronize();

#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"parallel_time: "<<sum_time<<endl;
#endif
    
    BN_WORD_copy(temp_result,result);
    BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,one,n,n0_inverse,temp_result, u, u_temp, v, v_temp, m, c, any_value);
    hipDeviceSynchronize();
    BN_WORD_copy(temp_result,result);
    return 0;

}


__host__ int BN_WORD_parallel_mont_exp(const BN_WORD *a, const BN_WORD *e, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;

#ifdef CUDA_TIMING
    timeval start, stop;
    double sum_time;
#endif

    int *any_value;
    BN_ULONG n0_inverse;
    BN_WORD *a_pro, *temp_result,*u,*u_temp,*v,*v_temp,*m,*c,*one, *zero,*R_pro;
    hipMallocManaged((void **)&(any_value),dmax*sizeof(int));
    a_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    u=BN_WORD_new(dmax);
    u_temp=BN_WORD_new(dmax);
    v=BN_WORD_new(dmax);
    v_temp=BN_WORD_new(dmax);
    m=BN_WORD_new(dmax);
    c=BN_WORD_new(dmax);
    one=BN_WORD_new(dmax);
    zero=BN_WORD_new(dmax);
    R_pro=BN_WORD_new(dmax);
    BN_WORD_setone(one);
    BN_WORD_setzero(zero);
    BN_WORD_copy(a,a_pro);
    while(BN_WORD_cmp(a_pro,n)==1){
        BN_WORD_sub(a_pro,n,a_pro);
    }
    if(BN_WORD_cmp(a_pro,n)==0){
        BN_WORD_setzero(result);
	return 0;
    }
    BN_WORD_sub(zero,n,R_pro);
    while(BN_WORD_cmp(R_pro,n)==1){
        BN_WORD_sub(R_pro,n,R_pro);
    }
    BN_WORD_mul_mod_host(a_pro,R_pro,n,temp_result);//
    BN_WORD_copy(temp_result,a_pro);
    BN_ULONG_inverse(n->d[0],n0_inverse);//
    BN_WORD_copy(R_pro,result);

#ifdef CUDA_TIMING
    gettimeofday(&start,0);
#endif

    for(int i=dmax-1;i>=0;i--){
        for(int j=sizeof(BN_ULONG)*8-1;j>=0;j--){
	     BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,result,n,n0_inverse,temp_result, u, u_temp, v, v_temp, m, c, any_value);
	     hipDeviceSynchronize();
	     BN_WORD_copy(temp_result,result);
	     if(get_bit(e->d[i],j)==(BN_ULONG)1){
		 BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,a_pro,n,n0_inverse,temp_result, u, u_temp, v, v_temp, m, c, any_value);
		 hipDeviceSynchronize();       
		 BN_WORD_copy(temp_result,result);
	     }
	}
    }
    BN_WORD_parallel_mont_mul<<<1,dmax>>>(result,one,n,n0_inverse,temp_result, u, u_temp, v, v_temp, m, c, any_value);
    hipDeviceSynchronize();
    BN_WORD_copy(temp_result,result);

#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"parallel_time: "<<sum_time<<endl;
#endif

    return 0;
}
