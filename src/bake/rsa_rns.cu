#include "hip/hip_runtime.h"
#include "rsa_rns.h"

namespace namespace_rsa_final{

__global__ void RNS_mul_mod_kernel(BN_WORD *bn_a,BN_WORD *bn_b,int base_num,BN_PART *m1, BN_PART *m2,BN_PART *d,BN_PART *e,BN_PART *a, BN_PART *a_2,BN_PART *b,BN_PART*b_2,BN_PART *c,BN_PART *x_result){
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x;
    unsigned int mask=0xffffffff;
    BN_PART x_1, x_2, y_1, y_2,s_1,s_2;
    BN_PART theta,xi,theta_k,sigma,sigma_add,sigma_k,s_1_add,L1,L2;
    float L1_float,L2_float;
    BN_WORD_BN_PART_mod_device(bn_a, m1[thread_id], x_1);
    BN_WORD_BN_PART_mod_device(bn_a, m2[thread_id], x_2);
    BN_WORD_BN_PART_mod_device(bn_b, m1[thread_id], y_1);
    BN_WORD_BN_PART_mod_device(bn_b, m2[thread_id], y_2);
    BN_PART_mul_mod(x_1,y_1,m1[thread_id],x_1);
    BN_PART_mul_mod(x_2,y_2,m2[thread_id],x_2);
    BN_PART_mul_mod(x_1,d[thread_id],m1[thread_id],theta);
    BN_PART_mul_mod(x_2,e[thread_id],m2[thread_id],xi);
    L1_float=0;
    sigma=0;
    for(int k=0;k<base_num;k++){
        theta_k=__shfl_sync(mask,theta,k,32);
        L1_float+=(float)theta_k/m1[k];
        BN_PART_mul_mod(a[thread_id*base_num+k],theta_k,m2[thread_id],sigma_add);
        BN_PART_add_mod(sigma,sigma_add,m2[thread_id],sigma);
    }
  L1=(BN_PART)L1_float;
    BN_PART_add_mod(sigma,xi,m2[thread_id],sigma);
    BN_PART_mul_mod(L1,a_2[thread_id],m2[thread_id],sigma_add);
    BN_PART_add_mod(sigma,sigma_add,m2[thread_id],sigma);
    L2_float=0;
    s_1=0;
    for(int k=0;k<base_num;k++){
        sigma_k=__shfl_sync(mask,sigma,k,32);
        L2_float+=(float)sigma_k/m2[k];
        BN_PART_mul_mod(b[thread_id*base_num+k],sigma_k,m1[thread_id],s_1_add);
        BN_PART_add_mod(s_1,s_1_add,m1[thread_id],s_1);
    }
    L2=(BN_PART)L2_float;
    BN_PART_mul_mod(L2,b_2[thread_id],m1[thread_id],s_1_add);
    BN_PART_add_mod(s_1,s_1_add,m1[thread_id],s_1);
    BN_PART_mul_mod(sigma,c[thread_id],m2[thread_id],s_2);
    BN_PART_mul_mod(s_1,d[thread_id],m1[thread_id],theta);
    BN_PART_mul_mod(s_2,e[thread_id],m2[thread_id],xi);
    L1_float=0;
    sigma=0;
   for(int k=0;k<base_num;k++){
        theta_k=__shfl_sync(mask,theta,k,32);
        L1_float+=(float)theta_k/m1[k];
        BN_PART_mul_mod(a[thread_id*base_num+k],theta_k,m2[thread_id],sigma_add);
        sigma=BN_PART_add_mod(sigma,sigma_add,m2[thread_id],sigma);
    }
    L1=(BN_PART)L1_float;
    BN_PART_add_mod(sigma,xi,m2[thread_id],sigma);
    BN_PART_mul_mod(L1,a_2[thread_id],m2[thread_id],sigma_add);
    BN_PART_add_mod(sigma,sigma_add,m2[thread_id],sigma);
    L2_float=0;
    s_1=0;
    for(int k=0;k<base_num;k++){
        sigma_k=__shfl_sync(mask,sigma,k,32);
        L2_float+=(float)sigma_k/m2[k];
        BN_PART_mul_mod(b[thread_id*base_num+k],sigma_k,m1[thread_id],s_1_add);
        BN_PART_add_mod(s_1,s_1_add,m1[thread_id],s_1);
    }
    L2=(BN_PART)L2_float;
    BN_PART_mul_mod(L2,b_2[thread_id],m1[thread_id],s_1_add);
    BN_PART_add_mod(s_1,s_1_add,m1[thread_id],x_result[thread_id]);
}


__host__ RNS_N:: RNS_N(RSA_N *rsa_n){
    m_rsa_n=rsa_n;
    m_base_num=m_rsa_n->n->dmax;
    hipMallocManaged((void**)&(m_m1),BASE_MAX*sizeof(BN_PART));
    hipMallocManaged((void**)&(m_m2),BASE_MAX*sizeof(BN_PART));
    m_M1=BN_WORD_new(m_base_num);
    m_M2=BN_WORD_new(m_base_num);
    m_M1_n=BN_WORD_new(m_base_num);
    m_M2_n=BN_WORD_new(m_base_num);
    m_M1_i=(BN_WORD**)malloc(m_base_num*sizeof(BN_WORD*));
    m_M2_i=(BN_WORD**)malloc(m_base_num*sizeof(BN_WORD*));
    m_M1_red_i=(BN_WORD**)malloc(m_base_num*sizeof(BN_WORD*));
    m_M2_red_i=(BN_WORD**)malloc(m_base_num*sizeof(BN_WORD*));
    for(int i=0;i<m_base_num;i++){
        m_M1_i[i]=BN_WORD_new(m_base_num);
    }
    for(int i=0;i<m_base_num;i++){
        m_M2_i[i]=BN_WORD_new(m_base_num);
    }
    for(int i=0;i<m_base_num;i++){
        m_M1_red_i[i]=BN_WORD_new(m_base_num);
    }
    for(int i=0;i<m_base_num;i++){
        m_M2_red_i[i]=BN_WORD_new(m_base_num);
    }
    hipMallocManaged((void**)&(m_d),m_base_num*sizeof(BN_PART));
    hipMallocManaged((void**)&(m_e),m_base_num*sizeof(BN_PART));
    hipMallocManaged((void**)&(m_a),m_base_num*m_base_num*sizeof(BN_PART));
    hipMallocManaged((void**)&(m_a_2),m_base_num*sizeof(BN_PART));
    hipMallocManaged((void**)&(m_b),m_base_num*m_base_num*sizeof(BN_PART));
    hipMallocManaged((void**)&(m_b_2),m_base_num*sizeof(BN_PART));
    hipMallocManaged((void**)&(m_c),m_base_num*sizeof(BN_PART));
    BN_PART M, M_inverse, p_temp, M1_i_inverse,M1_i_m1_i, M2_i_inverse,M2_i_m2_i;
    BN_WORD *temp_result, *bn_m, *q, *r, *bn_M1_i_inverse, *bn_M2_i_inverse;
    temp_result=BN_WORD_new(m_base_num);
    bn_m=BN_WORD_new(m_base_num);
    q=BN_WORD_new(m_base_num);
    r=BN_WORD_new(m_base_num);
    bn_M1_i_inverse=BN_WORD_new(m_base_num);
    bn_M2_i_inverse=BN_WORD_new(m_base_num);

    m_m1[0] =0xffffffffffffffc5;
    m_m1[1] =0xffffffffffffffad;
    m_m1[2] =0xffffffffffffffa1;
    m_m1[3] =0xffffffffffffff4d;
    m_m1[4] =0xffffffffffffff43;
    m_m1[5] =0xfffffffffffffeff;
    m_m1[6] =0xfffffffffffffee9;
    m_m1[7] =0xfffffffffffffebd;
    m_m1[8] =0xfffffffffffffe9f;
    m_m1[9] =0xfffffffffffffe95;
    m_m1[10]=0xfffffffffffffe57;
    m_m1[11]=0xfffffffffffffe3b;
    m_m1[12]=0xfffffffffffffe09;
    m_m1[13]=0xfffffffffffffd19;
    m_m1[14]=0xfffffffffffffcc7;
    m_m1[15]=0xfffffffffffffcb5;
    m_m1[16]=0xfffffffffffffcb3;
    m_m1[17]=0xfffffffffffffc7f;
    m_m1[18]=0xfffffffffffffc7d;
    m_m1[19]=0xfffffffffffffc59;
    m_m1[20]=0xfffffffffffffc4f;
    m_m1[21]=0xfffffffffffffc01;
    m_m1[22]=0xfffffffffffffbff;
    m_m1[23]=0xfffffffffffffbcb;
    m_m1[24]=0xfffffffffffffbc9;
    m_m1[25]=0xfffffffffffffb2d;
    m_m1[26]=0xfffffffffffffb05;
    m_m1[27]=0xfffffffffffffad5;
    m_m1[28]=0xfffffffffffffa9d;
    m_m1[29]=0xfffffffffffffa43;
    m_m1[30]=0xfffffffffffffa3d;
    m_m1[31]=0xfffffffffffffa31;

    m_m2[0] =0xfffffffffffffa1f;
    m_m2[1] =0xfffffffffffffa13;
    m_m2[2] =0xfffffffffffff9df;
    m_m2[3] =0xfffffffffffff9d1;
    m_m2[4] =0xfffffffffffff9b9;
    m_m2[5] =0xfffffffffffff97f;
    m_m2[6] =0xfffffffffffff925;
    m_m2[7] =0xfffffffffffff8f9;
    m_m2[8] =0xfffffffffffff8f3;
    m_m2[9] =0xfffffffffffff8d1;
    m_m2[10]=0xfffffffffffff8bd;
    m_m2[11]=0xfffffffffffff8a5;
    m_m2[12]=0xfffffffffffff863;
    m_m2[13]=0xfffffffffffff835;
    m_m2[14]=0xfffffffffffff82d;
    m_m2[15]=0xfffffffffffff80f;
    m_m2[16]=0xfffffffffffff803;
    m_m2[17]=0xfffffffffffff7cf;
    m_m2[18]=0xfffffffffffff7ab;
    m_m2[19]=0xfffffffffffff781;
    m_m2[20]=0xfffffffffffff733;
    m_m2[21]=0xfffffffffffff713;
    m_m2[22]=0xfffffffffffff70f;
    m_m2[23]=0xfffffffffffff6fb;
    m_m2[24]=0xfffffffffffff6b5;
    m_m2[25]=0xfffffffffffff661;
    m_m2[26]=0xfffffffffffff643;
    m_m2[27]=0xfffffffffffff60b;
    m_m2[28]=0xfffffffffffff605;
    m_m2[29]=0xfffffffffffff5db;
    m_m2[30]=0xfffffffffffff5b7;
    m_m2[31]=0xfffffffffffff563;

    BN_WORD_setone(m_M1);
    BN_WORD_setone(m_M2);
    for(int i=0;i<m_base_num;i++){
        BN_PART_BN_WORD_transform(m_m1[i],bn_m);
        BN_WORD_mul(bn_m,m_M1,m_M1);
    }
    for(int i=0;i<m_base_num;i++){
        BN_PART_BN_WORD_transform(m_m2[i],bn_m);
        BN_WORD_mul(bn_m,m_M2,m_M2);
    }
    BN_WORD_mod(m_M1,m_rsa_n->n,m_M1_n);
    BN_WORD_mod(m_M2,m_rsa_n->n,m_M2_n);
    for(int i=0;i<m_base_num;i++){
        BN_PART_BN_WORD_transform(m_m1[i], bn_m);
        BN_WORD_div(m_M1,bn_m,m_M1_i[i],r);
        BN_PART_BN_WORD_transform(m_m2[i], bn_m);
        BN_WORD_div(m_M2,bn_m,m_M2_i[i],r);
        BN_WORD_BN_PART_mod(m_M1_i[i],m_m1[i],M1_i_m1_i);
        BN_PART_mod_inverse(M1_i_m1_i,m_m1[i],M1_i_inverse);
        BN_PART_BN_WORD_transform(M1_i_inverse,bn_M1_i_inverse);
        BN_WORD_mul_mod(bn_M1_i_inverse,m_M1_i[i],m_M1,m_M1_red_i[i]);
        BN_WORD_BN_PART_mod(m_M2_i[i],m_m2[i],M2_i_m2_i);
	BN_PART_mod_inverse(M2_i_m2_i,m_m2[i],M2_i_inverse);
        BN_PART_BN_WORD_transform(M2_i_inverse, bn_M2_i_inverse);
        BN_WORD_mul_mod(bn_M2_i_inverse,m_M2_i[i],m_M2,m_M2_red_i[i]);
    }
    for(int i=0;i<m_base_num;i++){
        BN_WORD_BN_PART_mod(m_M1_i[i],m_m1[i],M);
        BN_WORD_BN_PART_mod(m_rsa_n->n,m_m1[i],p_temp);
	BN_PART_mul_mod(M,p_temp,m_m1[i],M);
	BN_PART_mod_inverse(M,m_m1[i],M_inverse);
        m_d[i]=m_m1[i]-M_inverse;
    }
    for(int i=0;i<m_base_num;i++){
        BN_WORD_BN_PART_mod(m_M2_i[i],m_m2[i],M);
        BN_WORD_BN_PART_mod(m_M1,m_m2[i],M_inverse);
	BN_PART_mul_mod(M,M_inverse,m_m2[i],M);
	BN_PART_mod_inverse(M,m_m2[i],m_e[i]);
    }
    for(int i=0;i<m_base_num;i++){
        for(int j=0;j<m_base_num;j++){
            BN_WORD_BN_PART_mod(m_M2_i[i],m_m2[i],M);
	    BN_PART_mul_mod(M,m_m1[j],m_m2[i],M);
            BN_PART_mod_inverse(M,m_m2[i],M_inverse);
            BN_WORD_BN_PART_mod(m_rsa_n->n,m_m2[i],p_temp);
	    BN_PART_mul_mod(M_inverse,p_temp,m_m2[i],m_a[i*m_base_num+j]);
        }
    }
    for(int i=0;i<m_base_num;i++){
        BN_WORD_BN_PART_mod(m_M2_i[i],m_m2[i],M);
        M=m_m2[i]-M;
        BN_PART_mod_inverse(M,m_m2[i],M_inverse);
        BN_WORD_BN_PART_mod(m_rsa_n->n,m_m2[i],p_temp);
	BN_PART_mul_mod(M_inverse,p_temp,m_m2[i],m_a_2[i]);
    }
    for(int i=0;i<m_base_num;i++){
        for(int j=0;j<m_base_num;j++){
            BN_WORD_BN_PART_mod(m_M2_i[j],m_m1[i],M);
            m_b[i*m_base_num+j]=M;
        }
    }
    for(int i=0;i<m_base_num;i++){
        BN_WORD_BN_PART_mod(m_M2,m_m1[i],M);
        M=m_m1[i]-M;
        m_b_2[i]=M;
    }
    for(int i=0;i<m_base_num;i++){
        BN_WORD_BN_PART_mod(m_M2_i[i],m_m2[i],M);
        m_c[i]=M;
    }
    BN_WORD_free(temp_result);
    BN_WORD_free(q);
    BN_WORD_free(bn_m);
    BN_WORD_free(r);
    BN_WORD_free(bn_M1_i_inverse);
}


__host__ RNS_N:: ~RNS_N(){
    hipFree(m_m1);
    hipFree(m_m2);
    BN_WORD_free(m_M1);
    BN_WORD_free(m_M2);
    BN_WORD_free(m_M1_n);
    BN_WORD_free(m_M2_n);
    for(int i=0;i<m_base_num;i++){
        BN_WORD_free(m_M1_i[i]);
        BN_WORD_free(m_M1_red_i[i]);
    }
    free(m_M1_i);
    free(m_M2_i);
    free(m_M1_red_i);
    free(m_M2_red_i);
    hipFree(m_d);
    hipFree(m_e);
    hipFree(m_a);
    hipFree(m_a_2);
    hipFree(m_b);
    hipFree(m_b_2);
    hipFree(m_c);
}

int RNS_N :: RNS_MUL_MOD(BN_WORD *a, BN_WORD *b, BN_WORD *result){
       int dmax=a->dmax;
    BN_WORD *a_temp, *b_temp;
    BN_PART *x_result;
    a_temp=BN_WORD_new(dmax);
    b_temp=BN_WORD_new(dmax);
    hipMallocManaged((void**)&(x_result),m_base_num*sizeof(BN_PART));
    BN_WORD_mul_mod(a, m_M1, m_rsa_n->n, a_temp); //a=a*M mod n
    BN_WORD_mul_mod(b, m_M1, m_rsa_n->n, b_temp);
    RNS_mul_mod_kernel<<<1,dmax>>>(a_temp,b_temp,m_base_num, m_m1,m_m2,m_d,m_e,m_a,m_a_2,m_b,m_b_2,m_c,x_result);
    hipDeviceSynchronize();
    RSA_RNS_reduction1(x_result,result);
    if(BN_WORD_cmp(result,m_rsa_n->n)==1){
        BN_WORD_sub(result,m_rsa_n->n,result);
    }
    BN_WORD_free(a_temp);
    BN_WORD_free(b_temp);
    hipFree(x_result);
    return 0;    
}

int RNS_N:: RSA_RNS_reduction1(BN_PART *x_result, BN_WORD *result){
    BN_WORD *result_add, *bn_x;
    bn_x=BN_WORD_new(m_base_num);
    result_add=BN_WORD_new(m_base_num);
    BN_WORD_setzero(result);
    for(int i=0;i<m_base_num;i++){
        BN_PART_BN_WORD_transform(x_result[i],bn_x);
        BN_WORD_mul_mod(bn_x,m_M1_red_i[i],m_M1,result_add);
        BN_WORD_add_mod(result,result_add,m_M1,result);
    }
    return 0;
}

}
