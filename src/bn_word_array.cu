#include "rsa_final.h"

namespace namespace_rsa_final{

__host__ BN_WORD_ARRAY *BN_WORD_ARRAY_new(int word_num, int dmax){
    BN_WORD_ARRAY *a;
    hipMallocManaged((void**)&(a),sizeof(BN_WORD_ARRAY));
    a->word_num=word_num;
    hipMallocManaged((void**)&(a->bn_word),sizeof(BN_WORD*));
    for(int i=0;i<word_num; i++){
        a->bn_word[i]=BN_WORD_new(dmax);
    }
    return a;
}

__host__ void BN_WORD_ARRAY_free(BN_WORD_ARRAY *a){
    for(int i=0;i<a->word_num;i++){
        BN_WORD_free(a->bn_word[i]);
    } 
    hipFree(a->bn_word);
    hipFree(a);

}

}
