#include "bn_num_operation.h"
#include "stdio.h"


BN_ULONG get_bit(BN_ULONG a,int i){
    return  (a&((BN_ULONG)1<<i))/((BN_ULONG)1<<i);
}


__host__ BN_NUM *BN_NUM_new(int wmax,int dmax){
    BN_NUM *a;
    hipMallocManaged((void**)&(a),sizeof(BN_NUM));
    a->wmax=wmax;
    hipMallocManaged((void**)&(a->word),sizeof(BN_WORD*)*wmax);
    for(int i=0;i<wmax;i++){
        *(a->word+i)=BN_WORD_new(dmax);
    }
    return a;
}

__device__ BN_NUM *BN_NUM_new_device(int wmax,int dmax){
    BN_NUM *a;
    a=(BN_NUM*)malloc(sizeof(BN_NUM));
    a->wmax=wmax;
    a->word=(BN_WORD **)malloc(sizeof(BN_WORD*)*wmax);
    for(int i=0;i<wmax;i++){
        *(a->word+i)=BN_WORD_new_device(dmax);
    }
    return a;

}
__host__ void BN_NUM_free(BN_NUM *a){
    for(int i=0;i<a->wmax;i++){
        BN_WORD_free(a->word[i]);
    }
    hipFree(a->word);
    hipFree(a);
}

__device__ void BN_NUM_free_device(BN_NUM *a){
    for(int i=0;i<a->wmax;i++){
        BN_WORD_free_device(a->word[i]);
    }
    free(a->word);
    free(a);
}

__host__ __device__ void BN_NUM_copy(const BN_NUM *a, BN_NUM *b){
    for(int i=0;i<a->wmax;i++){
        BN_WORD_copy(a->word[i],b->word[i]);
    }
}

__host__ __device__ void BN_NUM_setzero(BN_NUM *a){
    for(int i=0;i<a->wmax;i++){
        BN_WORD_setzero(a->word[i]);
    }
}

__host__ __device__ void BN_NUM_setone(BN_NUM *a){
    BN_WORD_setone(a->word[0]);
    for(int i=1;i<a->wmax;i++){
        BN_WORD_setzero(a->word[i]);
    }
}

__host__ __device__ int BN_NUM_cmp(const BN_NUM *a,const BN_NUM *b){
    int cmp;
    if(a->wmax!=b->wmax)
	    return -1;
    for(int i=a->wmax-1;i>=0;i--){
	cmp=BN_WORD_cmp(a->word[i],b->word[i]);
        if(cmp==1){
            return 1;
        }
        if(cmp==2){
            return 2;
        }
    }
    return 0;
}

__host__ __device__ void BN_NUM_print(const BN_NUM *a){
    printf("wmax:%d\n",a->wmax);
    for(int i=a->wmax-1;i>=0;i--){
        BN_WORD_print(a->word[i]);
     }
}

__host__ int BN_NUM_add(const BN_NUM *a,const BN_NUM *b,BN_NUM *result){
    int wmax=a->wmax;
    int dmax=a->word[0]->dmax;
    int cmp;
    BN_WORD *carry1,*carry2,*temp;
    carry1=BN_WORD_new(dmax);
    carry2=BN_WORD_new(dmax);
    temp=BN_WORD_new(dmax);
    BN_WORD_setzero(carry1);
    BN_WORD_setzero(carry2);
    BN_WORD_setzero(temp);
    for (int i=0;i<wmax;i++){
        BN_WORD_copy(carry1,carry2);
        BN_WORD_setzero(carry1);
	BN_WORD_add(a->word[i],carry2,temp);
	cmp=BN_WORD_cmp(a->word[i],temp);
        if(cmp==1){
            BN_WORD_setone(carry1);
        }
	BN_WORD_add(temp,b->word[i],result->word[i]);
	cmp=BN_WORD_cmp(b->word[i],result->word[i]);
        if(cmp==1){
            BN_WORD_setone(carry1);
        }
    }
    BN_WORD_free(carry1);
    BN_WORD_free(carry2);
    BN_WORD_free(temp);
    return 0;
}


__device__ int BN_NUM_add_device(const BN_NUM *a,const BN_NUM *b,BN_NUM *result){
    int wmax=a->wmax;
    int dmax=a->word[0]->dmax;
    int cmp;
    BN_WORD *carry1,*carry2,*temp;
    carry1=BN_WORD_new_device(dmax);
    carry2=BN_WORD_new_device(dmax);
    temp=BN_WORD_new_device(dmax);
    BN_WORD_setzero(carry1);
    BN_WORD_setzero(carry2);
    BN_WORD_setzero(temp);
    for (int i=0;i<wmax;i++){
        BN_WORD_copy(carry1,carry2);
        BN_WORD_setzero(carry1);
        BN_WORD_add(a->word[i],carry2,temp);
        cmp=BN_WORD_cmp(a->word[i],temp);
        if(cmp==1){
            BN_WORD_setone(carry1);
        }
        BN_WORD_add(temp,b->word[i],result->word[i]);
        cmp=BN_WORD_cmp(b->word[i],result->word[i]);
        if(cmp==1){
            BN_WORD_setone(carry1);
        }
    }
    BN_WORD_free_device(carry1);
    BN_WORD_free_device(carry2);
    BN_WORD_free_device(temp);
    return 0;
}


__host__ int BN_NUM_sub(const BN_NUM *a,const BN_NUM *b,BN_NUM *result){
    int wmax=a->wmax;
    int cmp;
    int dmax=a->word[0]->dmax;
    BN_WORD *carry1, *carry2,*temp;
    carry1=BN_WORD_new(dmax);
    carry2=BN_WORD_new(dmax);
    temp=BN_WORD_new(dmax);
    BN_WORD_setzero(carry1);
    BN_WORD_setzero(carry2);
    BN_WORD_setzero(temp);
    for(int i=0;i<wmax;i++){
        BN_WORD_copy(carry1,carry2);
        BN_WORD_setzero(carry1);
        BN_WORD_sub(a->word[i],b->word[i],temp);
        cmp=BN_WORD_cmp(a->word[i],temp);
        if(cmp==2){
            BN_WORD_setone(carry1);
        }
        BN_WORD_sub(temp,carry2,result->word[i]);
        cmp=BN_WORD_cmp(temp,result->word[i]);
        if(cmp==2){
            BN_WORD_setone(carry1);
        }
    }
    BN_WORD_free(carry1);
    BN_WORD_free(carry2);
    BN_WORD_free(temp);
    return 0;
}


__device__ int BN_NUM_sub_device(const BN_NUM *a,const BN_NUM *b,BN_NUM *result){
    int wmax=a->wmax;
    int cmp;
    int dmax=a->word[0]->dmax;
    BN_WORD *carry1, *carry2,*temp;
    carry1=BN_WORD_new_device(dmax);
    carry2=BN_WORD_new_device(dmax);
    temp=BN_WORD_new_device(dmax);
    BN_WORD_setzero(carry1);
    BN_WORD_setzero(carry2);
    BN_WORD_setzero(temp);
    for(int i=0;i<wmax;i++){
	BN_WORD_copy(carry1,carry2);
	BN_WORD_setzero(carry1);
        BN_WORD_sub(a->word[i],b->word[i],temp);
	cmp=BN_WORD_cmp(a->word[i],temp);
	if(cmp==2){
	    BN_WORD_setone(carry1);
	}
	BN_WORD_sub(temp,carry2,result->word[i]);
	cmp=BN_WORD_cmp(temp,result->word[i]);
	if(cmp==2){
	    BN_WORD_setone(carry1);
	}
    }
    BN_WORD_free_device(carry1);
    BN_WORD_free_device(carry2);
    BN_WORD_free_device(temp);
    return 0;
}

__host__ int BN_NUM_left_shift_bits(const BN_NUM *a,BN_NUM *b,int bits){
    int wmax=a->wmax;
    int dmax=a->word[0]->dmax;
    int bits_bn_ulong=sizeof(BN_ULONG)*8;
    int shift_num=bits/(bits_bn_ulong);
    int real_bits=bits%(bits_bn_ulong);
    int j,i_w,i_d,j_w,j_d,i_sub_w,i_sub_d;
    for(int i=0;i<wmax*dmax-shift_num;i++){
        j=i+shift_num;
	i_w=i/dmax;
	i_d=i%dmax;
	j_w=j/dmax;
	j_d=j%dmax;
	i_sub_w=(i-1)/dmax;
	i_sub_d=(i-1)%dmax;
	if(i==0){
	b->word[j_w]->d[j_d]=(a->word[i_w]->d[i_d])<<real_bits;
	}
	else{
	    if(real_bits==0){
		    b->word[j_w]->d[j_d]=((a->word[i_w]->d[i_d])<<real_bits)
                +((a->word[i_sub_w]->d[i_sub_d])/((BN_ULONG)1<<(bits_bn_ulong-real_bits-1))/((BN_ULONG)1<<1));
	    }    
	    else{
	    	    b->word[j_w]->d[j_d]=((a->word[i_w]->d[i_d])<<real_bits)
		+((a->word[i_sub_w]->d[i_sub_d])/((BN_ULONG)1<<(bits_bn_ulong-real_bits)));
	    }
	}
    }
    for(int j=0;j<shift_num;j++){
	j_w=j/dmax;
        j_d=j%dmax;
	b->word[j_w]->d[j_d]=0;
    }
    return 0;
}
/*
__host__ int BN_NUM_left_shift_bits(const BN_NUM *a,BN_NUM *b,int bits){
    int wmax=a->wmax;
    int dmax=a->word[0]->dmax;
    int bits_bn_ulong=sizeof(BN_ULONG)*8;
    int shift_num=bits/(bits_bn_ulong);
    int real_bits=bits%(bits_bn_ulong);
    int j,i_w,i_d,j_w,j_d,i_sub_w,i_sub_d;
    for(int i=0;i<wmax*dmax-shift_num;i++){
        j=i+shift_num;
        i_w=i/dmax;
        i_d=i%dmax;
        j_w=j/dmax;
        j_d=j%dmax;
        i_sub_w=(i-1)/dmax;
        i_sub_d=(i-1)%dmax;
        if(i==0){
        b->word[j_w]->d[j_d]=(a->word[i_w]->d[i_d])<<real_bits;
        }
        b->word[j_w]->d[j_d]=((a->word[i_w]->d[i_d])<<real_bits)
                +((a->word[i_sub_w]->d[i_sub_d])/((BN_ULONG)1<<(bits_bn_ulong-real_bits)));
    }
    return 0;
}
*/

__host__ int BN_NUM_right_shift_bits(const BN_NUM *a,BN_NUM *b,int bits){
    int wmax=a->wmax;
    int dmax=a->word[0]->dmax;
    int bits_bn_ulong=sizeof(BN_ULONG)*8;
    int shift_num=bits/(bits_bn_ulong);
    int real_bits=bits%(bits_bn_ulong);
    int j,i_w,i_d,j_w,j_d,i_add_w,i_add_d;
    for(int i=shift_num;i<wmax*dmax;i++){
        j=i-shift_num;
        i_w=i/dmax;
        i_d=i%dmax;
        j_w=j/dmax;
        j_d=j%dmax;
        i_add_w=(i+1)/dmax;
        i_add_d=(i+1)%dmax;
	if(i==(dmax*wmax-1)){
        b->word[j_w]->d[j_d]=(a->word[i_w]->d[i_d])/((BN_ULONG)1<<real_bits);
	}
	else{
	    if(real_bits==0){
		    b->word[j_w]->d[j_d]=((a->word[i_w]->d[i_d])/((BN_ULONG)1<<real_bits))
                +((a->word[i_add_w]->d[i_add_d])<<(bits_bn_ulong-real_bits-1)<<1);
	    }
	    else{
	    	    b->word[j_w]->d[j_d]=((a->word[i_w]->d[i_d])/((BN_ULONG)1<<real_bits))
                +((a->word[i_add_w]->d[i_add_d])<<(bits_bn_ulong-real_bits));	    
	    }
	}
    }
    for(int j=wmax*dmax-shift_num;j<wmax*dmax;j++){
	j_w=j/dmax;
        j_d=j%dmax;
        b->word[j_w]->d[j_d]=0;
    }
    return 0;
}




__host__ int BN_NUM_mul(const BN_NUM *a, const BN_NUM *b, BN_NUM *result){
    int wmax=a->wmax;
    int dmax=a->word[0]->dmax;
    int bits_bn_ulong=sizeof(BN_ULONG)*8;
    int shift_bits;
    BN_ULONG mul_value;
    BN_NUM *a_shift;
    BN_NUM *temp_result;
    a_shift=BN_NUM_new(wmax,dmax);
    temp_result=BN_NUM_new(wmax,dmax);
    BN_NUM_setzero(result);
    for(int w=0;w<wmax;w++){
        for(int d=0;d<dmax;d++){
	    for(int i=0;i<bits_bn_ulong;i++){
	        shift_bits=w*dmax*bits_bn_ulong+d*bits_bn_ulong+i;
		mul_value=get_bit(b->word[w]->d[d],i);
		if(mul_value==(BN_ULONG)1){
		    BN_NUM_setzero(a_shift);
		    BN_NUM_left_shift_bits(a,a_shift,shift_bits);
		    BN_NUM_add(result,a_shift,temp_result);
		    BN_NUM_copy(temp_result,result);
		}
	    }
	}
    }
   return 0; 
}

__host__ int BN_NUM_div(const BN_NUM *a, const BN_NUM *b, BN_NUM *q, BN_NUM *r){
    int wmax=a->wmax;
    int dmax=a->word[0]->dmax;
    BN_NUM_setzero(q);
    BN_NUM *one,*a_temp,*b_temp,*temp_result,*div_temp;
    one=BN_NUM_new(wmax,dmax);
    a_temp=BN_NUM_new(wmax,dmax);
    b_temp=BN_NUM_new(wmax,dmax);
    temp_result=BN_NUM_new(wmax,dmax);
    div_temp=BN_NUM_new(wmax,dmax);
    BN_NUM_setone(one);
    int shift_num=0;
    if(BN_NUM_cmp(a,b)==2){
        BN_NUM_setzero(q);
	BN_NUM_copy(a,r);
	return 0;
    }
    if(BN_NUM_cmp(a,b)==0){
        BN_NUM_setone(q);
	BN_NUM_setzero(r);
	return 0;
    }
    BN_NUM_copy(a,a_temp);
    while((BN_NUM_cmp(a_temp,b)==1)||(BN_NUM_cmp(a_temp,b)==0)){
        shift_num ++;
	BN_NUM_right_shift_bits(a_temp,temp_result,1);
	BN_NUM_copy(temp_result,a_temp);
    }
    shift_num --;
  //  printf("shift_num:%d\n",shift_num);
    BN_NUM_copy(a,a_temp);
    BN_NUM_left_shift_bits(b,b_temp,shift_num);
    for(int i=shift_num;i>=0;i--){
        if(BN_NUM_cmp(a_temp,b_temp)==1){
	    BN_NUM_sub(a_temp,b_temp,a_temp);
	    BN_NUM_left_shift_bits(one,div_temp,i);
//	    printf("div_temp:\n");
//	    BN_NUM_print(div_temp);
	    BN_NUM_add(q,div_temp,q);
	}
	BN_NUM_right_shift_bits(b_temp,temp_result,1);
	BN_NUM_copy(temp_result,b_temp);
    }
    BN_NUM_copy(a_temp,r);
    return 0;
}


/*
__host__ int BN_NUM_mul(const BN_NUM *a, const BN_NUM *b, BN_NUM *result){
    int wmax=a->wmax;
    int dmax=a->word[0]->dmax;
    int i,j,m;
    BN_WORD *carry1, *carry2, *carry3, *carry4, *temp_result, *temp_mul_result;
    carry1=BN_WORD_new(2*dmax);
    carry2=BN_WORD_new(2*dmax);
    carry3=BN_WORD_new(2*dmax);
    carry4=BN_WORD_new(2*dmax);
    temp_result=BN_WORD_new(2*dmax);
    temp_mul_result=BN_WORD_new(2*dmax);
    for(m=0;m<2*wmax-1;m++){
        for(i=0;i<=m;i++){
	    j=m-i;
	    BN_WORD_mul(a->word[i],b->word[j],temp_mul_result);
	    BN_WORD_add(temp_result,temp_mul_result,temp_result);
	    if(temp_result->carry==1){
	        temp_result->carry=0;
		BN_WORD_add(carry4,one,carry4);
	    }
	}
	BN_WORD_add()
    }
}
*/

