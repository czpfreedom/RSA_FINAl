#include "bn_num_operation.h"

__host__ BN_NUM *BN_NUM_new(int wmax,int dmax){
    BN_NUM *a;
    hipMallocManaged((void**)&(a),sizeof(BN_NUM));
    a->wmax=wmax;
    hipMallocManaged((void**)&(a->word),sizeof(BN_WORD*)*wmax);
    for(int i=0;i<wmax;i++){
        *(a->word+i)=BN_WORD_new(dmax);
    }
    return a;
}

__device__ BN_NUM *BN_NUM_new_device(int wmax,int dmax){
    BN_NUM *a;
    a=(BN_NUM*)malloc(sizeof(BN_NUM));
    a->wmax=wmax;
    a->word=(BN_WORD **)malloc(sizeof(BN_WORD*)*wmax);
    for(int i=0;i<wmax;i++){
        *(a->word+i)=BN_WORD_new_device(dmax);
    }
    return a;

}
__host__ void BN_NUM_free(BN_NUM *a){
    for(int i=0;i<a->wmax;i++){
        BN_WORD_free(*(a->word+1));
    }
    hipFree(a->word);
    hipFree(a);
}

__device__ void BN_NUM_free_device(BN_NUM *a){
    for(int i=0;i<a->wmax;i++){
        BN_WORD_free_device(*(a->word+1));
    }
    free(a->word);
    free(a);
}

