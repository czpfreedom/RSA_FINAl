#include "bn_word.h"
#include "stdlib.h"
#include <stdio.h>

__host__ BN_WORD *BN_WORD_new(int dmax){
    BN_WORD *a;
    hipMallocManaged((void**)&(a),sizeof(BN_WORD));
    a->dmax=dmax;
    hipMallocManaged((void**)&(a->d),dmax*sizeof(BN_PART));
    return a;
}

__host__ void BN_WORD_free(BN_WORD *a){
    hipFree(a->d);
    hipFree(a);
}

__device__ BN_WORD *BN_WORD_new_device(int dmax){
    BN_WORD *a;
    a=(BN_WORD*)malloc(sizeof(BN_WORD));
    a->dmax=dmax;
    a->d=(BN_PART*)malloc(dmax*sizeof(BN_PART));
    return a;
}

__device__ void BN_WORD_free_device(BN_WORD *a){
    free(a->d);
    free(a);
}

__host__ __device__ void BN_WORD_setzero(BN_WORD *a){
    for(int i=0;i<a->dmax;i++){
        a->d[i]=0;
    }
}

__host__ __device__ void BN_WORD_setone(BN_WORD *a){
    a->d[0]=1;
    for(int i=1;i<a->dmax;i++){
        a->d[i]=0;
    }
}


__host__ __device__ int BN_WORD_copy(const BN_WORD *a,BN_WORD *b){
    if(a->dmax!=b->dmax){
        return -1;
    }
    for(int i=0;i<a->dmax;i++){
        b->d[i]=a->d[i];
    }
    return 0;
}

__host__ int BN_WORD_print(const BN_WORD *a){
    printf("dmax:%d\n",a->dmax);
    for(int i=(a->dmax)-1;i>=0;i--){
#ifdef BN_PART_32
	printf("%x,",a->d[i]);
#endif
#ifdef BN_PART_64
        printf("%lx,",a->d[i]);
#endif
    }
    printf("\n");
    return 0;
}

__device__ int BN_WORD_print_device(const BN_WORD *a){
    printf("dmax:%d\n",a->dmax);
    for(int i=(a->dmax)-1;i>=0;i--){
#ifdef BN_PART_32
	printf("%x,",a->d[i]);
#endif
#ifdef BN_PART_64
        printf("%lx,",a->d[i]);
#endif
    }
    printf("\n");
    return 0;
}

__host__ __device__ int BN_WORD_cmp(const BN_WORD *a,const BN_WORD *b){
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    for(int i=(a->dmax)-1;i>=0;i--){
        if(a->d[i]>b->d[i]){
            return 1;
        }
        if(a->d[i]<b->d[i]){
            return 2;
        }
    }
    return 0;
}

__host__ __device__ BN_PART bn_word_get_bit(const BN_WORD *a, int i){
    return get_bit(a->d[i/(sizeof(BN_PART)*8)],i%(sizeof(BN_PART)*8));
}

__host__ __device__ int BN_WORD_left_shift(const BN_WORD *a,BN_WORD *b,int words){
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    if((a->dmax)<words){
        return -2;
    }
    for(int i=(a->dmax)-1;i>=words;i--){
        b->d[i]=a->d[i-words];
    }
    for(int i=words-1;i>=0;i--){
        b->d[i]=0;
    }
    return 0;
}


__host__ __device__ int BN_WORD_left_shift_bits(const BN_WORD *a,BN_WORD *b,int bits){
    int num_bits=bits%(sizeof(BN_PART)*8);
    int num_bnpart=bits/(sizeof(BN_PART)*8);
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    b->d[num_bnpart]=a->d[0]<<num_bits;
    for (int i=1+num_bnpart;i<a->dmax;i++){
	if(num_bits==0){
	    b->d[i]=((a->d[i-num_bnpart])<<num_bits);
	}
	else{
	    b->d[i]=((a->d[i-num_bnpart])<<num_bits)+((a->d[i-1-num_bnpart])>>(sizeof(BN_PART)*8-num_bits));
	}
    }
    for (int i=0;i<num_bnpart;i++){
        b->d[i]=0;
    }
    return 0;
}

__host__ __device__ int BN_WORD_right_shift(const BN_WORD *a,BN_WORD *b,int words){
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    if((a->dmax)<words){
        return -3;
    }
    for(int i=0;i<a->dmax-words;i++){
        b->d[i]=a->d[i+words];
    }
    for(int i=a->dmax-words;i<a->dmax;i++){
        b->d[i]=0;
    }
    return 0;
}

__host__ __device__ int BN_WORD_right_shift_bits(const BN_WORD *a,BN_WORD *b,int bits){
    int num_bits=bits%(sizeof(BN_PART)*8);
    int num_bnpart=bits/(sizeof(BN_PART)*8);
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    for (int i=0;i<a->dmax-1-num_bnpart;i++){
	if(num_bits==0){
	    b->d[i]=(a->d[i+num_bnpart])>>num_bits;
	}
	else{
	    b->d[i]=((a->d[i+num_bnpart])>>num_bits)+((a->d[i+num_bnpart+1])<<(sizeof(BN_PART)*8-num_bits));
	}
    }
    b->d[a->dmax-1-num_bnpart]=(a->d[a->dmax-1])>>num_bits;
    for(int i=a->dmax-num_bnpart;i<a->dmax;i++){
        b->d[i]=0;
    }
    return 0;
}

__host__ __device__ int BN_WORD_add(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    BN_PART mid_value;
    BN_PART carry1=0;
    BN_PART carry2=0;
    for (int i=0;i<a->dmax;i++){
        carry2=carry1;
        carry1=0;
        mid_value=a->d[i]+carry2;
        if(mid_value<a->d[i]){
            carry1=1;
        }
        mid_value=mid_value+b->d[i];
        if(mid_value<b->d[i]){
            carry1=1;
        }
        result->d[i]=mid_value;
    }
    return 0;
}

__host__ __device__ int BN_WORD_sub(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    BN_PART mid_value1, mid_value;
    BN_PART carry1,carry2;
    int cmp=BN_WORD_cmp(a,b);
    if(cmp==0){
        BN_WORD_setzero(result);
	return 0;
    }
    result->dmax=a->dmax;
    carry2=0;
    carry1=0;
    for(int i=0;i<a->dmax;i++){
        carry2=carry1;
	carry1=0;
	mid_value1=a->d[i]-carry2;
	if(mid_value1>a->d[i]){
	    carry1=1;
	}
	mid_value=mid_value1-b->d[i];
	if(mid_value>mid_value1){
	    carry1=1;
	}
	result->d[i]=mid_value;
    }
    return 0;
}

__host__ int BN_WORD_mul(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    int dmax=a->dmax;
    BN_WORD *result_temp;
    BN_WORD *a_temp;
    result_temp=BN_WORD_new(dmax);
    a_temp=BN_WORD_new(dmax);
    BN_WORD_setzero(result_temp);
    BN_WORD_copy(a,a_temp);
    for (int i=0;i<dmax;i++){
        for(int j=0;j<sizeof(BN_PART)*8;j++){
	    if(get_bit(b->d[i],j)==1){
	        BN_WORD_add(result_temp,a_temp,result_temp);
	    }
	    BN_WORD_left_shift_bits(a,a_temp,i*sizeof(BN_PART)*8+j+1);
	}
    }
    BN_WORD_copy(result_temp,result);
    return 0;
}

__device__ int BN_WORD_mul_device(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    int dmax=a->dmax;
    BN_WORD *result_temp;
    BN_WORD *a_temp;
    result_temp=BN_WORD_new_device(dmax);
    a_temp=BN_WORD_new_device(dmax);
    BN_WORD_setzero(result_temp);
    BN_WORD_copy(a,a_temp);
    for (int i=0;i<dmax;i++){
        for(int j=0;j<sizeof(BN_PART)*8;j++){
            if(get_bit(b->d[i],j)==1){
                BN_WORD_add(result_temp,a_temp,result_temp);
            }
            BN_WORD_left_shift_bits(a,a_temp,i*sizeof(BN_PART)*8+j);
        }
    }
    BN_WORD_copy(result_temp,result);
    return 0;
}

__host__ int BN_WORD_div(const BN_WORD *a, const BN_WORD *b, BN_WORD *q, BN_WORD *r){
    int dmax=a->dmax;
    BN_WORD_setzero(q);
    BN_WORD *one,*a_temp,*b_temp,*temp_result,*div_temp;
    one=BN_WORD_new(dmax);
    a_temp=BN_WORD_new(dmax);
    b_temp=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    div_temp=BN_WORD_new(dmax);
    BN_WORD_setone(one);
    int shift_num=0;
    if(BN_WORD_cmp(a,b)==0){
        BN_WORD_setone(q);
        BN_WORD_setzero(r);
        return 0;
    }
    BN_WORD_copy(a,a_temp);
    while((BN_WORD_cmp(a_temp,b)==1)||(BN_WORD_cmp(a_temp,b)==0)){
        shift_num ++;
        BN_WORD_right_shift_bits(a_temp,temp_result,1);
        BN_WORD_copy(temp_result,a_temp);
    }
    shift_num --;
    BN_WORD_copy(a,a_temp);
    BN_WORD_left_shift_bits(b,b_temp,shift_num);
    BN_WORD_setzero(q);
    for(int i=shift_num;i>=0;i--){
        if((BN_WORD_cmp(a_temp,b_temp)==1)||(BN_WORD_cmp(a_temp,b_temp)==0)){
            BN_WORD_sub(a_temp,b_temp,temp_result);
	    BN_WORD_copy(temp_result,a_temp);
            BN_WORD_left_shift_bits(one,div_temp,i);
            BN_WORD_add(q,div_temp,temp_result);
	    BN_WORD_copy(temp_result,q);
        }
        BN_WORD_right_shift_bits(b_temp,temp_result,1);
        BN_WORD_copy(temp_result,b_temp);
    }
    BN_WORD_copy(a_temp,r);
    BN_WORD_free(one);
    BN_WORD_free(a_temp);
    BN_WORD_free(b_temp);
    BN_WORD_free(temp_result);
    BN_WORD_free(div_temp);
    return 0;
}

__device__ int BN_WORD_div_device(const BN_WORD *a, const BN_WORD *b, BN_WORD *q, BN_WORD *r){
    int dmax=a->dmax;
    BN_WORD_setzero(q);
    BN_WORD *one,*a_temp,*b_temp,*temp_result,*div_temp;
    one=BN_WORD_new_device(dmax);
    a_temp=BN_WORD_new_device(dmax);
    b_temp=BN_WORD_new_device(dmax);
    temp_result=BN_WORD_new_device(dmax);
    div_temp=BN_WORD_new_device(dmax);
    BN_WORD_setone(one);
    int shift_num=0;
    if(BN_WORD_cmp(a,b)==0){
        BN_WORD_setone(q);
        BN_WORD_setzero(r);
        return 0;
    }
    BN_WORD_copy(a,a_temp);
    while((BN_WORD_cmp(a_temp,b)==1)||(BN_WORD_cmp(a_temp,b)==0)){
        shift_num ++;
        BN_WORD_right_shift_bits(a_temp,temp_result,1);
        BN_WORD_copy(temp_result,a_temp);
    }
    shift_num --;
    BN_WORD_copy(a,a_temp);
    BN_WORD_left_shift_bits(b,b_temp,shift_num);
    BN_WORD_setzero(q);
    for(int i=shift_num;i>=0;i--){
        if((BN_WORD_cmp(a_temp,b_temp)==1)||(BN_WORD_cmp(a_temp,b_temp)==0)){
            BN_WORD_sub(a_temp,b_temp,temp_result);
            BN_WORD_copy(temp_result,a_temp);
            BN_WORD_left_shift_bits(one,div_temp,i);
            BN_WORD_add(q,div_temp,temp_result);
            BN_WORD_copy(temp_result,q);
        }
        BN_WORD_right_shift_bits(b_temp,temp_result,1);
        BN_WORD_copy(temp_result,b_temp);
    }
    BN_WORD_copy(a_temp,r);
    BN_WORD_free_device(one);
    BN_WORD_free_device(a_temp);
    BN_WORD_free_device(b_temp);
    BN_WORD_free_device(temp_result);
    BN_WORD_free_device(div_temp);
    return 0;
}

__host__ int BN_WORD_mod (const BN_WORD *a, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;
    BN_WORD *q;
    q =BN_WORD_new(dmax);
    BN_WORD_div(a,n,q,result);
    BN_WORD_free(q);
    return 0;
}

__device__ int BN_WORD_mod_device (const BN_WORD *a, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;
    BN_WORD *q;
    q =BN_WORD_new_device(dmax);
    BN_WORD_div_device(a,n,q,result);
    BN_WORD_free_device(q);
    return 0;
}

__host__ int BN_WORD_add_mod(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;
    BN_WORD *q, *a_temp, *b_temp, *temp_result;
    q=BN_WORD_new(dmax);
    a_temp=BN_WORD_new(dmax);
    b_temp=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    BN_WORD_div(a,n,q,a_temp);
    BN_WORD_div(b,n,q,b_temp);
    BN_WORD_add(a_temp,b_temp,temp_result);
    if(BN_WORD_cmp(a_temp,temp_result)==1){
        BN_WORD_sub(temp_result,n,temp_result);
    }
    BN_WORD_copy(temp_result,result);
    return 0;
}

__host__ int BN_WORD_mul_mod(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;
    int bit;
    BN_WORD *a_sub, *b_sub, *temp_result1, *temp_result2;
    a_sub=BN_WORD_new(dmax);
    b_sub=BN_WORD_new(dmax);
    temp_result1=BN_WORD_new(dmax);
    temp_result2=BN_WORD_new(dmax);
    BN_WORD_copy(a,a_sub);
    BN_WORD_copy(b,b_sub);
    BN_WORD_mod(a_sub,n,a_sub);
    if(BN_WORD_cmp(a_sub,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    BN_WORD_mod(b_sub,n,b_sub);
    if(BN_WORD_cmp(b_sub,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    BN_WORD_setzero(temp_result1);
    BN_WORD_setzero(temp_result2);
    for(int i=dmax-1;i>=0;i--){
        for(int j=sizeof(BN_PART)*8-1;j>=0;j--){
                bit=get_bit(b_sub->d[i],j);
                BN_WORD_add(temp_result1,temp_result1,temp_result2);
                if((BN_WORD_cmp(temp_result1, temp_result2)==2)||(BN_WORD_cmp(temp_result1,n)==1)||(BN_WORD_cmp(temp_result1,n)==0)){
                    BN_WORD_sub(temp_result2,n,temp_result1);
                }
                else {
                    BN_WORD_copy(temp_result2,temp_result1);
                }
                if(bit==1){
                    BN_WORD_add(temp_result1,a_sub,temp_result2);
                    if((BN_WORD_cmp(temp_result1, temp_result2)==2)||(BN_WORD_cmp(temp_result1,n)==1)||(BN_WORD_cmp(temp_result1,n)==0)){
                            BN_WORD_sub(temp_result1,n,temp_result2);
                    }
                    else {
                            BN_WORD_copy(temp_result2,temp_result1);
                    }
                }
        }
    }
    BN_WORD_copy(temp_result1,result);
    BN_WORD_free(a_sub);
    BN_WORD_free(b_sub);
    BN_WORD_free(temp_result1);
    BN_WORD_free(temp_result2);
    return 0;
}

