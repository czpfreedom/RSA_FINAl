#include "bn_word_operation.h"
#include "bn_openssl.h"
#include "stdlib.h"
#include "iostream"
#include <stdio.h>


__host__ BN_WORD_32 *BN_WORD_32_new(int dmax){
    BN_WORD_32 *a;
    hipMallocManaged((void**)&(a),sizeof(BN_WORD_32));
    a->dmax=dmax;
    hipMallocManaged((void**)&(a->d),dmax*sizeof(BN_PART_32));
    return a;
}

__host__ void BN_WORD_32_free(BN_WORD_32 *a){
    hipFree(a->d);
    hipFree(a);
}

__device__ BN_WORD_32 *BN_WORD_new_device(int dmax){
    BN_WORD_32 *a;
    a=(BN_WORD_32*)malloc(sizeof(BN_WORD_32));
    a->dmax=dmax;
    a->d=(BN_PART_32 *)malloc(dmax*sizeof(BN_PART_32));
    return a;
}

__device__ void BN_WORD_32_free_device(BN_WORD_32 *a){
    free(a->d);
    free(a);
}

__host__ __device__ void BN_WORD_32_setzero(BN_WORD_32 *a){
    for(int i=0;i<a->dmax;i++){
        a->d[i]=0;
    }
}

__host__ __device__ void BN_WORD_setone(BN_WORD *a){
    a->d[0]=1;
    for(int i=1;i<a->dmax;i++){
        a->d[i]=0;
    }
}


__host__ __device__ int BN_WORD_copy(const BN_WORD *a,BN_WORD *b){
    if(a->dmax!=b->dmax){
        return -1;
    }
    for(int i=0;i<a->dmax;i++){
        b->d[i]=a->d[i];
    }
    return 0;
}

__host__ __device__ void BN_WORD_print(const BN_WORD *a){
    printf("dmax:%d\n",a->dmax);
    for(int i=(a->dmax)-1;i>=0;i--){
#ifdef BN_PART_32
	printf("%x,",a->d[i]);
#endif
#ifdef BN_PART_64
        printf("%lx,",a->d[i]);
#endif
    }
    printf("\n");
}

__host__ __device__ int BN_WORD_cmp(const BN_WORD *a,const BN_WORD *b){
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    for(int i=(a->dmax)-1;i>=0;i--){
        if(a->d[i]>b->d[i]){
            return 1;
        }
        if(a->d[i]<b->d[i]){
            return 2;
        }
    }
    return 0;
}

__host__ __device__ int BN_WORD_left_shift(const BN_WORD *a,BN_WORD *b,int words){
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    if((a->dmax)<words){
        return -3;
    }
    for(int i=(a->dmax)-1;i>=words;i--){
        b->d[i]=a->d[i-words];
    }
    for(int i=words-1;i>=0;i--){
        b->d[i]=0;
    }
    return 0;
}


__host__ __device__ int BN_WORD_left_shift_bits(const BN_WORD *a,BN_WORD *b,int bits){
    int num_bits=bits%(sizeof(BN_PART)*8);
    int num_bnpart=bits/(sizeof(BN_PART)*8);
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    b->d[num_bnpart]=a->d[0]<<num_bits;
    for (int i=1+num_bnpart;i<a->dmax;i++){
	if(num_bits==0){
	    b->d[i]=((a->d[i-num_bnpart])<<num_bits);
	}
	else{
		b->d[i]=((a->d[i-num_bnpart])<<num_bits)+((a->d[i-1-num_bnpart])>>(sizeof(BN_PART)*8-num_bits));
	}
    }
    for (int i=0;i<num_bnpart;i++){
        b->d[i]=0;
    }
    return 0;
}

__host__ __device__ int BN_WORD_right_shift(const BN_WORD *a,BN_WORD *b,int words){
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    if((a->dmax)<words){
        return -3;
    }
    for(int i=0;i<a->dmax-words;i++){
        b->d[i]=a->d[i+words];
    }
    for(int i=a->dmax-words;i<a->dmax;i++){
        b->d[i]=0;
    }
    return 0;
}

__host__ __device__ int BN_WORD_right_shift_bits(const BN_WORD *a,BN_WORD *b,int bits){
    int num_bits=bits%(sizeof(BN_PART)*8);
    int num_bnpart=bits/(sizeof(BN_PART)*8);
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    for (int i=0;i<a->dmax-1-num_bnpart;i++){
	if(num_bits==0){
	    b->d[i]=(a->d[i+num_bnpart])>>num_bits;
	}
	else{
	    b->d[i]=((a->d[i+num_bnpart])>>num_bits)+((a->d[i+num_bnpart+1])<<(sizeof(BN_PART)*8-num_bits));
	}
    }
    b->d[a->dmax-1-num_bnpart]=(a->d[a->dmax-1])>>num_bits;
    for(int i=a->dmax-num_bnpart;i<a->dmax;i++){
        b->d[i]=0;
    }
    return 0;
}

__host__ __device__ int BN_WORD_add(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    BN_PART mid_value;
    BN_PART carry1=0;
    BN_PART carry2=0;
    if((a->dmax!=b->dmax)||(a->dmax!=result->dmax)){
        return -1;
    }
    for (int i=0;i<a->dmax;i++){
        carry2=carry1;
        carry1=0;
        mid_value=a->d[i]+carry2;
        if(mid_value<a->d[i]){
            carry1=1;
        }
        mid_value=mid_value+b->d[i];
        if(mid_value<b->d[i]){
            carry1=1;
        }
        result->d[i]=mid_value;
    }
    return 0;
}

__host__ __device__ int BN_WORD_sub(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    BN_PART mid_value1, mid_value;
    BN_PART carry1,carry2;
    int cmp=BN_WORD_cmp(a,b);
    if(cmp==-1){
        return -1;
    }
    if(cmp==-2){
        return -2;
    }
    if(cmp==0){
        BN_WORD_setzero(result);
	return 0;
    }
    result->dmax=a->dmax;
    carry2=0;
    carry1=0;
    for(int i=0;i<a->dmax;i++){
        carry2=carry1;
	carry1=0;
	mid_value1=a->d[i]-carry2;
	if(mid_value1>a->d[i]){
	    carry1=1;
	}
	mid_value=mid_value1-b->d[i];
	if(mid_value>mid_value1){
	    carry1=1;
	}
	result->d[i]=mid_value;
    }
    return 0;
}

__host__ int BN_WORD_mul(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    int dmax=a->dmax;
    BN_WORD *result_temp;
    BN_WORD *a_temp;
    result_temp=BN_WORD_new(dmax);
    a_temp=BN_WORD_new(dmax);
    BN_WORD_setzero(result_temp);
    BN_WORD_copy(a,a_temp);
    for (int i=0;i<dmax;i++){
        for(int j=0;j<sizeof(BN_PART)*8;j++){
	    if(get_bit(b->d[i],j)==1){
	        BN_WORD_add(result_temp,a_temp,result_temp);
	    }
	    BN_WORD_left_shift_bits(a,a_temp,i*sizeof(BN_PART)*8+j+1);
	}
    }
    BN_WORD_copy(result_temp,result);
    return 0;
}

__device__ int BN_WORD_mul_device(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    int dmax=a->dmax;
    BN_WORD *result_temp;
    BN_WORD *a_temp;
    result_temp=BN_WORD_new_device(dmax);
    a_temp=BN_WORD_new_device(dmax);
    BN_WORD_setzero(result_temp);
    BN_WORD_copy(a,a_temp);
    for (int i=0;i<dmax;i++){
        for(int j=0;j<sizeof(BN_PART)*8;j++){
            if(get_bit(b->d[i],j)==1){
                BN_WORD_add(result_temp,a_temp,result_temp);
            }
            BN_WORD_left_shift_bits(a,a_temp,i*sizeof(BN_PART)*8+j);
        }
    }
    BN_WORD_copy(result_temp,result);
    return 0;
}

__host__ int BN_WORD_div(const BN_WORD *a, const BN_WORD *b, BN_WORD *q, BN_WORD *r){
    int dmax=a->dmax;
    BN_WORD_setzero(q);
    BN_WORD *one,*a_temp,*b_temp,*temp_result,*div_temp;
    one=BN_WORD_new(dmax);
    a_temp=BN_WORD_new(dmax);
    b_temp=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    div_temp=BN_WORD_new(dmax);
    BN_WORD_setone(one);
    int shift_num=0;
    if(BN_WORD_cmp(a,b)==0){
        BN_WORD_setone(q);
        BN_WORD_setzero(r);
        return 0;
    }
    BN_WORD_copy(a,a_temp);
    while((BN_WORD_cmp(a_temp,b)==1)||(BN_WORD_cmp(a_temp,b)==0)){
        shift_num ++;
        BN_WORD_right_shift_bits(a_temp,temp_result,1);
        BN_WORD_copy(temp_result,a_temp);
    }
    shift_num --;
    BN_WORD_copy(a,a_temp);
    BN_WORD_left_shift_bits(b,b_temp,shift_num);
    BN_WORD_setzero(q);
    for(int i=shift_num;i>=0;i--){
        if((BN_WORD_cmp(a_temp,b_temp)==1)||(BN_WORD_cmp(a_temp,b_temp)==0)){
            BN_WORD_sub(a_temp,b_temp,temp_result);
	    BN_WORD_copy(temp_result,a_temp);
            BN_WORD_left_shift_bits(one,div_temp,i);
            BN_WORD_add(q,div_temp,temp_result);
	    BN_WORD_copy(temp_result,q);
        }
        BN_WORD_right_shift_bits(b_temp,temp_result,1);
        BN_WORD_copy(temp_result,b_temp);
    }
    BN_WORD_copy(a_temp,r);
    BN_WORD_free(one);
    BN_WORD_free(a_temp);
    BN_WORD_free(b_temp);
    BN_WORD_free(temp_result);
    BN_WORD_free(div_temp);
    return 0;
}

__host__ int BN_WORD_add_mod_host(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;
    BN_WORD *q, *a_temp, *b_temp;
    q=BN_WORD_new(dmax);
    a_temp=BN_WORD_new(dmax);
    b_temp=BN_WORD_new(dmax);
    BN_WORD_div(a,n,q,a_temp);
    BN_WORD_div(b,n,q,b_temp);
    BN_WORD_add(a_temp,b_temp,result);
    if(BN_WORD_cmp(a_temp,result)==1){
        BN_WORD_sub(result,n,result);
    }
    return 0;
}

__host__ int BN_WORD_mul_mod_host(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, BN_WORD *result){
    int dmax=a->dmax;
    int bit;
    BN_WORD *a_sub, *b_sub, *temp_result;
    a_sub=BN_WORD_new(dmax);
    b_sub=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    BN_WORD_copy(a,a_sub);
    BN_WORD_copy(b,b_sub);
    while(BN_WORD_cmp(a_sub,n)==1){
        BN_WORD_sub(a_sub,n,a_sub);
    }
    if(BN_WORD_cmp(a_sub,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    while(BN_WORD_cmp(b_sub,n)==1){
        BN_WORD_sub(b_sub,n,b_sub);
    }
    if(BN_WORD_cmp(b_sub,n)==0){
        BN_WORD_setzero(result);
        return 0;
    }
    BN_WORD_setzero(result);
    for(int i=dmax-1;i>=0;i--){
        for(int j=sizeof(BN_PART)*8-1;j>=0;j--){
                bit=get_bit(b_sub->d[i],j);
                BN_WORD_add(result,result,temp_result);
                if((BN_WORD_cmp(temp_result, result)==2)||(BN_WORD_cmp(temp_result,n)==1)||(BN_WORD_cmp(temp_result,n)==0)){
                    BN_WORD_sub(temp_result,n,result);
                }
                else {
                    BN_WORD_copy(temp_result,result);
                }
                if(bit==1){
                    BN_WORD_add(result,a_sub,temp_result);
                    if((BN_WORD_cmp(temp_result, result)==2)||(BN_WORD_cmp(temp_result,n)==1)||(BN_WORD_cmp(temp_result,n)==0)){
                            BN_WORD_sub(temp_result,n,result);
                    }
                    else {
                            BN_WORD_copy(temp_result,result);
                    }
                }
        }
    }
    BN_WORD_free(a_sub);
    BN_WORD_free(b_sub);
    BN_WORD_free(temp_result);
    return 0;
}

/*
something useless

BN_WORD *zero;
BN_WORD *one;

BN_WORD_zero(DMAX);

BN_WORD_setzero(zero);
BN_WORD_setzero(one);

__host__ BN_WORD* BN_WORD_CTX_new(int dmax, int num){
    BN_WORD *a;
    hipMallocManaged((void**)&(a),num*sizeof(BN_WORD));
    for(int i=0;i<num;i++){
        (a+i)->dmax=dmax;
	(a+i)->carry=0;
	hipMallocManaged((void**)&((a+i)->d),dmax*sizeof(BN_PART));
    }
    return a;
}

__host__ void BN_WORD_CTX_free(BN_WORD *a,int num){
    for(int i=0;i<num;i++){
        hipFree((a+i)->d);
    }
    hipFree(a);
}

__device__ BN_WORD* BN_WORD_CTX_new_device(int dmax, int num){
    BN_WORD *a;
    a=(BN_WORD*)malloc(num*sizeof(BN_WORD));
    for(int i=0;i<num;i++){
        (a+i)->dmax=dmax;
        (a+i)->carry=0;
	(a+i)->d=(BN_PART *)malloc(dmax*sizeof(BN_PART));
    }
    return a;
}

__device__ void BN_WORD_CTX_free_device(BN_WORD *a,int num){
    for(int i=0;i<num;i++){
        free((a+i)->d);
    }
    free(a);
}

__host__ __device__ int BN_WORD_CTX_mul_part(const BN_WORD *a, const BN_PART b, BN_PART &u, BN_WORD *v){
    int dmax=a->dmax;
    BN_PART temp_u, temp_v;
    u=0;
    BN_WORD_setzero(v);
    for(int i=0;i<dmax-1;i++){
        BN_PART_mul(a->d[i],b,temp_u,temp_v);
	v->d[i]=v->d[i]+temp_v;
	if(v->d[i]<temp_v){
	    v->d[i+1]=temp_u+1;
	}
	else{
	    v->d[i+1]=temp_u;
	}
    }
    BN_PART_mul(a->d[dmax-1],b,temp_u,temp_v);
    v->d[dmax-1]=v->d[dmax-1]+temp_v;
    if(v->d[dmax-1]<temp_v){
        u=temp_u+1;
    }
    else{
        u=temp_u;
    }
    return 0;
}

__host__ __device__ int BN_WORD_CTX_mul(const BN_WORD *a, const BN_WORD *b, BN_WORD *u, BN_WORD *v, BN_WORD *ctx){
    int dmax=a->dmax;
    BN_WORD *temp_v, *temp_shifted_u, *temp_shifted_v, *one;
    BN_PART temp_u;
    temp_v=ctx;
    temp_shifted_u=ctx+1;
    temp_shifted_v=ctx+2;
    one=ctx+3;
    BN_WORD_setone(one);
    BN_WORD_setzero(u);
    BN_WORD_setzero(v);
    for(int i=0; i<dmax;i++){
        BN_WORD_CTX_mul_part(a,b->d[i],temp_u,temp_v);
	BN_WORD_left_shift(temp_v,temp_shifted_v,i);
	BN_WORD_right_shift(temp_v,temp_shifted_u,(dmax-i));
	temp_shifted_u->d[i]=temp_u;
    	BN_WORD_add(v,temp_shifted_v,v);
	BN_WORD_add(u,temp_shifted_u,u);
	if(BN_WORD_cmp(v,temp_shifted_v)==2){
            BN_WORD_add(u,one,u);		
	}
    }
    return 0;
}

__host__ __device__ void BN_WORD_mul_word_bnulong(BN_WORD *a, BN_PART b,BN_WORD *result, BN_WORD *mid_value1, BN_WORD *mid_value2, 
		BN_WORD *mid_value3, int *return_value, int *mid_return_value){
    if(*(a->carry)!=0){
        *return_value=-1;
	return;
    }
    BN_WORD_setzero(result);
    BN_WORD_setzero(mid_value1);
    BN_WORD_setzero(mid_value2);
    BN_WORD_setzero(mid_value3);
    for(int i=0;i<sizeof(BN_PART)*8;i++){
        if(get_bit(b,i)==1){
	    BN_WORD_left_shift_bits(a,mid_value2,i,mid_return_value);
	    *(mid_value1->carry)=0;
	    *(mid_value3->carry)=0;
	    BN_WORD_add(mid_value1,mid_value2,mid_value3,mid_return_value);
	    *(result->carry)=*(result->carry)+(a->d[*(a->dmax)-1])/((BN_PART)1<<(sizeof(BN_PART)*8-i))+*(mid_value3->carry);
	    for(int j=0;j<*(a->dmax);j++){
	        mid_value1->d[j]=mid_value3->d[j];
	    }
	}
    }
//    BN_WORD_print(mid_value1);
    for(int i=0;i<*(a->dmax);i++){
        result->d[i]=mid_value3->d[i];
    }
    *return_value=0;
    return;
}

__host__ __device__ void BN_WORD_mul(BN_WORD *a, BN_WORD *b, BN_WORD *result_u,BN_WORD *result_v,BN_WORD *mul_word_result,  BN_WORD *mid_value1, 
		BN_WORD *mid_value2, BN_WORD *mid_value3,BN_WORD *mid_value4, BN_WORD *mid_value5, 
		int *return_value,int *add_return_value,int *mid_return_value){
    BN_WORD_setzero(mid_value3);
    BN_WORD_setzero(mid_value4);
    BN_WORD_setzero(result_u);
    for (int i=0;i<*(b->dmax);i++){
        BN_WORD_mul_word_bnulong(a,b->d[i], mul_word_result, mid_value1,mid_value2,mid_value5,add_return_value,mid_return_value);
	*(mid_value3->carry)=0;
	BN_WORD_left_shift(mul_word_result,mid_value4,i,mid_return_value);
	*(mid_value4->carry)=0;
	BN_WORD_setone(mid_value5);
	BN_WORD_add(mid_value3,mid_value4,mid_value5,mid_return_value);
	for (int j=0;j<*(b->dmax);j++){
	    mid_value3->d[j]=mid_value5->d[j];
	}
	BN_WORD_right_shift(mul_word_result,mid_value1,*(a->dmax)-i,mid_return_value);
	*(mid_value1->carry)=0;
	mid_value1->d[i]=*(mul_word_result->carry);
	BN_WORD_setzero(mid_value2);
	if(*(mid_value3->carry)!=0){
	    BN_WORD_setone(mid_value2);
	}
	BN_WORD_add(result_u,mid_value1,mid_value5,mid_return_value);
	for (int j=0;j<*(b->dmax);j++){
            result_u->d[j]=mid_value5->d[j];
        }
	BN_WORD_add(result_u,mid_value2,mid_value5,mid_return_value);
	for (int j=0;j<*(b->dmax);j++){
            result_u->d[j]=mid_value5->d[j];
        }
    }
    for(int i=0;i<*(a->dmax);i++){
        result_v->d[i]=mid_value3->d[i];
    }
    *(result_u->carry)=0;
    *(result_v->carry)=0;
    *(return_value)=0;
    return;
}

__host__ int BN_WORD_mul_half(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    BN_WORD *mid_value;
    BN_WORD *temp_result;
    int dmax=a->dmax;
    if((b->dmax!=dmax)||(result->dmax!=2*dmax)){
        return -1;
    }
    mid_value=BN_WORD_new(dmax*2);
    temp_result=BN_WORD_new(dmax*2);
    BN_WORD_setzero(result);
    for(int i=0;i<dmax;i++){
        BN_WORD_setzero(mid_value);
        BN_WORD_setzero(mid_value);
        for(int j=0;j<dmax;j++){
            mid_value->d[i+j]=(b->d[i])*(a->d[j]);
        }
        BN_WORD_add(result,mid_value,temp_result);
        BN_WORD_copy(temp_result,result);
    }
    BN_WORD_free(mid_value);
    BN_WORD_free(temp_result);
    return 0;
}

 __device__ int BN_WORD_mul_half_device(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    BN_WORD *mid_value;
    BN_WORD *temp_result;
    int dmax=a->dmax;
    if((b->dmax!=dmax)||(result->dmax!=2*dmax)){
        return -1;
    }
    mid_value=BN_WORD_new_device(dmax*2);
    temp_result=BN_WORD_new_device(dmax*2);
    BN_WORD_setzero(result);
    for(int i=0;i<dmax;i++){
	BN_WORD_setzero(mid_value);
	BN_WORD_setzero(mid_value);
	for(int j=0;j<dmax;j++){
	    mid_value->d[i+j]=(b->d[i])*(a->d[j]);
	}
	BN_WORD_add(result,mid_value,temp_result);
	BN_WORD_copy(temp_result,result);
    }
    BN_WORD_free_device(mid_value);
    BN_WORD_free_device(temp_result);
    return 0;
}

__host__ __device__ void BN_WORD_high (const BN_WORD *a, BN_WORD *b){
    b->dmax=a->dmax;
    for(int i=0;i<a->dmax;i++){
        b->d[i]=(a->d[i])/((BN_PART)1<<sizeof(BN_PART)*4);
    }	
}

__host__ __device__ void BN_WORD_low (const BN_WORD *a, BN_WORD *b){
    b->dmax=a->dmax;
    for(int i=0;i<a->dmax;i++){
#ifdef BN_PART_32
        b->d[i]=(a->d[i])&INT_MASK2l;
#endif
#ifdef BN_PART_64
        b->d[i]=(a->d[i])&LONG_MASK2l;
#endif
    }
}

__host__ int BN_WORD_mul(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    int dmax;
    dmax=a->dmax;
    if((b->dmax!=dmax)||(result->dmax!=2*dmax)){
        return -1;
    }
    BN_WORD *a_half, *b_half, *mid_value, *temp_result;
    a_half=BN_WORD_new(dmax);
    b_half=BN_WORD_new(dmax);
    mid_value=BN_WORD_new(dmax*2);
    temp_result=BN_WORD_new(dmax*2);

    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_low(a,a_half);
    BN_WORD_low(b,b_half);

    BN_WORD_mul_half(a_half,b_half,mid_value);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);

    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_high(a,a_half);
    BN_WORD_low(b,b_half);
    BN_WORD_mul_half(a_half,b_half,mid_value);
    BN_WORD_left_shift_bits(mid_value,temp_result,sizeof(BN_PART)*4);
    BN_WORD_copy(temp_result,mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);

    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_low(a,a_half);
    BN_WORD_high(b,b_half);
    BN_WORD_mul_half(a_half,b_half,mid_value);
    BN_WORD_left_shift_bits(mid_value,temp_result,sizeof(BN_PART)*4);
    BN_WORD_copy(temp_result,mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);

    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_high(a,a_half);
    BN_WORD_high(b,b_half);
    BN_WORD_mul_half(a_half,b_half,mid_value);
    BN_WORD_left_shift(mid_value,temp_result,1);
    BN_WORD_copy(temp_result,mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);

    BN_WORD_free(a_half);
    BN_WORD_free(b_half);
    BN_WORD_free(mid_value);
    BN_WORD_free(temp_result);
    return 0;
}

__device__ int BN_WORD_mul_device(const BN_WORD *a, const BN_WORD *b, BN_WORD *result){
    int dmax;
    dmax=a->dmax;
    if((b->dmax!=dmax)||(result->dmax!=2*dmax)){
        return -1;
    }
    BN_WORD *a_half, *b_half, *mid_value, *temp_result;
    a_half=BN_WORD_new_device(dmax);
    b_half=BN_WORD_new_device(dmax);
    mid_value=BN_WORD_new_device(dmax*2);
    temp_result=BN_WORD_new_device(dmax*2);

    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_low(a,a_half);
    BN_WORD_low(b,b_half);

    BN_WORD_mul_half_device(a_half,b_half,mid_value);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);

    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_high(a,a_half);
    BN_WORD_low(b,b_half);
    BN_WORD_mul_half_device(a_half,b_half,mid_value);
    BN_WORD_left_shift_bits(mid_value,temp_result,sizeof(BN_PART)*4);
    BN_WORD_copy(temp_result,mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);



    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_low(a,a_half);
    BN_WORD_high(b,b_half);
    BN_WORD_mul_half_device(a_half,b_half,mid_value);
    BN_WORD_left_shift_bits(mid_value,temp_result,sizeof(BN_PART)*4);
    BN_WORD_copy(temp_result,mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);


    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_high(a,a_half);
    BN_WORD_high(b,b_half);
    BN_WORD_mul_half_device(a_half,b_half,mid_value);
    BN_WORD_left_shift(mid_value,temp_result,1);
    BN_WORD_copy(temp_result,mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);


    BN_WORD_free_device(a_half);
    BN_WORD_free_device(b_half);
    BN_WORD_free_device(mid_value);
    BN_WORD_free_device(temp_result);
    return 0;
}
*/
