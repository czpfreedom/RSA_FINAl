#include "bn_word_operation.h"
#include "bn_openssl.h"
#include "stdlib.h"
#include "iostream"
#include <stdio.h>

#ifndef BN_MASK2L
#define BN_MASK2L (0xffffffffffffffffL)// unsigned long
#endif

#ifndef BN_MASK2l
#define BN_MASK2l (0xffffffffL)        // unsigned long also  but with 32 zeros at the top
#endif




#ifndef get_bit
#define get_bit(a,i)     (a&((BN_ULONG)1<<i))>>i
#endif


BN_WORD* BN_WORD_new(int dmax){
    BN_WORD *a;
    a=new BN_WORD();
    hipMallocManaged((void**)&(a->dmax),sizeof(int));
    hipMallocManaged((void**)&(a->carry),sizeof(int));
    *(a->dmax)=dmax;
    *(a->carry)=0;
    hipMallocManaged((void**)&(a->d),*(a->dmax)*sizeof(BN_ULONG));
    return a;
}

int BN_WORD_free(BN_WORD *a){
    hipFree(a->dmax);
    hipFree(a->carry);
    hipFree(a->d);
    delete(a);
    return 0;
}

__host__ __device__ int BN_WORD_setzero(BN_WORD *a){
    *(a->carry)=0;
    for(int i=0;i<*a->dmax;i++){
        a->d[i]=0;
    }
    return 0;
}

__host__ __device__ int BN_WORD_setone(BN_WORD *a){
    *(a->carry)=0;
    a->d[0]=1;
    for(int i=1;i<*a->dmax;i++){
        a->d[i]=0;
    }
    return 0;
}


/*int BN_WORD_copy(BN_WORD *a,BN_WORD *b){
    *(b->carry)=*(a->carry);
    if(*(a->dmax)==*(b->dmax)){
        ;
    }
    else {
        *(b->dmax)=*(a->dmax);
        hipFree((b->d));
        hipMallocManaged((void **)&(b->d),*(b->dmax)*sizeof(BN_ULONG));
        for(int i=0;i<*(b->dmax);i++){
            b->d[i]=a->d[i];
        }
    }
}
*/

__host__ __device__ void BN_WORD_print(BN_WORD *a){
    printf("dmax:%d\n",*(a->dmax));
    printf("carry:%lx\n",*(a->carry));
    for(int i=*(a->dmax)-1;i>=0;i--){
        printf("%lx,",a->d[i]);
    }
    printf("\n");
}

__host__ __device__ void BN_WORD_cmp(BN_WORD *a, BN_WORD *b, int *return_value){
    *return_value=0;
    if ((*(a->carry)!=0)||(*(b->carry)!=0)){
        *return_value=-1;
        return;
    }
    if(*(a->dmax)!=*(b->dmax)){
        *return_value=-2;
        return ;
    }
    for(int i=*(a->dmax)-1;i>=0;i--){
        if(a->d[i]>b->d[i]){
            *return_value=1;
            return;
        }
        if(a->d[i]<b->d[i]){
            *return_value=2;
            return;
        }
    }
    *return_value=0;
    return ;
}

__host__ __device__ void BN_WORD_left_shift(BN_WORD *a,BN_WORD *b,int words,int *return_value){
    if(*(a->dmax)<words){
        *return_value=-1;
	return;
    }
    if(*(a->dmax)!=*(b->dmax)){
        *return_value=-2;
	return;
    }
    *(b->carry)=*(a->carry);
    *(b->dmax)=*(a->dmax);
    for(int i=*(a->dmax)-1;i>=words;i--){
        b->d[i]=a->d[i-words];
    }
    for(int i=words-1;i>=0;i--){
        b->d[i]=0;
    }
    *return_value=0;
    return ;
}



__host__ __device__ void BN_WORD_left_shift_bits(BN_WORD *a,BN_WORD *b,int bits,int *return_value){
    if(bits>sizeof(BN_ULONG)*8){
        *return_value=-1;
	return;
    }
    if(*(a->dmax)!=*(b->dmax)){
        *return_value=-2;
        return;
    }
    *(b->carry)=*(a->carry);
    *(b->dmax)=*(a->dmax);
    b->d[0]=a->d[0]<<bits;
    for (int i=1;i<*(a->dmax);i++){
        b->d[i]=(a->d[i]<<bits)+(a->d[i-1])/((BN_ULONG)1<<(sizeof(BN_ULONG)*8-bits));
    }
    *return_value=0;
    return;
}



__host__ __device__ void BN_WORD_right_shift(BN_WORD *a,BN_WORD *b,int words, int *return_value){
    if(*(a->dmax)<words){
        *return_value=-1;
        return;
    }
    if(*(a->dmax)!=*(b->dmax)){
        *return_value=-2;
        return;
    }
    for(int i=0;i<*(a->dmax)-words;i++){
        b->d[i]=a->d[i+words];
    }
    for(int i=*(a->dmax)-words;i<*(a->dmax);i++){
        b->d[i]=0;
    }
    *return_value=0;
    return;
}

__host__ __device__ void BN_WORD_right_shift_bits(BN_WORD *a,BN_WORD *b,int bits,int *return_value){
    if(bits>sizeof(BN_ULONG)*8){
        *return_value=-1;
        return;
    }
    if(*(a->dmax)!=*(b->dmax)){
        *return_value=-2;
        return;
    }
    *(b->carry)=*(a->carry);
    *(b->dmax)=*(a->dmax);
    for (int i=0;i<*(a->dmax)-1;i++){
        b->d[i]=(a->d[i])/((BN_ULONG)1<<bits)+((a->d[i+1])<<(sizeof(BN_ULONG)*8-bits));
//	printf("%lx\n",(a->d[i])/(1<<bits));
//	printf("%lx\n",(a->d[i+1])<<(sizeof(BN_ULONG)*8-bits));
    }
    b->d[*(a->dmax)-1]=(a->d[*(a->dmax)-1])/((BN_ULONG)1<<bits);
    *return_value=0;
    return;
}



__host__ __device__ void BN_WORD_add(BN_WORD *a, BN_WORD *b, BN_WORD *result, int *return_value){
    BN_ULONG carry2=0;
    BN_ULONG carry1=0;
    BN_ULONG mid_value;
    if ((*(a->carry)!=0)||(*(b->carry)!=0)){
        *return_value=-1;
	return ;
    }
    if(*(a->dmax)!=*(b->dmax)){
	*return_value=-2;
        return ;
    }
    *(result->dmax)=*(a->dmax);
    for (int i=0;i<*(a->dmax);i++){
	carry2=carry1;
	carry1=0;
        mid_value=(a->d[i]+carry2)&BN_MASK2L;
        if(mid_value<a->d[i]){
            carry1=1;
        }
        mid_value=(mid_value+b->d[i])&BN_MASK2L;
        if(mid_value<b->d[i]){
            carry1=1;
        }
        result->d[i]=mid_value;
    }
    *(result->carry)=carry1;
    *return_value=0;
    return ;
}

__host__ __device__ void BN_WORD_sub(BN_WORD *a, BN_WORD *b, BN_WORD *result, int *cmp_return_value,int *return_value){
    BN_ULONG mid_value1, mid_value;
    BN_ULONG carry1,carry2;
    BN_WORD_cmp(a,b,cmp_return_value);
    if(*(cmp_return_value)==-1){
        *return_value=-1;
        return;
    }
    if(*(cmp_return_value)==-2){
        *return_value=-2;
	return;
    }
    if(*(cmp_return_value)==2){
        *return_value=-3;
	return;
    }
    if(*(cmp_return_value)==0){
	*return_value=0;
        BN_WORD_setzero(result);
	return;
    }
    if(*(cmp_return_value)==1){
        *(result->dmax)=*(a->dmax);
	*(result->carry)=0;
        carry2=0;
	carry1=0;
        for(int i=0;i<*(a->dmax);i++){
	    carry2=carry1;
	    carry1=0;
            mid_value1=(a->d[i]-carry2)&BN_MASK2L;
            if(mid_value1>a->d[i]){
                carry1=1;
            }
            mid_value=(mid_value1-b->d[i])&BN_MASK2L;
            if(mid_value>mid_value1){
                carry1=1;
            }
            result->d[i]=mid_value;
        }
        *return_value=0;
    }
}

__host__ __device__ void BN_WORD_high (BN_WORD *a, BN_WORD *b){
    *(b->dmax)=*(a->dmax);
    *(b->carry)=*(a->dmax);
    for(int i=0;i<*(a->dmax);i++){
        b->d[i]=(a->d[i])/((BN_ULONG)1<<sizeof(BN_ULONG)*4);
    }	
}

__host__ __device__ void BN_WORD_low (BN_WORD *a, BN_WORD *b){
    *(b->dmax)=*(a->dmax);
    *(b->carry)=*(a->dmax);
    for(int i=0;i<*(a->dmax);i++){
        b->d[i]=(a->d[i])&BN_MASK2l;
    }
}

__host__ __device__ void BN_WORD_mul_half(BN_WORD *a, BN_WORD *b, BN_WORD *result, BN_WORD *mid_value1,
	       	BN_WORD *mid_value2,int *add_return_value){
    int dmax=*(a->dmax);
    BN_WORD_setzero(result);
    for(int i=0;i<dmax;i++){
	BN_WORD_setzero(mid_value1);
	BN_WORD_setzero(mid_value2);
	for(int j=0;j<dmax;j++){
	    mid_value1->d[i+j]=(b->d[i])*(a->d[j]);
	}
	BN_WORD_add(result,mid_value1,mid_value2,add_return_value);
	for(int j=0;j<dmax*2;j++){
	    result->d[j]=mid_value2->d[j];
	}
    }
}

__host__ __device__ void BN_WORD_mul(BN_WORD *a, BN_WORD *b, BN_WORD *a_half, BN_WORD *b_half,BN_WORD *result,
		BN_WORD *mid_value1,  BN_WORD *mid_value2, BN_WORD *mid_value3, BN_WORD *temp_result,
		int *add_return_value, int *shift_return_value){
    int dmax=*(a->dmax);
    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value1);
    BN_WORD_setzero(mid_value2);
    BN_WORD_setzero(mid_value3);
    BN_WORD_setzero(temp_result);
    BN_WORD_low(a,a_half);
    BN_WORD_low(b,b_half);
    BN_WORD_mul_half(a_half,b_half,mid_value3,mid_value1,mid_value2,add_return_value);
    BN_WORD_add(result,mid_value3,temp_result,add_return_value);
    for(int i=0;i<dmax*2;i++){
        result->d[i]=temp_result->d[i];
    }


    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value1);
    BN_WORD_setzero(mid_value2);
    BN_WORD_setzero(mid_value3);
    BN_WORD_setzero(temp_result);
    BN_WORD_high(a,a_half);
    BN_WORD_low(b,b_half);
    BN_WORD_mul_half(a_half,b_half,mid_value3,mid_value1,mid_value2,add_return_value);
    BN_WORD_left_shift_bits(mid_value3,temp_result,sizeof(BN_ULONG)*4,shift_return_value);
    for(int i=0;i<dmax*2;i++){
        mid_value3->d[i]=temp_result->d[i];
    }

    BN_WORD_add(result,mid_value3,temp_result,add_return_value);
    for(int i=0;i<dmax*2;i++){
        result->d[i]=temp_result->d[i];
    }

    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value1);
    BN_WORD_setzero(mid_value2);
    BN_WORD_setzero(mid_value3);
    BN_WORD_setzero(temp_result);
    BN_WORD_low(a,a_half);
    BN_WORD_high(b,b_half);
    BN_WORD_mul_half(a_half,b_half,mid_value3,mid_value1,mid_value2,add_return_value);
    BN_WORD_left_shift_bits(mid_value3,temp_result,sizeof(BN_ULONG)*4,shift_return_value);
    for(int i=0;i<dmax*2;i++){
        mid_value3->d[i]=temp_result->d[i];
    }

    BN_WORD_add(result,mid_value3,temp_result,add_return_value);
    for(int i=0;i<dmax*2;i++){
        result->d[i]=temp_result->d[i];
    }

    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value1);
    BN_WORD_setzero(mid_value2);
    BN_WORD_setzero(mid_value3);
    BN_WORD_setzero(temp_result);
    BN_WORD_high(a,a_half);
    BN_WORD_high(b,b_half);
    BN_WORD_mul_half(a_half,b_half,mid_value3,mid_value1,mid_value2,add_return_value);
    BN_WORD_left_shift(mid_value3,temp_result,1,shift_return_value);
    for(int i=0;i<dmax*2;i++){
        mid_value3->d[i]=temp_result->d[i];
    }

    BN_WORD_add(result,mid_value3,temp_result,add_return_value);
    for(int i=0;i<dmax*2;i++){
        result->d[i]=temp_result->d[i];
    }
}

/*

__host__ __device__ void BN_WORD_mul_word_bnulong(BN_WORD *a, BN_ULONG b,BN_WORD *result, BN_WORD *mid_value1, BN_WORD *mid_value2, 
		BN_WORD *mid_value3, int *return_value, int *mid_return_value){
    if(*(a->carry)!=0){
        *return_value=-1;
	return;
    }
    BN_WORD_setzero(result);
    BN_WORD_setzero(mid_value1);
    BN_WORD_setzero(mid_value2);
    BN_WORD_setzero(mid_value3);
    for(int i=0;i<sizeof(BN_ULONG)*8;i++){
        if(get_bit(b,i)==1){
	    BN_WORD_left_shift_bits(a,mid_value2,i,mid_return_value);
	    *(mid_value1->carry)=0;
	    *(mid_value3->carry)=0;
	    BN_WORD_add(mid_value1,mid_value2,mid_value3,mid_return_value);
	    *(result->carry)=*(result->carry)+(a->d[*(a->dmax)-1])/((BN_ULONG)1<<(sizeof(BN_ULONG)*8-i))+*(mid_value3->carry);
	    for(int j=0;j<*(a->dmax);j++){
	        mid_value1->d[j]=mid_value3->d[j];
	    }
	}
    }
//    BN_WORD_print(mid_value1);
    for(int i=0;i<*(a->dmax);i++){
        result->d[i]=mid_value3->d[i];
    }
    *return_value=0;
    return;
}

__host__ __device__ void BN_WORD_mul(BN_WORD *a, BN_WORD *b, BN_WORD *result_u,BN_WORD *result_v,BN_WORD *mul_word_result,  BN_WORD *mid_value1, 
		BN_WORD *mid_value2, BN_WORD *mid_value3,BN_WORD *mid_value4, BN_WORD *mid_value5, 
		int *return_value,int *add_return_value,int *mid_return_value){
    BN_WORD_setzero(mid_value3);
    BN_WORD_setzero(mid_value4);
    BN_WORD_setzero(result_u);
    for (int i=0;i<*(b->dmax);i++){
        BN_WORD_mul_word_bnulong(a,b->d[i], mul_word_result, mid_value1,mid_value2,mid_value5,add_return_value,mid_return_value);
	*(mid_value3->carry)=0;
	BN_WORD_left_shift(mul_word_result,mid_value4,i,mid_return_value);
	*(mid_value4->carry)=0;
	BN_WORD_setone(mid_value5);
	BN_WORD_add(mid_value3,mid_value4,mid_value5,mid_return_value);
	for (int j=0;j<*(b->dmax);j++){
	    mid_value3->d[j]=mid_value5->d[j];
	}
	BN_WORD_right_shift(mul_word_result,mid_value1,*(a->dmax)-i,mid_return_value);
	*(mid_value1->carry)=0;
	mid_value1->d[i]=*(mul_word_result->carry);
	BN_WORD_setzero(mid_value2);
	if(*(mid_value3->carry)!=0){
	    BN_WORD_setone(mid_value2);
	}
	BN_WORD_add(result_u,mid_value1,mid_value5,mid_return_value);
	for (int j=0;j<*(b->dmax);j++){
            result_u->d[j]=mid_value5->d[j];
        }
	BN_WORD_add(result_u,mid_value2,mid_value5,mid_return_value);
	for (int j=0;j<*(b->dmax);j++){
            result_u->d[j]=mid_value5->d[j];
        }
    }
    for(int i=0;i<*(a->dmax);i++){
        result_v->d[i]=mid_value3->d[i];
    }
    *(result_u->carry)=0;
    *(result_v->carry)=0;
    *(return_value)=0;
    return;
}
*/
