#include "bn_word_operation.h"
#include "bn_openssl.h"
#include "stdlib.h"
#include "iostream"
#include <stdio.h>

#ifndef BN_MASK2L
#define BN_MASK2L (0xffffffffffffffffL)// unsigned long
#endif

#ifndef BN_MASK2l
#define BN_MASK2l (0xffffffffL)        // unsigned long also  but with 32 zeros at the top
#endif




#ifndef get_bit
#define get_bit(a,i)     (a&((BN_ULONG)1<<i))>>i
#endif



__host__ BN_WORD* BN_WORD_new(int dmax){
    BN_WORD *a;
    hipMallocManaged((void**)&(a),sizeof(BN_WORD));
    a->dmax=dmax;
    a->carry=0;
    hipMallocManaged((void**)&(a->d),(a->dmax)*sizeof(BN_ULONG));
    return a;
}

__host__ void BN_WORD_free(BN_WORD *a){
    hipFree(a->d);
    hipFree(a);
}

__device__ BN_WORD* BN_WORD_new_device(int dmax){
    BN_WORD *a;
    a=(BN_WORD*)malloc(sizeof(BN_WORD));
    a->dmax=dmax;
    a->carry=0;
    a->d=(BN_ULONG *)malloc((a->dmax)*sizeof(BN_ULONG));
    return a;
}

__device__ void BN_WORD_free_device(BN_WORD *a){
    free(a->d);
    free(a);
}


__host__ __device__ void BN_WORD_setzero(BN_WORD *a){
    a->carry=0;
    for(int i=0;i<a->dmax;i++){
        a->d[i]=0;
    }
}

__host__ __device__ void BN_WORD_setone(BN_WORD *a){
    a->carry=0;
    a->d[0]=1;
    for(int i=1;i<a->dmax;i++){
        a->d[i]=0;
    }
}


__host__ __device__ int BN_WORD_copy(BN_WORD *a,BN_WORD *b){
    if(a->dmax!=b->dmax){
        return -1;
    }
    b->carry=a->carry;
    for(int i=0;i<a->dmax;i++){
        b->d[i]=a->d[i];
    }
    return 0;
}

__host__ __device__ void BN_WORD_print(BN_WORD *a){
    printf("dmax:%d\n",a->dmax);
    printf("carry:%lx\n",a->carry);
    for(int i=(a->dmax)-1;i>=0;i--){
        printf("%lx,",a->d[i]);
    }
    printf("\n");
}

__host__ __device__ int BN_WORD_cmp(BN_WORD *a, BN_WORD *b){
    if (((a->carry)!=0)||((b->carry)!=0)){
        return -2;
    }
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    for(int i=(a->dmax)-1;i>=0;i--){
        if(a->d[i]>b->d[i]){
            return 1;
        }
        if(a->d[i]<b->d[i]){
            return 2;
        }
    }
    return 0;
}

__host__ __device__ int BN_WORD_left_shift(BN_WORD *a,BN_WORD *b,int words){
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    if((a->dmax)<words){
        return -3;
    }
    b->carry=a->carry;
    for(int i=(a->dmax)-1;i>=words;i--){
        b->d[i]=a->d[i-words];
    }
    for(int i=words-1;i>=0;i--){
        b->d[i]=0;
    }
    return 0;
}


__host__ __device__ int BN_WORD_left_shift_bits(BN_WORD *a,BN_WORD *b,int bits){
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    if(bits>sizeof(BN_ULONG)*8){
        return -3;
    }
    b->carry=a->carry;
    b->d[0]=a->d[0]<<bits;
    for (int i=1;i<a->dmax;i++){
        b->d[i]=(a->d[i]<<bits)+(a->d[i-1])/((BN_ULONG)1<<(sizeof(BN_ULONG)*8-bits));
    }
    return 0;
}


__host__ __device__ int BN_WORD_right_shift(BN_WORD *a,BN_WORD *b,int words){
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    if((a->dmax)<words){
        return -3;
    }
    b->carry=a->carry;
    for(int i=0;i<a->dmax-words;i++){
        b->d[i]=a->d[i+words];
    }
    for(int i=a->dmax-words;i<a->dmax;i++){
        b->d[i]=0;
    }
    return 0;
}



__host__ __device__ int BN_WORD_right_shift_bits(BN_WORD *a,BN_WORD *b,int bits){
    if((a->dmax)!=(b->dmax)){
        return -1;
    }
    if(bits>sizeof(BN_ULONG)*8){
        return -3;
    }
    b->carry=a->carry;
    for (int i=0;i<a->dmax-1;i++){
        b->d[i]=(a->d[i])/((BN_ULONG)1<<bits)+((a->d[i+1])<<(sizeof(BN_ULONG)*8-bits));
    }
    b->d[a->dmax-1]=(a->d[a->dmax-1])/((BN_ULONG)1<<bits);
    return 0;
}



__host__ __device__ int BN_WORD_add(BN_WORD *a, BN_WORD *b, BN_WORD *result){
    BN_ULONG carry2=0;
    BN_ULONG carry1=0;
    BN_ULONG mid_value;
    if (((a->carry)!=0)||((b->carry)!=0)){
        return -2;
    }
    if((a->dmax!=b->dmax)||(a->dmax!=result->dmax)){
        return -1;
    }
    for (int i=0;i<a->dmax;i++){
        carry2=carry1;
        carry1=0;
        mid_value=(a->d[i]+carry2)&BN_MASK2L;
        if(mid_value<a->d[i]){
            carry1=1;
        }
        mid_value=(mid_value+b->d[i])&BN_MASK2L;
        if(mid_value<b->d[i]){
            carry1=1;
        }
        result->d[i]=mid_value;
    }
    result->carry=carry1;
    return 0;
}


__host__ __device__ int BN_WORD_sub(BN_WORD *a, BN_WORD *b, BN_WORD *result){
    BN_ULONG mid_value1, mid_value;
    BN_ULONG carry1,carry2;
    int cmp=BN_WORD_cmp(a,b);
    if(cmp==-1){
        return -1;
    }
    if(cmp==-2){
        return -2;
    }
    if(cmp==2){
        return -4;
    }
    if(cmp==0){
        BN_WORD_setzero(result);
	return 0;
    }
    if(cmp==1){
        result->dmax=a->dmax;
        result->carry=0;
        carry2=0;
        carry1=0;
        for(int i=0;i<a->dmax;i++){
            carry2=carry1;
            carry1=0;
            mid_value1=(a->d[i]-carry2)&BN_MASK2L;
            if(mid_value1>a->d[i]){
                carry1=1;
            }
            mid_value=(mid_value1-b->d[i])&BN_MASK2L;
            if(mid_value>mid_value1){
                carry1=1;
            }
            result->d[i]=mid_value;
        }
	return 0;
    }
    return 0;
}




__host__ __device__ void BN_WORD_high (BN_WORD *a, BN_WORD *b){
    b->dmax=a->dmax;
    b->carry=a->carry;
    for(int i=0;i<a->dmax;i++){
        b->d[i]=(a->d[i])/((BN_ULONG)1<<sizeof(BN_ULONG)*4);
    }	
}

__host__ __device__ void BN_WORD_low (BN_WORD *a, BN_WORD *b){
    b->dmax=a->dmax;
    b->carry=a->dmax;
    for(int i=0;i<a->dmax;i++){
        b->d[i]=(a->d[i])&BN_MASK2l;
    }
}

 __device__ int BN_WORD_mul_half(BN_WORD *a, BN_WORD *b, BN_WORD *result){
    BN_WORD *mid_value;
    BN_WORD *temp_result;
    int dmax=a->dmax;
    if((b->dmax!=dmax)||(result->dmax!=2*dmax)){
        return -1;
    }
    mid_value=BN_WORD_new_device(dmax*2);
    temp_result=BN_WORD_new_device(dmax*2);
    BN_WORD_setzero(result);
    for(int i=0;i<dmax;i++){
	BN_WORD_setzero(mid_value);
	BN_WORD_setzero(mid_value);
	for(int j=0;j<dmax;j++){
	    mid_value->d[i+j]=(b->d[i])*(a->d[j]);
	}
	BN_WORD_add(result,mid_value,temp_result);
	BN_WORD_copy(temp_result,result);
    }
    BN_WORD_free_device(mid_value);
    BN_WORD_free_device(temp_result);
    return 0;
}

__device__ int BN_WORD_mul(BN_WORD *a, BN_WORD *b, BN_WORD *result){
    int dmax;
    dmax=a->dmax;
    if((b->dmax!=dmax)||(result->dmax!=2*dmax)){
        return -1;
    }
    if((a->carry!=0)||(b->carry!=0)){
        return -2;
    }
    BN_WORD *a_half, *b_half, *mid_value, *temp_result;
    a_half=BN_WORD_new_device(dmax);
    b_half=BN_WORD_new_device(dmax);
    mid_value=BN_WORD_new_device(dmax*2);
    temp_result=BN_WORD_new_device(dmax*2);

    result->carry=0;

    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_low(a,a_half);
    BN_WORD_low(b,b_half);

    BN_WORD_mul_half(a_half,b_half,mid_value);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);

    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_high(a,a_half);
    BN_WORD_low(b,b_half);
    BN_WORD_mul_half(a_half,b_half,mid_value);
    BN_WORD_left_shift_bits(mid_value,temp_result,sizeof(BN_ULONG)*4);
    BN_WORD_copy(temp_result,mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);



    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_low(a,a_half);
    BN_WORD_high(b,b_half);
    BN_WORD_mul_half(a_half,b_half,mid_value);
    BN_WORD_left_shift_bits(mid_value,temp_result,sizeof(BN_ULONG)*4);
    BN_WORD_copy(temp_result,mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);


    BN_WORD_setzero(a_half);
    BN_WORD_setzero(b_half);
    BN_WORD_setzero(mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_high(a,a_half);
    BN_WORD_high(b,b_half);
    BN_WORD_mul_half(a_half,b_half,mid_value);
    BN_WORD_left_shift(mid_value,temp_result,1);
    BN_WORD_copy(temp_result,mid_value);
    BN_WORD_setzero(temp_result);
    BN_WORD_add(result,mid_value,temp_result);
    BN_WORD_copy(temp_result,result);


    BN_WORD_free_device(a_half);
    BN_WORD_free_device(b_half);
    BN_WORD_free_device(mid_value);
    BN_WORD_free_device(temp_result);
    return 0;
}




/*

__host__ __device__ void BN_WORD_mul_word_bnulong(BN_WORD *a, BN_ULONG b,BN_WORD *result, BN_WORD *mid_value1, BN_WORD *mid_value2, 
		BN_WORD *mid_value3, int *return_value, int *mid_return_value){
    if(*(a->carry)!=0){
        *return_value=-1;
	return;
    }
    BN_WORD_setzero(result);
    BN_WORD_setzero(mid_value1);
    BN_WORD_setzero(mid_value2);
    BN_WORD_setzero(mid_value3);
    for(int i=0;i<sizeof(BN_ULONG)*8;i++){
        if(get_bit(b,i)==1){
	    BN_WORD_left_shift_bits(a,mid_value2,i,mid_return_value);
	    *(mid_value1->carry)=0;
	    *(mid_value3->carry)=0;
	    BN_WORD_add(mid_value1,mid_value2,mid_value3,mid_return_value);
	    *(result->carry)=*(result->carry)+(a->d[*(a->dmax)-1])/((BN_ULONG)1<<(sizeof(BN_ULONG)*8-i))+*(mid_value3->carry);
	    for(int j=0;j<*(a->dmax);j++){
	        mid_value1->d[j]=mid_value3->d[j];
	    }
	}
    }
//    BN_WORD_print(mid_value1);
    for(int i=0;i<*(a->dmax);i++){
        result->d[i]=mid_value3->d[i];
    }
    *return_value=0;
    return;
}

__host__ __device__ void BN_WORD_mul(BN_WORD *a, BN_WORD *b, BN_WORD *result_u,BN_WORD *result_v,BN_WORD *mul_word_result,  BN_WORD *mid_value1, 
		BN_WORD *mid_value2, BN_WORD *mid_value3,BN_WORD *mid_value4, BN_WORD *mid_value5, 
		int *return_value,int *add_return_value,int *mid_return_value){
    BN_WORD_setzero(mid_value3);
    BN_WORD_setzero(mid_value4);
    BN_WORD_setzero(result_u);
    for (int i=0;i<*(b->dmax);i++){
        BN_WORD_mul_word_bnulong(a,b->d[i], mul_word_result, mid_value1,mid_value2,mid_value5,add_return_value,mid_return_value);
	*(mid_value3->carry)=0;
	BN_WORD_left_shift(mul_word_result,mid_value4,i,mid_return_value);
	*(mid_value4->carry)=0;
	BN_WORD_setone(mid_value5);
	BN_WORD_add(mid_value3,mid_value4,mid_value5,mid_return_value);
	for (int j=0;j<*(b->dmax);j++){
	    mid_value3->d[j]=mid_value5->d[j];
	}
	BN_WORD_right_shift(mul_word_result,mid_value1,*(a->dmax)-i,mid_return_value);
	*(mid_value1->carry)=0;
	mid_value1->d[i]=*(mul_word_result->carry);
	BN_WORD_setzero(mid_value2);
	if(*(mid_value3->carry)!=0){
	    BN_WORD_setone(mid_value2);
	}
	BN_WORD_add(result_u,mid_value1,mid_value5,mid_return_value);
	for (int j=0;j<*(b->dmax);j++){
            result_u->d[j]=mid_value5->d[j];
        }
	BN_WORD_add(result_u,mid_value2,mid_value5,mid_return_value);
	for (int j=0;j<*(b->dmax);j++){
            result_u->d[j]=mid_value5->d[j];
        }
    }
    for(int i=0;i<*(a->dmax);i++){
        result_v->d[i]=mid_value3->d[i];
    }
    *(result_u->carry)=0;
    *(result_v->carry)=0;
    *(return_value)=0;
    return;
}
*/
