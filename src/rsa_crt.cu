#include "hip/hip_runtime.h"
#include "rsa_crt.h"
#include "rsa_final.h"
#include "stdlib.h"
#include "string.h"

namespace namespace_rsa_final{

CRT_N :: CRT_N(){

}

CRT_N :: CRT_N (RSA_N rsa_n){
    m_rsa_n=rsa_n;
    m_zero.setzero();
    m_one.setone();
    m_R.setR();
    BN_PART_mod_inverse(m_rsa_n.m_n.m_data[0], 0, m_n0_inverse);
    m_n0_inverse=0-m_n0_inverse;
//n_neg
    struct timeval tv;
    gettimeofday(&tv,NULL);
    m_time_stamp = * new Time_Stamp(tv);
    m_time_system = * new Time_System();
    log_create();
}

CRT_N :: CRT_N (CRT_N &crt_n){
    m_rsa_n = crt_n.m_rsa_n;
    m_zero = crt_n.m_zero;
    m_one = crt_n.m_one;
    m_R =crt_n.m_R;
    m_n0_inverse = crt_n.m_n0_inverse;

    m_log_file = crt_n.m_log_file;
    m_time_stamp = crt_n.m_time_stamp;
    m_time_system = crt_n.m_time_system;
}

CRT_N& CRT_N :: operator= (CRT_N &crt_n){

    m_rsa_n = crt_n.m_rsa_n;
    m_zero = crt_n.m_zero;
    m_one = crt_n.m_one;
    m_R =crt_n.m_R;
    m_n0_inverse = crt_n.m_n0_inverse;

    m_log_file = crt_n.m_log_file;
    m_time_stamp = crt_n.m_time_stamp;
    m_time_system = crt_n.m_time_system;

    return * this;
}


CRT_N :: ~CRT_N (){
    log_quit();
}

int CRT_N :: CRT_MOD_MUL(BN_WORD a, BN_WORD b, BN_WORD &result){
    BN_WORD aR, bR;
    BN_WORD n=m_rsa_n.m_n;
    aR=(a*m_R)%n;
    bR=(b*m_R)%n;
   
    BN_PART *bp_a;
    BN_PART *bp_b;
    BN_PART *bp_n;
    BN_PART *bp_result;
    hipMallocManaged((void**)&(bp_a),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_b),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_n),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_result),WARP_SIZE*sizeof(BN_PART));


    for(int i=0;i<WARP_SIZE;i++){
        bp_a[i]=0;
        bp_b[i]=0;
        bp_n[i]=0;
        bp_result[i]=0;
    }

    for(int i=0;i<aR.m_top;i++){
	bp_a[i]=aR.m_data[i];
    }
    
    for(int i=0;i<bR.m_top;i++){
	bp_b[i]=bR.m_data[i];
    }
    for(int i=0;i<n.m_top;i++){
	bp_n[i]=n.m_data[i];
    }
    
    GPU_WORD_mul_mod<<<1,WARP_SIZE>>>(bp_a,bp_b,bp_n,m_n0_inverse,bp_result);
    hipDeviceSynchronize();

    result.setzero(); 
    for(int i=0;i<WARP_SIZE;i++){
	result.m_data[i]=bp_result[i];
    }
    result.m_top=32;
    
    hipFree(bp_result);
    return 1;
}

int CRT_N :: log_create(){

    char file_name[200];
    snprintf(file_name,sizeof(file_name), "%s%s.log",RSA_FINAL_LOG, m_time_stamp.m_abbr);
    m_log_file = fopen(file_name, "a+");
    return 1;

}

int CRT_N :: log_info(LOG_TYPE log_type){
    return 1;
}

int CRT_N :: time_info(LOG_TYPE log_type, TIME_TYPE time_type){
    return 1;
}

int CRT_N :: log_quit(){

    fclose(m_log_file);
    return 1; 
    
}

__device__ int  GPU_WORD_parallel_Mon(BN_PART *A, BN_PART *B, BN_PART *N, BN_PART n0_inverse, BN_PART *M, BN_PART *U, BN_PART *V, BN_PART *C, BN_PART *result, int thread_id){
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;
    p_a=A[thread_id];
    p_n=N[thread_id];
    p_u=0;
    p_v=0;
    M[thread_id]=0;
    C[thread_id]=0;

    for(int i=0;i<WARP_SIZE;i++){
        p_b=B[i];
        BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mul_lo(p_v,n0_inverse,M[thread_id]);
        __syncthreads();
        p_m=M[0];
        BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        V[thread_id]=p_v;
        __syncthreads();
        p_v=V[int_mod(thread_id+1,WARP_SIZE)];
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
    }
    C[thread_id]=p_u;
    U[thread_id]=p_u;
    __syncthreads();//
    while(BN_PART_any(U,WARP_SIZE)==0){
        p_u=U[int_mod(thread_id-1,WARP_SIZE)];
        if(thread_id==0){
            p_u=0;
        }
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        C[thread_id]=C[thread_id]+p_u;
        U[thread_id]=p_u;
        __syncthreads();
    }
    result[thread_id]=p_v;

    return 1;	
}


__device__ int GPU_WORD_delete_carry(BN_PART *result, BN_PART *N, BN_PART c){
    GPU_WORD gw_result, gw_n;
    gw_result.setzero();
    gw_n.setzero();
    for(int i=0;i<WARP_SIZE;i++){
        gw_result.m_data[i]=result[i];
	gw_n.m_data[i]=N[i];
    }
    gw_result.m_data[WARP_SIZE]=c;
    gw_result.m_top=WARP_SIZE+1;
    gw_n.m_top=WARP_SIZE;
    gw_result.check_top();
    gw_result=gw_result%gw_n;
    for(int i=0;i<WARP_SIZE;i++){
        result[i]=gw_result.m_data[i];
    }
    return 1;
}



__global__ void  GPU_WORD_mul_mod(BN_PART *A, BN_PART *B, BN_PART *N, BN_PART n0_inverse, BN_PART *result){
   int j=threadIdx.x;
   
   __shared__ BN_PART M[WARP_SIZE]; 
   __shared__ BN_PART U[WARP_SIZE]; 
   __shared__ BN_PART V[WARP_SIZE]; 
   __shared__ BN_PART C[WARP_SIZE]; 

   BN_PART c;
   
   __shared__ BN_PART ONE[WARP_SIZE];
   ONE[j]=0;
   ONE[0]=1;

   GPU_WORD_parallel_Mon(A,B,N,n0_inverse,M,U,V,C,result,j);

   c=C[WARP_SIZE-1];

   if(j==0){
       GPU_WORD_delete_carry(result, N, c); 
   }
   __syncthreads();

   GPU_WORD_parallel_Mon(result,ONE,N,n0_inverse,M,U,V,C,result,j);

   c=C[WARP_SIZE-1];

   if(j==0){
       GPU_WORD_delete_carry(result, N, c); 
   }
   __syncthreads();

}

}
