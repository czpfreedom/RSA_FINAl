#include "rsa_crt.h"
#include "rsa_final.h"
#include "stdlib.h"
#include "string.h"

namespace namespace_rsa_final{

CRT_N :: CRT_N(){
    log_info(CRT_CREATE_LOG);

}

CRT_N :: CRT_N (RSA_N rsa_n){
    m_rsa_n=rsa_n;
    m_zero.setzero();
    m_one.setone();
    int R_top=rsa_n.m_n.m_top+1;
    m_R.setR(R_top);
    m_R=m_R%m_rsa_n.m_n;
    BN_PART_mod_inverse(m_rsa_n.m_n.m_data[0], 0, m_n0_inverse);
    m_n0_inverse=0-m_n0_inverse;

    log_info(CRT_CREATE_LOG);

}

CRT_N :: CRT_N (CRT_N &crt_n){
    m_rsa_n = crt_n.m_rsa_n;
    m_zero = crt_n.m_zero;
    m_one = crt_n.m_one;
    m_R =crt_n.m_R;
    m_n0_inverse = crt_n.m_n0_inverse;

    memcpy(m_log_file_name,crt_n.m_log_file_name,LOG_FILE_NAME_LENGTH);
}

CRT_N& CRT_N :: operator= (CRT_N &crt_n){

    m_rsa_n = crt_n.m_rsa_n;
    m_zero = crt_n.m_zero;
    m_one = crt_n.m_one;
    m_R =crt_n.m_R;
    m_n0_inverse = crt_n.m_n0_inverse;

    memcpy(m_log_file_name,crt_n.m_log_file_name,LOG_FILE_NAME_LENGTH);
    return * this;
}

CRT_N :: ~CRT_N (){
}

int CRT_N :: CRT_MOD_MUL(BN_WORD a, BN_WORD b, BN_WORD &result){
    BN_WORD aR, bR;
    aR=(a*m_R)%m_rsa_n.m_n;
    bR=(b*m_R)%m_rsa_n.m_n;

    int n_top=m_rsa_n.m_n.m_top;
   
    BN_PART *bp_a, *bp_b, *bp_n, *bp_result;
    hipMallocManaged((void**)&(bp_a),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_b),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_n),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_result),WARP_SIZE*sizeof(BN_PART));

    memset(bp_a,0,WARP_SIZE);
    memset(bp_b,0,WARP_SIZE);
    memset(bp_n,0,WARP_SIZE);
    memset(bp_result,0,WARP_SIZE);

    memcpy(bp_a,aR.m_data,aR.m_top*sizeof(BN_PART));
    memcpy(bp_b,bR.m_data,bR.m_top*sizeof(BN_PART));
    memcpy(bp_n,m_rsa_n.m_n.m_data,m_rsa_n.m_n.m_top*sizeof(BN_PART));
    
    GPU_WORD_mod_mul<<<1,WARP_SIZE>>>(bp_a,bp_b,bp_n,m_n0_inverse,bp_result, n_top);
    hipDeviceSynchronize();

    result.setzero(); 
    memcpy(result.m_data,bp_result,n_top*sizeof(BN_PART));
    result.check_top();

    log_info(CRT_MOD_MUL_LOG,a,b,result);

    hipFree(bp_a);
    hipFree(bp_b);
    hipFree(bp_n);
    hipFree(bp_result);
    return 1;
}

int CRT_N :: CRT_MOD_EXP(BN_WORD a, BN_WORD e, BN_WORD &result){
    BN_WORD aR;
    aR=(a*m_R)%m_rsa_n.m_n;

    int n_top=m_rsa_n.m_n.m_top;
   
    BN_PART *bp_a,*bp_e,*bp_r,*bp_n,*bp_result;
    int E_bits=e.m_top*sizeof(BN_PART)*8;

    hipMallocManaged((void**)&(bp_a),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_e),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_r),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_n),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_result),WARP_SIZE*sizeof(BN_PART));

    memset(bp_a,0,WARP_SIZE);
    memset(bp_e,0,WARP_SIZE);
    memset(bp_n,0,WARP_SIZE);
    memset(bp_r,0,WARP_SIZE);
    memset(bp_result,0,WARP_SIZE);
    
    memcpy(bp_a,aR.m_data,aR.m_top*sizeof(BN_PART));
    memcpy(bp_e,e.m_data,e.m_top*sizeof(BN_PART));
    memcpy(bp_n,m_rsa_n.m_n.m_data,m_rsa_n.m_n.m_top*sizeof(BN_PART));
    memcpy(bp_r,m_R.m_data,m_R.m_top*sizeof(BN_PART));

    GPU_WORD_mod_exp<<<1,WARP_SIZE*2>>>(bp_a,bp_e,E_bits,bp_r,bp_n,m_n0_inverse,bp_result, n_top);
    hipDeviceSynchronize();

    result.setzero(); 
    memcpy(result.m_data,bp_result,n_top*sizeof(BN_PART));
    result.check_top();

    log_info(CRT_MOD_EXP_LOG,a,e,result);
    
    hipFree(bp_a);
    hipFree(bp_e);
    hipFree(bp_n);
    hipFree(bp_r);
    hipFree(bp_result);
    return 1;
}

int CRT_N :: CRT_MOD_EXP_ARRAY(BN_WORD_ARRAY a, BN_WORD e, BN_WORD_ARRAY &result){
    int bn_word_num=a.m_bn_word_num;
    BN_WORD_ARRAY aR(bn_word_num);
    for(int i=0;i<bn_word_num;i++){
        aR.m_bn_word[i]=(a.m_bn_word[i]*m_R)%m_rsa_n.m_n;    
    }

    BN_PART *bp_a,*bp_e,*bp_r,*bp_n,*bp_result;
    int E_bits=e.m_top*sizeof(BN_PART)*8;
    int n_top=m_rsa_n.m_n.m_top;

    hipMallocManaged((void**)&(bp_a),bn_word_num*WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_e),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_r),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_n),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_result),bn_word_num*WARP_SIZE*sizeof(BN_PART));

    memset(bp_a,0,bn_word_num*WARP_SIZE);
    memset(bp_e,0,WARP_SIZE);
    memset(bp_n,0,WARP_SIZE);
    memset(bp_r,0,WARP_SIZE);
    memset(bp_result,0,bn_word_num*WARP_SIZE);

    for(int i=0;i<bn_word_num;i++){
        memcpy(bp_a+i*WARP_SIZE,aR.m_bn_word[i].m_data,aR.m_bn_word[i].m_top*sizeof(BN_PART));
    }
    memcpy(bp_e,e.m_data,e.m_top*sizeof(BN_PART));
    memcpy(bp_n,m_rsa_n.m_n.m_data,m_rsa_n.m_n.m_top*sizeof(BN_PART));
    memcpy(bp_r,m_R.m_data,m_R.m_top*sizeof(BN_PART));

    GPU_WORD_ARRAY_mod_exp<<<bn_word_num,WARP_SIZE*2>>>(bp_a,bp_e,E_bits,bp_r,bp_n,m_n0_inverse,bp_result,n_top);
    hipDeviceSynchronize();

    for(int i=0;i<bn_word_num;i++){
        result.m_bn_word[i].setzero();    
	memcpy(result.m_bn_word[i].m_data,bp_result+i*WARP_SIZE,n_top*sizeof(BN_PART));
    	result.m_bn_word[i].check_top();
    }

    log_info(CRT_MOD_EXP_ARRAY_LOG,a,e,result);

    hipFree(bp_a);
    hipFree(bp_e);
    hipFree(bp_n);
    hipFree(bp_r);
    hipFree(bp_result);
    return 1;
}

}
