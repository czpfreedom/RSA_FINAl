#include "hip/hip_runtime.h"
#include "rsa_crt.h"
#include "stdio.h"

__global__ void BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, const BN_WORD *one, const BN_PART n0_inverse, BN_WORD *result){
    int j=threadIdx.x+blockIdx.x*blockDim.x;
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;
    int dmax=a->dmax;

    __shared__  BN_PART M[SHARED_SIZE];
    __shared__  BN_PART C[SHARED_SIZE];
    __shared__  BN_PART U[SHARED_SIZE];
    __shared__  BN_PART V[SHARED_SIZE];
    __shared__  BN_PART A[SHARED_SIZE];
    __shared__  BN_PART B[SHARED_SIZE];

    if(j==0){
	for(int i=0; i<dmax;i++){
	    A[i]=a->d[i];
	    B[i]=b->d[i];
	}
    }

    // Montgomery 
    p_a=A[j];
    p_n=n->d[j];
    p_u=0;
    p_v=0;
    M[j]=0;
    C[j]=0;

    for(int i=0;i<dmax;i++){
        p_b=B[i];
        BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mul_lo(p_v,n0_inverse,M[j]);
        __syncthreads();
        p_m=M[0];
        BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
	V[j]=p_v;
        __syncthreads();
	p_v=V[int_mod(j+1,dmax)];
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
    }
    printf("p_v[%x]:%lx\n",j,p_v);
    C[j]=p_u;
    U[j]=p_u;
    __syncthreads();//
    while(BN_PART_any(U,dmax)==0){
	U[int_mod(j-1,dmax)]=p_u;
        if(j==0){
            p_u=0;
        }
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
	C[j]=C[j]+p_u;
	U[j]=p_u;
        __syncthreads();
    }
    result->d[j]=p_v;
    C[j]=C[dmax-1];
    if(j==0){
        while(C[j]!=0){
	    BN_WORD_mod_device(result,n,result);
	    C[j]=C[j]-1;
    	    BN_WORD_sub(result,n,result);
    	}
	BN_WORD_mod_device(result,n,result);
    }

// Montgomery Reduce

    if(j==0){
        for(int i=0; i<dmax;i++){
            A[i]=result->d[i];
            B[i]=one->d[i];
        }
    }

    p_a=A[j];
    p_n=n->d[j];
    p_u=0;
    p_v=0;
    M[j]=0;
    C[j]=0;

    for(int i=0;i<dmax;i++){
        p_b=B[i];
        BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mul_lo(p_v,n0_inverse,M[j]);
        __syncthreads();
        p_m=M[0];
        BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
	V[j]=p_v;
        __syncthreads();
	p_v=V[int_mod(j+1,dmax)];
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
    }
    C[j]=p_u;
    U[j]=p_u;
    __syncthreads();//
    while(BN_PART_any(U,dmax)==0){
	U[int_mod(j-1,dmax)]=p_u;
        if(j==0){
            p_u=0;
        }
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
	C[j]=C[j]+p_u;
	U[j]=p_u;
        __syncthreads();
    }
    result->d[j]=p_v;
    C[j]=C[dmax-1];
    if(j==0){
        while(C[j]!=0){
	    BN_WORD_mod_device(result,n,result);
	    C[j]=C[j]-1;
    	    BN_WORD_sub(result,n,result);
    	}
	BN_WORD_mod_device(result,n,result);
    }

}
/*
__global__ void BN_WORD_parallel_mont_mul(const BN_WORD *square_1, const BN_WORD *square_2, const BN_WORD *result2, const BN_WORD *e, const BN_WORD *n, const BN_WORD *one, const BN_PART n0_inverse, BN_WORD *result){
    int dmax=square_1->dmax;
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x;
    int j= thread_id%dmax;
    BN_PART e_i;
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;

    __shared__  BN_PART M1[SHARED_SIZE];
    __shared__  BN_PART C1[SHARED_SIZE];
    __shared__  BN_PART U1[SHARED_SIZE];
    __shared__  BN_PART V1[SHARED_SIZE];
    __shared__  BN_PART M2[SHARED_SIZE];
    __shared__  BN_PART C2[SHARED_SIZE];
    __shared__  BN_PART U2[SHARED_SIZE];
    __shared__  BN_PART V2[SHARED_SIZE];

    for(int i=0; i<dmax*sizeof(BN_PART)*8;i++){
        e_i = bn_word_get_bit(e,i);
	if(thread_id<dmax){
	
	else {
	
	}

    }
}
*/


CRT_N ::CRT_N (RSA_N *rsa_n){

    m_rsa_n = rsa_n;
    int dmax = m_rsa_n->n->dmax;
    m_zero=BN_WORD_new(dmax);
    m_one=BN_WORD_new(dmax);
    m_R=BN_WORD_new(dmax);
    BN_WORD_setzero(m_zero);
    BN_WORD_setone(m_one);
    BN_WORD_sub(m_zero,m_rsa_n->n,m_R);

    BN_PART_inverse(m_rsa_n->n->d[0], 0, m_n0_inverse);

    printf("m_R\n");
    BN_WORD_print(m_R);
}

CRT_N :: ~CRT_N (){
    BN_WORD_free(m_zero);
    BN_WORD_free(m_R);

}

int CRT_N :: CRT_MUL_MOD(BN_WORD *a, BN_WORD *b, BN_WORD *result){
    int dmax = a->dmax;
    BN_WORD *a_pro, *b_pro, *temp_result;
    
    
    a_pro=BN_WORD_new(dmax);
    b_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    BN_WORD_mod(a,m_rsa_n->n,a_pro);
    BN_WORD_mod(b,m_rsa_n->n,b_pro);
    
    
    BN_WORD_mul_mod(a_pro,m_R,m_rsa_n->n,temp_result);
    BN_WORD_copy(temp_result,a_pro);
    BN_WORD_mul_mod(b_pro,m_R,m_rsa_n->n,temp_result);
    BN_WORD_copy(temp_result,b_pro);

    printf("a_pro:\n");
    BN_WORD_print(a_pro);
    printf("b_pro:\n");
    BN_WORD_print(b_pro);

    BN_WORD_parallel_mont_mul<<<1,dmax>>>(a_pro, b_pro, m_rsa_n->n, m_one, m_n0_inverse, result);
    hipDeviceSynchronize();
    return 0;
}

int CRT_N :: CRT_MUL_EXP(BN_WORD *a, BN_WORD *e, BN_WORD *result){
	/*
    int dmax = a->dmax;
    BN_WORD *square_1, *square_2, *result_2, *temp_result;
    square_1=BN_WORD_new(dmax);
    square_2=BN_WORD_new(dmax);
    result_2=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);

    BN_WORD_mod(a,m_rsa_n.m_n,square_1);
    BN_WORD_mul_mod(square_1,m_R,m_rsa_n.m_n,temp_result);
    BN_WORD_copy(temp_result,square_1);
    BN_WORD_copy(square_1,square_2);
    BN_WORD_copy(m_R,result_2);

    BN_WORD_parallel_mont_mul<<<1,dmax*2>>>(square_1, square_2, result_2, e, m_rsa_n.m_n , m_one, m_n0_inverse, result);
    hipDeviceSynchronize();
    */
    return 0;

}
