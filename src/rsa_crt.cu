#include "hip/hip_runtime.h"
#include "rsa_crt.h"
#include "rsa_final.h"
#include "stdlib.h"
#include "string.h"

namespace namespace_rsa_final{

CRT_N :: CRT_N(){

}

CRT_N :: CRT_N (RSA_N rsa_n){
    m_rsa_n=rsa_n;
    m_zero.setzero();
    m_one.setone();
    int R_top=rsa_n.m_n.m_top+1;
    m_R.setR(R_top);
    m_R=m_R%m_rsa_n.m_n;
    BN_PART_mod_inverse(m_rsa_n.m_n.m_data[0], 0, m_n0_inverse);
    m_n0_inverse=0-m_n0_inverse;
//n_neg
    struct timeval tv;
    gettimeofday(&tv,NULL);
    m_time_stamp = * new Time_Stamp(tv);
    m_time_system = * new Time_System();
    log_create();
}

CRT_N :: CRT_N (CRT_N &crt_n){
    m_rsa_n = crt_n.m_rsa_n;
    m_zero = crt_n.m_zero;
    m_one = crt_n.m_one;
    m_R =crt_n.m_R;
    m_n0_inverse = crt_n.m_n0_inverse;

    m_log_file = crt_n.m_log_file;
    m_time_stamp = crt_n.m_time_stamp;
    m_time_system = crt_n.m_time_system;
}

CRT_N& CRT_N :: operator= (CRT_N &crt_n){

    m_rsa_n = crt_n.m_rsa_n;
    m_zero = crt_n.m_zero;
    m_one = crt_n.m_one;
    m_R =crt_n.m_R;
    m_n0_inverse = crt_n.m_n0_inverse;

    m_log_file = crt_n.m_log_file;
    m_time_stamp = crt_n.m_time_stamp;
    m_time_system = crt_n.m_time_system;

    return * this;
}


CRT_N :: ~CRT_N (){
    log_quit();
}

int CRT_N :: CRT_MOD_MUL(BN_WORD a, BN_WORD b, BN_WORD &result){
    BN_WORD aR, bR;
    BN_WORD n=m_rsa_n.m_n;
    aR=(a*m_R)%n;
    bR=(b*m_R)%n;
   
    BN_PART *bp_a;
    BN_PART *bp_b;
    BN_PART *bp_n;
    BN_PART *bp_result;
    hipMallocManaged((void**)&(bp_a),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_b),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_n),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_result),WARP_SIZE*sizeof(BN_PART));

    for(int i=0;i<WARP_SIZE;i++){
        bp_a[i]=0;
        bp_b[i]=0;
        bp_n[i]=0;
        bp_result[i]=0;
    }

    for(int i=0;i<aR.m_top;i++){
	bp_a[i]=aR.m_data[i];
    }
    
    for(int i=0;i<bR.m_top;i++){
	bp_b[i]=bR.m_data[i];
    }
    for(int i=0;i<n.m_top;i++){
	bp_n[i]=n.m_data[i];
    }
    
    GPU_WORD_mod_mul<<<1,WARP_SIZE>>>(bp_a,bp_b,bp_n,m_n0_inverse,bp_result);
    hipDeviceSynchronize();

    result.setzero(); 
    for(int i=0;i<WARP_SIZE;i++){
	result.m_data[i]=bp_result[i];
    }
    result.m_top=32;
    
    hipFree(bp_result);
    return 1;
}

int CRT_N :: CRT_MOD_EXP(BN_WORD a, BN_WORD e, BN_WORD &result){
    BN_WORD aR;
    BN_WORD n=m_rsa_n.m_n;
    aR=(a*m_R)%n;
    
    int E_bits=e.m_top*sizeof(BN_PART)*8;
   
    BN_PART *bp_a;
    BN_PART *bp_e;
    BN_PART *bp_r;
    BN_PART *bp_n;
    BN_PART *bp_result;
    hipMallocManaged((void**)&(bp_a),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_e),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_r),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_n),WARP_SIZE*sizeof(BN_PART));
    hipMallocManaged((void**)&(bp_result),WARP_SIZE*sizeof(BN_PART));

    for(int i=0;i<WARP_SIZE;i++){
        bp_a[i]=0;
        bp_e[i]=0;
	bp_r[i]=0;
        bp_n[i]=0;
        bp_result[i]=0;
    }

    for(int i=0;i<aR.m_top;i++){
	bp_a[i]=aR.m_data[i];
    }
    
    for(int i=0;i<e.m_top;i++){
	bp_e[i]=e.m_data[i];
    }
    for(int i=0;i<n.m_top;i++){
	bp_n[i]=n.m_data[i];
    }
    for(int i=0;i<m_R.m_top;i++){
	bp_r[i]=m_R.m_data[i];
    }

    GPU_WORD_mod_exp<<<1,WARP_SIZE*2>>>(bp_a,bp_e,E_bits,bp_r,bp_n,m_n0_inverse,bp_result);
    hipDeviceSynchronize();

    result.setzero(); 
    for(int i=0;i<WARP_SIZE;i++){
	result.m_data[i]=bp_result[i];
    }
    result.m_top=32;
    
    hipFree(bp_result);
    return 1;
    
}


int CRT_N :: log_create(){

    char file_name[200];
    snprintf(file_name,sizeof(file_name), "%s%s.log",RSA_FINAL_LOG, m_time_stamp.m_abbr);
    m_log_file = fopen(file_name, "a+");
    return 1;

}

int CRT_N :: log_info(LOG_TYPE log_type){
    return 1;
}

int CRT_N :: time_info(LOG_TYPE log_type, TIME_TYPE time_type){
    return 1;
}

int CRT_N :: log_quit(){

    fclose(m_log_file);
    return 1; 
    
}

}
