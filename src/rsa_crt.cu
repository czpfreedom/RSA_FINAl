#include "hip/hip_runtime.h"
#include "rsa_crt.h"
#include "stdio.h"
#include <ctime>
#include "iostream"
#include <sys/time.h>

using namespace std;

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}


__global__ void BN_WORD_parallel_Mon(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, const BN_PART n0_inverse, BN_WORD *result){
    
    int j=threadIdx.x+blockIdx.x*blockDim.x;
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;
    int dmax=a->dmax;

    __shared__  BN_PART M[SHARED_SIZE];
    __shared__  BN_PART C[SHARED_SIZE];
    __shared__  BN_PART U[SHARED_SIZE];
    __shared__  BN_PART V[SHARED_SIZE];
    __shared__  BN_PART A[SHARED_SIZE];
    __shared__  BN_PART B[SHARED_SIZE];
    __shared__  BN_PART N[SHARED_SIZE];

    if(j==0){
	for(int i=0; i<dmax;i++){
	    A[i]=a->d[i];
	    B[i]=b->d[i];
	    N[i]=n->d[i];
	}
    }

    __syncthreads();

    p_a=A[j];
    p_n=N[j];
    p_u=0;
    p_v=0;
    M[j]=0;
    C[j]=0;

    for(int i=0;i<dmax;i++){
        p_b=B[i];
        BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mul_lo(p_v,n0_inverse,M[j]);
        __syncthreads();
        p_m=M[0];
        BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
	V[j]=p_v;
        __syncthreads();
	p_v=V[int_mod(j+1,dmax)];
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
    }
    C[j]=p_u;
    U[j]=p_u;
    __syncthreads();//
    while(BN_PART_any(U,dmax)==0){
	p_u=U[int_mod(j-1,dmax)];
        if(j==0){
            p_u=0;
        }
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
	C[j]=C[j]+p_u;
	U[j]=p_u;
        __syncthreads();
    }
    result->d[j]=p_v;
    C[j]=C[dmax-1];
    if(j==0){
        while(C[j]!=0){
	    BN_WORD_mod_device(result,n,result);
	    C[j]=C[j]-1;
    	    BN_WORD_sub(result,n,result);
    	}
	BN_WORD_mod_device(result,n,result);
    }

}

__global__ void BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, const BN_WORD *one, const BN_PART n0_inverse, BN_WORD *result){
    int j=threadIdx.x+blockIdx.x*blockDim.x;
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;
    int dmax=a->dmax;

    __shared__  BN_PART M[SHARED_SIZE];
    __shared__  BN_PART C[SHARED_SIZE];
    __shared__  BN_PART U[SHARED_SIZE];
    __shared__  BN_PART V[SHARED_SIZE];
    __shared__  BN_PART A[SHARED_SIZE];
    __shared__  BN_PART B[SHARED_SIZE];
    __shared__  BN_PART N[SHARED_SIZE];

    if(j==0){
	for(int i=0; i<dmax;i++){
	    A[i]=a->d[i];
	    B[i]=b->d[i];
	    N[i]=n->d[i];
	}
    }

    __syncthreads();

    // Montgomery 
    p_a=A[j];
    p_n=N[j];
    p_u=0;
    p_v=0;
    M[j]=0;
    C[j]=0;

    for(int i=0;i<dmax;i++){
        p_b=B[i];
        BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mul_lo(p_v,n0_inverse,M[j]);
        __syncthreads();
        p_m=M[0];
        BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
	V[j]=p_v;
        __syncthreads();
	p_v=V[int_mod(j+1,dmax)];
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
    }
    C[j]=p_u;
    U[j]=p_u;
    __syncthreads();//
    while(BN_PART_any(U,dmax)==0){
	p_u=U[int_mod(j-1,dmax)];
        if(j==0){
            p_u=0;
        }
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
	C[j]=C[j]+p_u;
	U[j]=p_u;
        __syncthreads();
    }
    result->d[j]=p_v;
    C[j]=C[dmax-1];
    if(j==0){
        while(C[j]!=0){
	    BN_WORD_mod_device(result,n,result);
	    C[j]=C[j]-1;
    	    BN_WORD_sub(result,n,result);
    	}
	BN_WORD_mod_device(result,n,result);
    }

// Montgomery Reduce
    if(j==0){
        for(int i=0; i<dmax;i++){
            A[i]=result->d[i];
            B[i]=one->d[i];
        }
    }

    __syncthreads();
    
    p_a=A[j];
    p_n=n->d[j];
    p_u=0;
    p_v=0;
    M[j]=0;
    C[j]=0;

    for(int i=0;i<dmax;i++){
        p_b=B[i];
        BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mul_lo(p_v,n0_inverse,M[j]);
        __syncthreads();
        p_m=M[0];
        BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
	V[j]=p_v;
        __syncthreads();
	p_v=V[int_mod(j+1,dmax)];
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
    }
    C[j]=p_u;
    U[j]=p_u;
    __syncthreads();//
    while(BN_PART_any(U,dmax)==0){
	p_u=U[int_mod(j-1,dmax)];
        if(j==0){
            p_u=0;
        }
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
	C[j]=C[j]+p_u;
	U[j]=p_u;
        __syncthreads();
    }
    result->d[j]=p_v;
    C[j]=C[dmax-1];
    if(j==0){
        while(C[j]!=0){
	    BN_WORD_mod_device(result,n,result);
	    C[j]=C[j]-1;
    	    BN_WORD_sub(result,n,result);
    	}
	BN_WORD_mod_device(result,n,result);
    }

}

__global__ void BN_WORD_parallel_mont_exp(int dmax, BN_WORD *square_1, BN_WORD *square_2, BN_WORD *result_2, const BN_WORD *e, const BN_WORD *n, const BN_WORD *one, const BN_PART n0_inverse, BN_WORD *result){
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x;
    int tix= thread_id%dmax;
    int tiy= thread_id/dmax;
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;

    __shared__  BN_PART A[2][SHARED_SIZE];
    __shared__  BN_PART B[2][SHARED_SIZE];
    __shared__  BN_PART M[2][SHARED_SIZE];
    __shared__  BN_PART C[2][SHARED_SIZE];
    __shared__  BN_PART U[2][SHARED_SIZE];
    __shared__  BN_PART V[2][SHARED_SIZE];
    __shared__  BN_PART N   [SHARED_SIZE];
    __shared__  BN_PART E   [SHARED_SIZE];

    if(tiy==0){
	N[tix]=n->d[tix];
	E[tix]=e->d[tix];
    }
    __syncthreads();

//Montgomery
    for(int i=0; i<dmax*sizeof(BN_PART)*8;i++){
//赋值	    

        if(tiy==0){
	    A[tiy][tix]=square_1->d[tix];	
	    B[tiy][tix]=square_1->d[tix];	
	}
        else{
	    A[tiy][tix]=square_2->d[tix];	
	    B[tiy][tix]=result_2->d[tix];	
	}
	__syncthreads();

	p_u=0;
	p_v=0;
	M[tiy][tix]=0;
	C[tiy][tix]=0;
	p_n=N[tix];
	p_a=A[tiy][tix];

	__syncthreads();

//循环计算
	for(int j=0;j<dmax;j++){
	    p_b=B[tiy][j];
	    BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
	    p_u=ptemp_u+p_u;
	    BN_PART_mul_lo(p_v,n0_inverse,M[tiy][tix]);
	    __syncthreads();
	    p_m=M[tiy][0];
	    BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
	    p_u=ptemp_u+p_u;
	    V[tiy][tix]=p_v;
	    __syncthreads();
	    p_v=V[tiy][int_mod(tix+1,dmax)];
	    p_v=p_u+p_v;
	    if(p_v<p_u){
		p_u=1;
	    }
	    else{
		p_u=0;
	    }
	    BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
	    p_u=ptemp_u+p_u;
	    BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
	    p_u=ptemp_u+p_u;
	} 
	C[tiy][tix]=p_u;
	U[tiy][tix]=p_u;

//去除carry值

	while(BN_PART_any(U[tiy],dmax)==0){
	    p_u=U[tiy][int_mod(tix-1,dmax)];
	    if(tix==0){
		p_u=0;
	    }
	    p_v=p_u+p_v;
	    if(p_v<p_u){
		p_u=1;
	    }
	    else{
		p_u=0;
	    }
	    C[tiy][tix]=C[tiy][tix]+p_u;
	    U[tiy][tix]=p_u;
	}
	
//模运算

	if(tiy==0){
	    A[tiy][tix]=p_v;
	    C[tiy][tix]=C[tiy][dmax-1];
	    square_1->d[tix]=A[tiy][tix];
	    if(tix==0){
	        while(C[tiy][tix]!=0){
		    BN_WORD_mod_device(square_1,n,square_1);
    		    C[tiy][tix]=C[tiy][tix]-1;
    		    BN_WORD_sub(square_1,n,square_1);
    		}
    		BN_WORD_mod_device(square_1,n,square_1);						    	    
	    }
	}
	else{
	    if(BN_PART_get_bit(E[i/(sizeof(BN_PART)*8)],i%(sizeof(BN_PART)*8))==1){
	        A[tiy][tix]=p_v;
    		C[tiy][tix]=C[tiy][dmax-1];
		result_2->d[tix]=A[tiy][tix];
    		if(tix==0){
    		    while(C[tiy][tix]!=0){
			BN_WORD_mod_device(result_2,n,result_2);
    			C[tiy][tix]=C[tiy][tix]-1;
    			BN_WORD_sub(result_2,n,result_2);
    		    }
    		    BN_WORD_mod_device(result_2,n,result_2);						    
    		}	        
	    }
	}
	__syncthreads();
	if((tiy==0)&&(tix==0)){
	    BN_WORD_copy(square_1,square_2);
	}
	__syncthreads();
    }

//Reduce
    A[tiy][tix]=result_2->d[tix];
    B[tiy][tix]=one->d[tix];
    p_a=A[tiy][tix];
    p_u=0;
    p_v=0;
    M[tiy][tix]=0;
    C[tiy][tix]=0;

    for(int j=0;j<dmax;j++){
	p_b=B[tiy][j];
	BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	BN_PART_mul_lo(p_v,n0_inverse,M[tiy][tix]);
	p_m=M[tiy][0];
	BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	V[tiy][tix]=p_v;
	p_v=V[tiy][int_mod(tix+1,dmax)];
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
	BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
    } 
    C[tiy][tix]=p_u;
    U[tiy][tix]=p_u;
    while(BN_PART_any(U[tiy],dmax)==0){
	p_u=U[tiy][int_mod(tix-1,dmax)];
	if(tix==0){
	    p_u=0;
	}
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
	C[tiy][tix]=C[tiy][tix]+p_u;
	U[tiy][tix]=p_u;
    }
    A[tiy][tix]=p_v;
    C[tiy][tix]=C[tiy][dmax-1];
    if(tiy==0){
	result->d[tix]=A[tiy][tix];
	if(tix==0){
	    while(C[tiy][tix]!=0){
		BN_WORD_mod_device(result,n,result);
    		C[tiy][tix]=C[tiy][tix]-1;
    		BN_WORD_sub(result,n,result);
    	    }
	    BN_WORD_mod_device(result,n,result);						    
	}
    }
}

__global__ void BN_WORD_ARRAY_parallel_mont_exp(int dmax, BN_WORD_ARRAY *square_1, BN_WORD_ARRAY *square_2, BN_WORD_ARRAY *result_2, const BN_WORD_ARRAY *e, const BN_WORD *n, const BN_WORD *one, const BN_PART n0_inverse, BN_WORD_ARRAY *result){
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x;
    int tix= (thread_id%(2*dmax))%dmax;
    int tiy= (thread_id%(2*dmax))/dmax;
    int tib= (thread_id/(2*dmax));
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;

    __shared__  BN_PART A[2][SHARED_SIZE];
    __shared__  BN_PART B[2][SHARED_SIZE];
    __shared__  BN_PART M[2][SHARED_SIZE];
    __shared__  BN_PART C[2][SHARED_SIZE];
    __shared__  BN_PART U[2][SHARED_SIZE];
    __shared__  BN_PART V[2][SHARED_SIZE];
    __shared__  BN_PART N   [SHARED_SIZE];
    __shared__  BN_PART E   [SHARED_SIZE];

    if(tiy==0){
	N[tix]=n->d[tix];
	E[tix]=e->bn_word[tib]->d[tix];
    }
    __syncthreads();

//Montgomery
    for(int i=0; i<dmax*sizeof(BN_PART)*8;i++){
//赋值	    

        if(tiy==0){
	    A[tiy][tix]=square_1->bn_word[tib]->d[tix];	
	    B[tiy][tix]=square_1->bn_word[tib]->d[tix];	
	}
        else{
	    A[tiy][tix]=square_2->bn_word[tib]->d[tix];	
	    B[tiy][tix]=result_2->bn_word[tib]->d[tix];	
	}
	__syncthreads();

	p_u=0;
	p_v=0;
	M[tiy][tix]=0;
	C[tiy][tix]=0;
	p_n=N[tix];
	p_a=A[tiy][tix];

	__syncthreads();

//循环计算
	for(int j=0;j<dmax;j++){
	    p_b=B[tiy][j];
	    BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
	    p_u=ptemp_u+p_u;
	    BN_PART_mul_lo(p_v,n0_inverse,M[tiy][tix]);
	    __syncthreads();
	    p_m=M[tiy][0];
	    BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
	    p_u=ptemp_u+p_u;
	    V[tiy][tix]=p_v;
	    __syncthreads();
	    p_v=V[tiy][int_mod(tix+1,dmax)];
	    p_v=p_u+p_v;
	    if(p_v<p_u){
		p_u=1;
	    }
	    else{
		p_u=0;
	    }
	    BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
	    p_u=ptemp_u+p_u;
	    BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
	    p_u=ptemp_u+p_u;
	} 
	C[tiy][tix]=p_u;
	U[tiy][tix]=p_u;

//去除carry值

	while(BN_PART_any(U[tiy],dmax)==0){
	    p_u=U[tiy][int_mod(tix-1,dmax)];
	    if(tix==0){
		p_u=0;
	    }
	    p_v=p_u+p_v;
	    if(p_v<p_u){
		p_u=1;
	    }
	    else{
		p_u=0;
	    }
	    C[tiy][tix]=C[tiy][tix]+p_u;
	    U[tiy][tix]=p_u;
	}
	
//模运算

	if(tiy==0){
	    A[tiy][tix]=p_v;
	    C[tiy][tix]=C[tiy][dmax-1];
	    square_1->bn_word[tib]->d[tix]=A[tiy][tix];
	    if(tix==0){
	        while(C[tiy][tix]!=0){
		    BN_WORD_mod_device(square_1->bn_word[tib],n,square_1->bn_word[tib]);
    		    C[tiy][tix]=C[tiy][tix]-1;
    		    BN_WORD_sub(square_1->bn_word[tib],n,square_1->bn_word[tib]);
    		}
    		BN_WORD_mod_device(square_1->bn_word[tib],n,square_1->bn_word[tib]);						    	    
	    }
	}
	else{
	    if(BN_PART_get_bit(E[i/(sizeof(BN_PART)*8)],i%(sizeof(BN_PART)*8))==1){
	        A[tiy][tix]=p_v;
    		C[tiy][tix]=C[tiy][dmax-1];
		result_2->bn_word[tib]->d[tix]=A[tiy][tix];
    		if(tix==0){
    		    while(C[tiy][tix]!=0){
			BN_WORD_mod_device(result_2->bn_word[tib],n,result_2->bn_word[tib]);
    			C[tiy][tix]=C[tiy][tix]-1;
    			BN_WORD_sub(result_2->bn_word[tib],n,result_2->bn_word[tib]);
    		    }
    		    BN_WORD_mod_device(result_2->bn_word[tib],n,result_2->bn_word[tib]);
    		}	        
	    }
	}
	__syncthreads();
	if((tiy==0)&&(tix==0)){
	    BN_WORD_copy(square_1->bn_word[tib],square_2->bn_word[tib]);
	}
	__syncthreads();
    }

//Reduce
    A[tiy][tix]=result_2->bn_word[tib]->d[tix];
    B[tiy][tix]=one->d[tix];
    p_a=A[tiy][tix];
    p_u=0;
    p_v=0;
    M[tiy][tix]=0;
    C[tiy][tix]=0;

    for(int j=0;j<dmax;j++){
	p_b=B[tiy][j];
	BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	BN_PART_mul_lo(p_v,n0_inverse,M[tiy][tix]);
	p_m=M[tiy][0];
	BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	V[tiy][tix]=p_v;
	p_v=V[tiy][int_mod(tix+1,dmax)];
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
	BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
	BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
	p_u=ptemp_u+p_u;
    } 
    C[tiy][tix]=p_u;
    U[tiy][tix]=p_u;
    while(BN_PART_any(U[tiy],dmax)==0){
	p_u=U[tiy][int_mod(tix-1,dmax)];
	if(tix==0){
	    p_u=0;
	}
	p_v=p_u+p_v;
	if(p_v<p_u){
	    p_u=1;
	}
	else{
	    p_u=0;
	}
	C[tiy][tix]=C[tiy][tix]+p_u;
	U[tiy][tix]=p_u;
    }
    A[tiy][tix]=p_v;
    C[tiy][tix]=C[tiy][dmax-1];
    if(tiy==0){
	result->bn_word[tib]->d[tix]=A[tiy][tix];
	if(tix==0){
	    while(C[tiy][tix]!=0){
		BN_WORD_mod_device(result->bn_word[tib],n,result->bn_word[tib]);
    		C[tiy][tix]=C[tiy][tix]-1;
    		BN_WORD_sub(result->bn_word[tib],n,result->bn_word[tib]);
    	    }
	    BN_WORD_mod_device(result->bn_word[tib],n,result->bn_word[tib]);						    
	}
    }
}

CRT_N ::CRT_N (RSA_N *rsa_n){

    m_rsa_n = rsa_n;
    int dmax = m_rsa_n->n->dmax;
    m_zero=BN_WORD_new(dmax);
    m_one=BN_WORD_new(dmax);
    m_R=BN_WORD_new(dmax);
    BN_WORD_setzero(m_zero);
    BN_WORD_setone(m_one);
    BN_WORD_sub(m_zero,m_rsa_n->n,m_R);

    BN_PART_mod_inverse(m_rsa_n->n->d[0], 0, m_n0_inverse);
    m_n0_inverse=0-m_n0_inverse;

}

CRT_N :: ~CRT_N (){
    BN_WORD_free(m_zero);
    BN_WORD_free(m_R);

}

int CRT_N :: CRT_MUL_MOD(BN_WORD *a, BN_WORD *b, BN_WORD *result){
    int dmax = a->dmax;
    BN_WORD *a_pro, *b_pro, *temp_result;
    
    a_pro=BN_WORD_new(dmax);
    b_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    BN_WORD_mod(a,m_rsa_n->n,a_pro);
    BN_WORD_mod(b,m_rsa_n->n,b_pro);
    
    
    BN_WORD_mul_mod(a_pro,m_R,m_rsa_n->n,temp_result);
    BN_WORD_copy(temp_result,a_pro);
    BN_WORD_mul_mod(b_pro,m_R,m_rsa_n->n,temp_result);
    BN_WORD_copy(temp_result,b_pro);
    BN_WORD_parallel_Mon<<<1,dmax>>>(a_pro, b_pro, m_rsa_n->n,m_n0_inverse, result);
    hipDeviceSynchronize();
    BN_WORD_parallel_Mon<<<1,dmax>>>(result,m_one, m_rsa_n->n,m_n0_inverse, result);
    hipDeviceSynchronize();
    BN_WORD_free(a_pro);
    BN_WORD_free(b_pro);
    BN_WORD_free(temp_result);
    return 0;
}

int CRT_N :: CRT_EXP_MOD(BN_WORD *a, BN_WORD *e, BN_WORD *result){
    
    int dmax = a->dmax;
    BN_WORD *a_pro, *temp_result, *square;
    a_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    square=BN_WORD_new(dmax);

    BN_WORD_mod(a,m_rsa_n->n,a_pro);
    BN_WORD_mul_mod(a_pro,m_R,m_rsa_n->n,temp_result);
    BN_WORD_copy(temp_result,a_pro);
    BN_WORD_copy(a_pro,square);
    BN_WORD_copy(m_R,result);
/*
    for(int i=dmax-1; i>=0;i--){
        for(int j=sizeof(BN_PART)*8-1;j>=0;j--){
	    BN_WORD_parallel_Mon<<<1,dmax>>>(result, result, m_rsa_n->n,m_n0_inverse, result); 
	    hipDeviceSynchronize();
	    if(BN_PART_get_bit(e->d[i],j)==1){
                BN_WORD_parallel_Mon<<<1,dmax>>>(result, a_pro, m_rsa_n->n,m_n0_inverse, result);
		hipDeviceSynchronize();
	    }
	}
    }
*/

    for(int i=0;i<dmax;i++){
        for(int j=0;j<sizeof(BN_PART)*8;j++){
	    if(BN_PART_get_bit(e->d[i],j)==1){
		BN_WORD_parallel_Mon<<<1,dmax>>>(result, square, m_rsa_n->n,m_n0_inverse, result);
		hipDeviceSynchronize();
	    }
	    BN_WORD_parallel_Mon<<<1,dmax>>>(square,square, m_rsa_n->n,m_n0_inverse, square);
	    hipDeviceSynchronize();
	}
    }

    BN_WORD_parallel_Mon<<<1,dmax>>>(result,m_one, m_rsa_n->n,m_n0_inverse, result);
    hipDeviceSynchronize();
    BN_WORD_free(a_pro);
    BN_WORD_free(square);
    BN_WORD_free(temp_result);
    return 0;
}

int CRT_N :: CRT_EXP_MOD_PARALL(BN_WORD *a, BN_WORD *e, BN_WORD *result){
    int dmax=a->dmax;
    BN_WORD *a_pro, *temp_result, *square_1, *square_2, *result_2;
    a_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    square_1=BN_WORD_new(dmax);
    square_2=BN_WORD_new(dmax);
    result_2=BN_WORD_new(dmax);
   
    BN_WORD_mod(a,m_rsa_n->n,a_pro);
    BN_WORD_mul_mod(a_pro,m_R,m_rsa_n->n,temp_result);
    BN_WORD_copy(temp_result,a_pro);
    BN_WORD_copy(a_pro,square_1);
    BN_WORD_copy(a_pro,square_2);
    BN_WORD_copy(m_R,result_2);
       
    BN_WORD_parallel_mont_exp<<<1,dmax*2>>>(dmax, square_1, square_2, result_2, e, m_rsa_n->n, m_one, m_n0_inverse, result);
    hipDeviceSynchronize();
    
    BN_WORD_free(a_pro);
    BN_WORD_free(temp_result);
    BN_WORD_free(square_1);
    BN_WORD_free(square_2);
    BN_WORD_free(result_2);
    return 0;
}

int CRT_N :: CRT_EXP_MOD_ARRAY(BN_WORD_ARRAY *a, BN_WORD_ARRAY *e, BN_WORD_ARRAY *result){
    int word_num = a->word_num;
    int dmax = a->bn_word[0]->dmax;
    BN_WORD_ARRAY *a_pro, *temp_result, *square_1, *square_2, *result_2;

    clock_t time_start=clock();
    
    a_pro=BN_WORD_ARRAY_new(word_num, dmax);
    temp_result=BN_WORD_ARRAY_new(word_num, dmax);
    square_1=BN_WORD_ARRAY_new(word_num, dmax);
    square_2=BN_WORD_ARRAY_new(word_num, dmax);
    result_2=BN_WORD_ARRAY_new(word_num, dmax);
    
    clock_t time_end=clock();
    cout<<"new time use:"<<1000*(time_end-time_start)/(double)CLOCKS_PER_SEC<<"ms"<<endl;

    time_start=clock();
    for(int i=0;i<word_num;i++){
	BN_WORD_mod(a->bn_word[i],m_rsa_n->n,a_pro->bn_word[i]);
    	BN_WORD_mul_mod(a_pro->bn_word[i],m_R,m_rsa_n->n,temp_result->bn_word[i]);
    	BN_WORD_copy(temp_result->bn_word[i],a_pro->bn_word[i]);
    	BN_WORD_copy(a_pro->bn_word[i],square_1->bn_word[i]);
    	BN_WORD_copy(a_pro->bn_word[i],square_2->bn_word[i]);
    	BN_WORD_copy(m_R,result_2->bn_word[i]);
    }

    time_end=clock();
    cout<<"init time use:"<<1000*(time_end-time_start)/(double)CLOCKS_PER_SEC<<"ms"<<endl;
    
    time_start=clock();
    double iStart = cpuSecond();
    BN_WORD_ARRAY_parallel_mont_exp<<<word_num,2*dmax>>>(dmax, square_1, square_2, result_2, e, m_rsa_n->n , m_one, m_n0_inverse, result);
    hipDeviceSynchronize();
    double iElaps = cpuSecond() - iStart;
    time_end=clock();
    cout<<"iElaps:"<<iElaps<<endl;
    cout<<"calculate time use:"<<1000*(time_end-time_start)/(double)CLOCKS_PER_SEC<<"ms"<<endl;

    time_start=clock();
    BN_WORD_ARRAY_free(a_pro);
    BN_WORD_ARRAY_free(temp_result);
    BN_WORD_ARRAY_free(square_1);
    BN_WORD_ARRAY_free(square_2);
    BN_WORD_ARRAY_free(result_2);
    time_end=clock();
    cout<<"free time use:"<<1000*(time_end-time_start)/(double)CLOCKS_PER_SEC<<"ms"<<endl;
    return 0;
}
