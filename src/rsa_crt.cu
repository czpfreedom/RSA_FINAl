#include "hip/hip_runtime.h"
#include "rsa_crt.h"
#include "stdio.h"

__global__ void BN_WORD_parallel_Mon(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, const BN_PART n0_inverse, BN_WORD *result){
    
    int j=threadIdx.x+blockIdx.x*blockDim.x;
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;
    int dmax=a->dmax;

    __shared__  BN_PART M[SHARED_SIZE];
    __shared__  BN_PART C[SHARED_SIZE];
    __shared__  BN_PART U[SHARED_SIZE];
    __shared__  BN_PART V[SHARED_SIZE];
    __shared__  BN_PART A[SHARED_SIZE];
    __shared__  BN_PART B[SHARED_SIZE];
    __shared__  BN_PART N[SHARED_SIZE];

    if(j==0){
	for(int i=0; i<dmax;i++){
	    A[i]=a->d[i];
	    B[i]=b->d[i];
	    N[i]=n->d[i];
	}
    }

    __syncthreads();

    p_a=A[j];
    p_n=N[j];
    p_u=0;
    p_v=0;
    M[j]=0;
    C[j]=0;

    for(int i=0;i<dmax;i++){
        p_b=B[i];
        BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mul_lo(p_v,n0_inverse,M[j]);
        __syncthreads();
        p_m=M[0];
        BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
	V[j]=p_v;
        __syncthreads();
	p_v=V[int_mod(j+1,dmax)];
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
    }
    C[j]=p_u;
    U[j]=p_u;
    __syncthreads();//
    while(BN_PART_any(U,dmax)==0){
	p_u=U[int_mod(j-1,dmax)];
        if(j==0){
            p_u=0;
        }
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
	C[j]=C[j]+p_u;
	U[j]=p_u;
        __syncthreads();
    }
    result->d[j]=p_v;
    C[j]=C[dmax-1];
    if(j==0){
        while(C[j]!=0){
	    BN_WORD_mod_device(result,n,result);
	    C[j]=C[j]-1;
    	    BN_WORD_sub(result,n,result);
    	}
	BN_WORD_mod_device(result,n,result);
    }

}

__global__ void BN_WORD_parallel_mont_mul(const BN_WORD *a, const BN_WORD *b, const BN_WORD *n, const BN_WORD *one, const BN_PART n0_inverse, BN_WORD *result){
    int j=threadIdx.x+blockIdx.x*blockDim.x;
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;
    int dmax=a->dmax;

    __shared__  BN_PART M[SHARED_SIZE];
    __shared__  BN_PART C[SHARED_SIZE];
    __shared__  BN_PART U[SHARED_SIZE];
    __shared__  BN_PART V[SHARED_SIZE];
    __shared__  BN_PART A[SHARED_SIZE];
    __shared__  BN_PART B[SHARED_SIZE];
    __shared__  BN_PART N[SHARED_SIZE];

    if(j==0){
	for(int i=0; i<dmax;i++){
	    A[i]=a->d[i];
	    B[i]=b->d[i];
	    N[i]=n->d[i];
	}
    }

    __syncthreads();

    // Montgomery 
    p_a=A[j];
    p_n=N[j];
    p_u=0;
    p_v=0;
    M[j]=0;
    C[j]=0;

    for(int i=0;i<dmax;i++){
        p_b=B[i];
        BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mul_lo(p_v,n0_inverse,M[j]);
        __syncthreads();
        p_m=M[0];
        BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
	V[j]=p_v;
        __syncthreads();
	p_v=V[int_mod(j+1,dmax)];
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
    }
    C[j]=p_u;
    U[j]=p_u;
    __syncthreads();//
    while(BN_PART_any(U,dmax)==0){
	p_u=U[int_mod(j-1,dmax)];
        if(j==0){
            p_u=0;
        }
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
	C[j]=C[j]+p_u;
	U[j]=p_u;
        __syncthreads();
    }
    result->d[j]=p_v;
    C[j]=C[dmax-1];
    if(j==0){
        while(C[j]!=0){
	    BN_WORD_mod_device(result,n,result);
	    C[j]=C[j]-1;
    	    BN_WORD_sub(result,n,result);
    	}
	BN_WORD_mod_device(result,n,result);
    }

// Montgomery Reduce
    if(j==0){
        for(int i=0; i<dmax;i++){
            A[i]=result->d[i];
            B[i]=one->d[i];
        }
    }

    __syncthreads();
    
    p_a=A[j];
    p_n=n->d[j];
    p_u=0;
    p_v=0;
    M[j]=0;
    C[j]=0;

    for(int i=0;i<dmax;i++){
        p_b=B[i];
        BN_PART_mad_lo(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mul_lo(p_v,n0_inverse,M[j]);
        __syncthreads();
        p_m=M[0];
        BN_PART_mad_lo(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
	V[j]=p_v;
        __syncthreads();
	p_v=V[int_mod(j+1,dmax)];
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
        BN_PART_mad_hi(p_a,p_b,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
        BN_PART_mad_hi(p_n,p_m,p_v,ptemp_u,p_v);
        p_u=ptemp_u+p_u;
    }
    C[j]=p_u;
    U[j]=p_u;
    __syncthreads();//
    while(BN_PART_any(U,dmax)==0){
	p_u=U[int_mod(j-1,dmax)];
        if(j==0){
            p_u=0;
        }
        p_v=p_u+p_v;
        if(p_v<p_u){
            p_u=1;
        }
        else{
            p_u=0;
        }
	C[j]=C[j]+p_u;
	U[j]=p_u;
        __syncthreads();
    }
    result->d[j]=p_v;
    C[j]=C[dmax-1];
    if(j==0){
        while(C[j]!=0){
	    BN_WORD_mod_device(result,n,result);
	    C[j]=C[j]-1;
    	    BN_WORD_sub(result,n,result);
    	}
	BN_WORD_mod_device(result,n,result);
    }

}
/*
__global__ void BN_WORD_parallel_mont_mul(const BN_WORD *square_1, const BN_WORD *square_2, const BN_WORD *result2, const BN_WORD *e, const BN_WORD *n, const BN_WORD *one, const BN_PART n0_inverse, BN_WORD *result){
    int dmax=square_1->dmax;
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x;
    int j= thread_id%dmax;
    BN_PART e_i;
    BN_PART p_a, p_b, p_u, p_v,ptemp_u,p_n,p_m;

    __shared__  BN_PART M1[SHARED_SIZE];
    __shared__  BN_PART C1[SHARED_SIZE];
    __shared__  BN_PART U1[SHARED_SIZE];
    __shared__  BN_PART V1[SHARED_SIZE];
    __shared__  BN_PART M2[SHARED_SIZE];
    __shared__  BN_PART C2[SHARED_SIZE];
    __shared__  BN_PART U2[SHARED_SIZE];
    __shared__  BN_PART V2[SHARED_SIZE];

    for(int i=0; i<dmax*sizeof(BN_PART)*8;i++){
        e_i = bn_word_get_bit(e,i);
	if(thread_id<dmax){
	
	else {
	
	}

    }
}
*/


CRT_N ::CRT_N (RSA_N *rsa_n){

    m_rsa_n = rsa_n;
    int dmax = m_rsa_n->n->dmax;
    m_zero=BN_WORD_new(dmax);
    m_one=BN_WORD_new(dmax);
    m_R=BN_WORD_new(dmax);
    BN_WORD_setzero(m_zero);
    BN_WORD_setone(m_one);
    BN_WORD_sub(m_zero,m_rsa_n->n,m_R);

    BN_PART_inverse(m_rsa_n->n->d[0], 0, m_n0_inverse);
    m_n0_inverse=0-m_n0_inverse;

}

CRT_N :: ~CRT_N (){
    BN_WORD_free(m_zero);
    BN_WORD_free(m_R);

}

int CRT_N :: CRT_MUL_MOD(BN_WORD *a, BN_WORD *b, BN_WORD *result){
    int dmax = a->dmax;
    BN_WORD *a_pro, *b_pro, *temp_result;
    
    a_pro=BN_WORD_new(dmax);
    b_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);
    BN_WORD_mod(a,m_rsa_n->n,a_pro);
    BN_WORD_mod(b,m_rsa_n->n,b_pro);
    
    
    BN_WORD_mul_mod(a_pro,m_R,m_rsa_n->n,temp_result);
    BN_WORD_copy(temp_result,a_pro);
    BN_WORD_mul_mod(b_pro,m_R,m_rsa_n->n,temp_result);
    BN_WORD_copy(temp_result,b_pro);
    BN_WORD_parallel_Mon<<<1,dmax>>>(a_pro, b_pro, m_rsa_n->n,m_n0_inverse, result);
    hipDeviceSynchronize();
    BN_WORD_parallel_Mon<<<1,dmax>>>(result,m_one, m_rsa_n->n,m_n0_inverse, result);
    hipDeviceSynchronize();
    BN_WORD_free(a_pro);
    BN_WORD_free(b_pro);
    BN_WORD_free(temp_result);
    return 0;
}

int CRT_N :: CRT_EXP_MOD(BN_WORD *a, BN_WORD *e, BN_WORD *result){
    
    int dmax = a->dmax;
    BN_WORD *a_pro, *temp_result;
    a_pro=BN_WORD_new(dmax);
    temp_result=BN_WORD_new(dmax);

    BN_WORD_mod(a,m_rsa_n->n,a_pro);
    BN_WORD_mul_mod(a_pro,m_R,m_rsa_n->n,temp_result);
    BN_WORD_copy(temp_result,a_pro);
    BN_WORD_copy(m_R,result);


    for(int i=dmax-1; i>=0;i--){
        for(int j=sizeof(BN_PART)*8-1;j>=0;j--){
	    BN_WORD_parallel_Mon<<<1,dmax>>>(result, result, m_rsa_n->n,m_n0_inverse, result); 
	    hipDeviceSynchronize();
	    if(BN_PART_get_bit(e->d[i],j)==1){
                BN_WORD_parallel_Mon<<<1,dmax>>>(result, a_pro, m_rsa_n->n,m_n0_inverse, result);
		hipDeviceSynchronize();
	    }
	}
    }

    BN_WORD_parallel_Mon<<<1,dmax>>>(result,m_one, m_rsa_n->n,m_n0_inverse, result);
    hipDeviceSynchronize();
    BN_WORD_free(a_pro);
    BN_WORD_free(temp_result);
    return 0;

}
