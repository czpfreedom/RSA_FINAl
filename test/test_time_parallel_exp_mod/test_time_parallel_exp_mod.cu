#include "hip/hip_runtime.h"
#include "bn_num_operation.h"
#include "bn_openssl.h"
#include "openssl/bn.h"
#include "iostream"
#include "parallel_mont_exp.h"

#define CUDA_TIMING

#ifndef DMAX
#define DMAX 1
#endif

#ifndef WMAX
#define WMAX 100
#endif

#ifdef CUDA_TIMING
#include "sys/time.h"
#endif

using namespace std;

int main(){



BIGNUM *open_a, *open_b,*open_e,*open_n,*open_result,*open_R, *open_temp;
BN_CTX *ctx;
BN_NUM *bn_a, *bn_b,*bn_e, *bn_n, *bn_result,*bn_word_result;

#ifdef CLOCKING
clock_t start, stop, sum_time;
#endif

#ifdef CUDA_TIMING
    timeval start, stop;
    double sum_time;
#endif

//test R_inverse and test mul_mod_host
open_a=BN_new();
open_b=BN_new();
open_n=BN_new();
open_result=BN_new();
ctx=BN_CTX_new();

bn_a=BN_NUM_new(WMAX,DMAX);
bn_b=BN_NUM_new(WMAX,DMAX);
bn_n=BN_NUM_new(WMAX,DMAX);
bn_result=BN_NUM_new(WMAX,DMAX);
bn_word_result=BN_NUM_new(WMAX,DMAX);

BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8)*WMAX,0,0);
BN_rand(open_b,DMAX*(sizeof(BN_ULONG)*8)*WMAX,0,0);
BN_rand(open_n,DMAX*(sizeof(BN_ULONG)*8)*WMAX,0,0);
while((open_n->d[0]%2)==0){
        BN_rand(open_n,DMAX*(sizeof(BN_ULONG)*8)*WMAX,0,0);
}
#ifdef CLOCKING
    start=clock();
#endif 

#ifdef CUDA_TIMING
    gettimeofday(&start,0);
#endif
for(int i=0;i<1;i++){
    BN_mod_mul(open_result, open_a, open_b, open_n, ctx);
}

#ifdef CLOCKING
    stop=time(NULL);
    sum_time= stop-start;
    cout<<"cpu_time:"<<(double)(sum_time)<<endl;
#endif

#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"cpu_time: "<<sum_time<<endl;
#endif

BN_NUM_openssl_transform(open_a,bn_a,WMAX,DMAX);
BN_NUM_openssl_transform(open_b,bn_b,WMAX,DMAX);
BN_NUM_openssl_transform(open_n,bn_n,WMAX,DMAX);
BN_NUM_openssl_transform(open_result,bn_result,WMAX,DMAX);
/*
cout<<"open_a"<<endl;
BN_NUM_print(bn_a);
cout<<"open_b"<<endl;
BN_NUM_print(bn_b);
cout<<"open_n"<<endl;
BN_NUM_print(bn_n);
cout<<"open_result"<<endl;
BN_NUM_print(bn_result);
*/
#ifdef CUDA_TIMING
    gettimeofday(&start,0);
#endif

for(int i=0;i<1;i++){
    BN_NUM_parallel_mod_mul(bn_a, bn_b, bn_n,bn_word_result);
}

#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"gpu_time: "<<sum_time<<endl;
#endif

/*
cout<<"bn_word_result"<<endl;
BN_NUM_print(bn_word_result);
*/
BN_free(open_a);
BN_free(open_b);
BN_free(open_n);
BN_free(open_result);
BN_CTX_free(ctx);
BN_NUM_free(bn_a);
BN_NUM_free(bn_b);
BN_NUM_free(bn_n);
BN_NUM_free(bn_result);
BN_NUM_free(bn_word_result);
/*
//test parallel_exp_mod

open_a=BN_new();
open_e=BN_new();
open_n=BN_new();
open_result=BN_new();
open_R=BN_new();
open_temp=BN_new();
ctx=BN_CTX_new();

bn_a=BN_NUM_new(WMAX,DMAX);
bn_e=BN_NUM_new(WMAX,DMAX);
bn_n=BN_NUM_new(WMAX,DMAX);
bn_result=BN_NUM_new(WMAX,DMAX);
bn_word_result=BN_NUM_new(WMAX,DMAX);
BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8)*WMAX,0,0);
BN_rand(open_e,DMAX*(sizeof(BN_ULONG)*8)*WMAX,0,0);
BN_rand(open_n,DMAX*(sizeof(BN_ULONG)*8)*WMAX,0,0);
BN_rand(open_R,DMAX*(sizeof(BN_ULONG)*8)*WMAX+8,0,0);
for(int i=0;i<WMAX*DMAX;i++){
    open_R->d[i]=0;
}
open_R->d[WMAX*DMAX]=1;
while((open_n->d[0]%2)==0){
        BN_rand(open_n,DMAX*(sizeof(BN_ULONG)*8)*WMAX,0,0);
}
BN_mod_exp(open_result, open_a, open_e, open_n, ctx);
BN_NUM_openssl_transform(open_a,bn_a,WMAX,DMAX);
BN_NUM_openssl_transform(open_e,bn_e,WMAX,DMAX);
BN_NUM_openssl_transform(open_n,bn_n,WMAX,DMAX);
BN_NUM_openssl_transform(open_result,bn_result,WMAX,DMAX);
cout<<"open_a"<<endl;
BN_NUM_print(bn_a);
cout<<"open_e"<<endl;
BN_NUM_print(bn_e);
cout<<"open_n"<<endl;
BN_NUM_print(bn_n);
cout<<"open_result"<<endl;
BN_NUM_print(bn_result);
BN_NUM_parallel_mont_exp(bn_a, bn_e, bn_n, WMAX, DMAX, bn_word_result);
cout<<"bn_word_result"<<endl;
BN_NUM_print(bn_word_result);

BN_free(open_a);
BN_free(open_e);
BN_free(open_n);
BN_free(open_result);
BN_CTX_free(ctx);
BN_NUM_free(bn_a);
BN_NUM_free(bn_e);
BN_NUM_free(bn_n);
BN_NUM_free(bn_result);
BN_NUM_free(bn_word_result);
*/

return 0;


}
