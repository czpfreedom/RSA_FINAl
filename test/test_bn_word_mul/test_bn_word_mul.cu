#include "hip/hip_runtime.h"
#include "bn_word_operation.h"
#include "bn_openssl.h"
#include "openssl/bn.h"
#include "iostream"

#define DMAX 3
#define LOOP_NUM 1
#define CUDA_TIMING

#define PRINT

//#define MUL
#define DIV

using namespace std;

#ifdef CUDA_TIMING
#include "sys/time.h"
#endif

__global__ void gpu_bn_word_mul(BN_WORD *a,BN_WORD *b,BN_WORD *result){
    BN_WORD_mul_device(a,b,result);
}


int main(){
    BIGNUM *open_a, *open_b, *open_result, *open_q, *open_r;
    BN_WORD *bn_a, *bn_b, *bn_result, *bn_word_result, *bn_q, *bn_r, *bn_word_q, *bn_word_r;
    BN_CTX *ctx;

#ifdef CUDA_TIMING
    timeval start, stop;
    double sum_time;
#endif

#ifdef MUL
//test mul
    cout<<"test_mul:"<<endl;
    open_a=BN_new();
    open_b=BN_new();
    open_result=BN_new();
    ctx=BN_CTX_new();
    BN_rand(open_a,DMAX*(sizeof(BN_PART)*8),0,0);
    BN_rand(open_b,DMAX*(sizeof(BN_PART)*8),0,0);

#ifdef CUDA_TIMING
    gettimeofday(&start,0);
#endif
    for(int i=0;i<LOOP_NUM;i++){
        BN_mul(open_result,open_a,open_b,ctx);
    }
#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"mul_cpu_time: "<<sum_time<<endl;
#endif

#ifdef CUDA_TIMING
    gettimeofday(&start,0);
#endif
    BN_WORD *test_word;
    for(int i=0;i<LOOP_NUM;i++){
        test_word=BN_WORD_new(DMAX);
	BN_WORD_free(test_word);
    }
#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"new_cpu_time: "<<sum_time<<endl;
#endif

    bn_a=BN_WORD_new(DMAX);
    bn_b=BN_WORD_new(DMAX);
    bn_result=BN_WORD_new(DMAX*2);
    bn_word_result=BN_WORD_new(DMAX);
    BN_WORD_openssl_transform(open_a,bn_a,DMAX);
    BN_WORD_openssl_transform(open_b,bn_b,DMAX);
    BN_WORD_openssl_transform(open_result,bn_result,DMAX);

#ifdef CUDA_TIMING
    gettimeofday(&start,0);
#endif
    for(int i=0;i<LOOP_NUM;i++){
	BN_WORD_mul(bn_a,bn_b,bn_word_result);
    }
#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"mul_gpu_time: "<<sum_time<<endl;
#endif
#ifdef PRINT    
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    cout<<"b:"<<endl;
    BN_WORD_print(bn_b);
    cout<<"open_result"<<endl;
    BN_WORD_print(bn_result);
    cout<<"bn_word_result"<<endl;
    BN_WORD_print(bn_word_result);
#endif

    BN_free(open_a);
    BN_free(open_b);
    BN_free(open_result);
    BN_CTX_free(ctx);
    BN_WORD_free(bn_a);
    BN_WORD_free(bn_b);
    BN_WORD_free(bn_result);
    BN_WORD_free(bn_word_result);
#endif
//test div
#ifdef DIV
    cout<<"test_div:"<<endl;
    open_a=BN_new();
    open_b=BN_new();
    open_q=BN_new();
    open_r=BN_new();
    ctx=BN_CTX_new();
    BN_rand(open_a,DMAX*(sizeof(BN_PART)*8),0,0);
    BN_rand(open_b,DMAX*(sizeof(BN_PART)*8),0,0);

#ifdef CUDA_TIMING
    gettimeofday(&start,0);
#endif
    for(int i=0;i<LOOP_NUM;i++){
        BN_div(open_q,open_r,open_a,open_b,ctx);
    }
#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"div_cpu_time: "<<sum_time<<endl;
#endif
    bn_a=BN_WORD_new(DMAX);
    bn_b=BN_WORD_new(DMAX);
    bn_q=BN_WORD_new(DMAX);
    bn_r=BN_WORD_new(DMAX);
    bn_word_q=BN_WORD_new(DMAX);
    bn_word_r=BN_WORD_new(DMAX);
    BN_WORD_openssl_transform(open_a,bn_a,DMAX);
    BN_WORD_openssl_transform(open_b,bn_b,DMAX);
    BN_WORD_openssl_transform(open_q,bn_q,DMAX)+BN_WORD_openssl_transform(open_r,bn_r,DMAX);

#ifdef CUDA_TIMING
    gettimeofday(&start,0);
#endif
        BN_WORD_div(bn_a,bn_b,bn_word_q,bn_word_r);

#ifdef CUDA_TIMING
    gettimeofday(&stop,0);
    sum_time = 1000000*(stop.tv_sec - start.tv_sec) + stop.tv_usec - start.tv_usec;
    cout<<"div_cpu_time: "<<sum_time<<endl;
#endif

#ifdef PRINT
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    cout<<"b:"<<endl;
    BN_WORD_print(bn_b);
    cout<<"open_q"<<endl;
    BN_WORD_print(bn_q);
    cout<<"open_r"<<endl;
    BN_WORD_print(bn_r);
    cout<<"bn_word_q"<<endl;
    BN_WORD_print(bn_word_q);
    cout<<"bn_word_r"<<endl;
    BN_WORD_print(bn_word_r);
#endif

    BN_free(open_a);
    BN_free(open_b);
    BN_free(open_q);
    BN_free(open_r);
    BN_CTX_free(ctx);
    BN_WORD_free(bn_a);
    BN_WORD_free(bn_b);
    BN_WORD_free(bn_q);
    BN_WORD_free(bn_r);
    BN_WORD_free(bn_word_q);
    BN_WORD_free(bn_word_r);
#endif
}

