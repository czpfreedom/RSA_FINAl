#include "bn_word_operation.h"
#include "bn_openssl.h"
#include "openssl/bn.h"
#include "iostream"


#define DMAX 3

using namespace std;

int main(){
    BIGNUM *open_a, *open_b, *open_result;
    BN_WORD *bn_a, *bn_b, *bn_result, *bn_word_result,*bn_a_half,*bn_b_half, *mid_value1, *mid_value2, *mid_value3,*temp_result;
    int transform_result;
    BN_CTX *ctx;
    int *shift_return_value, *add_return_value;

//test mul
    cout<<"test_mul:"<<endl;
    open_a=BN_new();
    open_b=BN_new();
    open_result=BN_new();
    ctx=BN_CTX_new();
    BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_rand(open_b,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_mul(open_result,open_a,open_b,ctx);
    bn_a=BN_WORD_new(DMAX);
    bn_b=BN_WORD_new(DMAX);
    bn_a_half=BN_WORD_new(DMAX);
    bn_b_half=BN_WORD_new(DMAX);
    bn_result=BN_WORD_new(DMAX*2);
    bn_word_result=BN_WORD_new(DMAX*2);
    mid_value1=BN_WORD_new(DMAX*2);
    mid_value2=BN_WORD_new(DMAX*2);
    mid_value3=BN_WORD_new(DMAX*2);
    temp_result=BN_WORD_new(DMAX*2);
    transform_result=BN_WORD_openssl_transform(open_a,bn_a,DMAX)+BN_WORD_openssl_transform(open_b,bn_b,DMAX)+BN_WORD_openssl_transform(open_result,bn_result,DMAX*2);
    if(transform_result!=0){
        cerr<<"Error: transform failed"<<endl;
        exit(1);
    }
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    cout<<"b:"<<endl;
    BN_WORD_print(bn_b);
    cout<<"open_result"<<endl;
    BN_WORD_print(bn_result);
    hipMallocManaged((void**)&(shift_return_value),sizeof(int));
    hipMallocManaged((void**)&(add_return_value),sizeof(int));
    BN_WORD_mul(bn_a,bn_b, bn_a_half, bn_b_half,bn_word_result,mid_value1,mid_value2, mid_value3, temp_result,add_return_value, shift_return_value);
    cout<<"bn_word_result"<<endl;
    BN_WORD_print(bn_word_result);







}

