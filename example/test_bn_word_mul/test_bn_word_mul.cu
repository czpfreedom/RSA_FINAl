#include "hip/hip_runtime.h"
#include "bn_word_operation.h"
#include "bn_openssl.h"
#include "openssl/bn.h"
#include "iostream"


#define DMAX 3


using namespace std;

__global__ void gpu_bn_word_mul(BN_WORD *a,BN_WORD *b,BN_WORD *result){
    BN_WORD_mul(a,b,result);
}


int main(){
    BIGNUM *open_a, *open_b, *open_result;
    BN_WORD *bn_a, *bn_b, *bn_result, *bn_word_result;
    int transform_result;
    BN_CTX *ctx;

//test mul
    cout<<"test_mul:"<<endl;
    open_a=BN_new();
    open_b=BN_new();
    open_result=BN_new();
    ctx=BN_CTX_new();
    BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_rand(open_b,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_mul(open_result,open_a,open_b,ctx);
    bn_a=BN_WORD_new(DMAX);
    bn_b=BN_WORD_new(DMAX);
    bn_result=BN_WORD_new(DMAX*2);
    bn_word_result=BN_WORD_new(DMAX*2);
    transform_result=BN_WORD_openssl_transform(open_a,bn_a,DMAX)+BN_WORD_openssl_transform(open_b,bn_b,DMAX)+BN_WORD_openssl_transform(open_result,bn_result,DMAX*2);
    if(transform_result!=0){
        cerr<<"Error: transform failed"<<endl;
        exit(1);
    }
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    cout<<"b:"<<endl;
    BN_WORD_print(bn_b);
    cout<<"open_result"<<endl;
    BN_WORD_print(bn_result);
    gpu_bn_word_mul<<<1,1>>>(bn_a,bn_b,bn_word_result);
    hipDeviceSynchronize();
    cout<<"bn_word_result"<<endl;
    BN_WORD_print(bn_word_result);
    BN_free(open_a);
    BN_free(open_b);
    BN_free(open_result);
    BN_CTX_free(ctx);
    BN_WORD_free(bn_a);
    BN_WORD_free(bn_b);
    BN_WORD_free(bn_result);
    BN_WORD_free(bn_word_result);

}

