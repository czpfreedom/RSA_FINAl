#include "bn_word_operation.h"
#include "bn_openssl.h"
#include "openssl/bn.h"
#include "iostream"


#define DMAX 32

using namespace std;

int main(){

	
    BIGNUM *open_a, *open_b,*open_result;
    BN_WORD *bn_a, *bn_b, *bn_result, *bn_word_result;
    int transform_result;
    int *cmp_return_value, *add_return_value, *sub_return_value,*shift_return_value;
// test add
    cout<<"test add:"<<endl;
    open_a=BN_new();
    open_b=BN_new();
    open_result=BN_new();
    BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_rand(open_b,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_add(open_result,open_a,open_b);
    
    bn_a=BN_WORD_new(DMAX);
    bn_b=BN_WORD_new(DMAX);
    bn_result=BN_WORD_new(DMAX);
    bn_word_result=BN_WORD_new(DMAX);
    transform_result=BN_WORD_openssl_transform(open_a,bn_a,DMAX)+BN_WORD_openssl_transform(open_b,bn_b,DMAX)+BN_WORD_openssl_transform(open_result,bn_result,DMAX);
    if(transform_result!=0){
        cerr<<"Error: transform failed"<<endl;    
	exit(1);
    }
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    cout<<"b:"<<endl;
    BN_WORD_print(bn_b);
    cout<<"open_result"<<endl;
    BN_WORD_print(bn_result);
    hipMallocManaged((void**)&(add_return_value),sizeof(int));
    BN_WORD_add(bn_a,bn_b,bn_word_result,add_return_value);
    if(*(add_return_value)!=0){
        cerr<<"Error: add failed"<<endl;
	exit(1);
    }
    cout<<"bn_word_result"<<endl;
    BN_WORD_print(bn_word_result);
    BN_free(open_a);
    BN_free(open_b);
    BN_free(open_result);
    BN_WORD_free(bn_a);
    BN_WORD_free(bn_b);
    BN_WORD_free(bn_result);
    BN_WORD_free(bn_word_result);
    hipFree(add_return_value);

//test sub

    cout<<"test sub:"<<endl;
    open_a=BN_new();
    open_b=BN_new();
    open_result=BN_new();
    BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8),1,0);
    BN_rand(open_b,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_sub(open_result,open_a,open_b);
    
    bn_a=BN_WORD_new(DMAX);
    bn_b=BN_WORD_new(DMAX);
    bn_result=BN_WORD_new(DMAX);
    bn_word_result=BN_WORD_new(DMAX);
    transform_result=BN_WORD_openssl_transform(open_a,bn_a,DMAX)+BN_WORD_openssl_transform(open_b,bn_b,DMAX)+BN_WORD_openssl_transform(open_result,bn_result,DMAX);
    if(transform_result!=0){
        cerr<<"Error: transform failed"<<endl;    
	exit(1);
    }
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    cout<<"b:"<<endl;
    BN_WORD_print(bn_b);
    cout<<"open_result"<<endl;
    BN_WORD_print(bn_result);
    hipMallocManaged((void**)&(sub_return_value),sizeof(int));
    hipMallocManaged((void**)&(cmp_return_value),sizeof(int));
    BN_WORD_cmp(bn_a,bn_b,cmp_return_value);
    BN_WORD_sub(bn_a,bn_b,bn_word_result,cmp_return_value,sub_return_value);
    cout<<"sub_return:"<<*(sub_return_value)<<endl;
    if(*(sub_return_value)!=0){
        cerr<<"Error: sub failed"<<endl;
	exit(1);
    }
    cout<<"bn_word_result"<<endl;
    BN_WORD_print(bn_word_result);
    BN_free(open_a);
    BN_free(open_b);
    BN_free(open_result);
    BN_WORD_free(bn_a);
    BN_WORD_free(bn_b);
    BN_WORD_free(bn_result);
    BN_WORD_free(bn_word_result);
    hipFree(cmp_return_value);
    hipFree(sub_return_value);

//test shift
    open_a=BN_new();
    BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8),1,0);
    bn_a=BN_WORD_new(DMAX);
    transform_result=BN_WORD_openssl_transform(open_a,bn_a,DMAX);
    bn_result=BN_WORD_new(DMAX);
    if(transform_result!=0){
        cerr<<"Error: transform failed"<<endl;
        exit(1);
    }
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    hipMallocManaged((void**)&(shift_return_value),sizeof(int));
    BN_WORD_left_shift(bn_a,bn_result,10,shift_return_value);
    cout<<"left_shift:"<<endl;
    BN_WORD_print(bn_result);
    BN_WORD_right_shift(bn_a,bn_result,10,shift_return_value);
    cout<<"right_shift:"<<endl;
    BN_WORD_print(bn_result);
    BN_free(open_a);
    BN_WORD_free(bn_a);
    BN_WORD_free(bn_result);
    hipFree(shift_return_value);


//test shift_bits
    open_a=BN_new();
    BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8),1,0);
    bn_a=BN_WORD_new(DMAX);
    transform_result=BN_WORD_openssl_transform(open_a,bn_a,DMAX);
    bn_result=BN_WORD_new(DMAX);
    if(transform_result!=0){
        cerr<<"Error: transform failed"<<endl;
        exit(1);
    }
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    hipMallocManaged((void**)&(shift_return_value),sizeof(int));
    BN_WORD_left_shift_bits(bn_a,bn_result,4,shift_return_value);
    cout<<"left_shift_bits:"<<endl;
    BN_WORD_print(bn_result);
    BN_WORD_right_shift_bits(bn_a,bn_result,4,shift_return_value);
    cout<<"right_shift_bits:"<<endl;
    BN_WORD_print(bn_result);
    BN_free(open_a);
    BN_WORD_free(bn_a);
    BN_WORD_free(bn_result);
    hipFree(shift_return_value);


}


