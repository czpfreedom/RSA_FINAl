#include "bn_openssl.h"
#include "openssl/bn.h"
#include "pseudo.h"
#include "iostream"



#define DMAX 3

using namespace std;

int main(){

    BIGNUM *open_a, *open_b, *open_c, *open_result, *open_mid_value1, *open_mid_value2, *pow_2_w ;
    BN_WORD *bn_a, *bn_b, *bn_c, *bn_result,*bn_word_result, *mad_lo_u,*mad_lo_v,*mad_hi_u,*mad_hi_v;
    BN_CTX *ctx;
    int transform_result, return_value;
// test mul_lo
    cout<<"test mul_lo:"<<endl;
    open_a=BN_new();
    open_b=BN_new();
    open_result=BN_new();
    ctx=BN_CTX_new();
    BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_rand(open_b,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_mul(open_result,open_a,open_b,ctx);
    bn_a=BN_WORD_new(DMAX);
    bn_b=BN_WORD_new(DMAX);
    bn_result=BN_WORD_new(DMAX);
    bn_word_result=BN_WORD_new(DMAX);
    transform_result=BN_WORD_openssl_transform(open_a,bn_a,DMAX)+BN_WORD_openssl_transform(open_b,bn_b,DMAX)+BN_WORD_openssl_transform(open_result,bn_result,DMAX);
    if(transform_result!=0){
        cerr<<"Error: transform failed"<<endl;
        exit(1);
    }
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    cout<<"b:"<<endl;
    BN_WORD_print(bn_b);
    cout<<"open_result"<<endl;
    BN_WORD_print(bn_result);
    return_value=mul_lo(bn_a,bn_b,bn_word_result);
    if(return_value!=0){
        cerr<<"Error: mul_lo failed"<<endl;
        exit(1);
    }
    cout<<"bn_word_result"<<endl;
    BN_WORD_print(bn_word_result);
    BN_free(open_a);
    BN_free(open_b);
    BN_free(open_result);
    BN_CTX_free(ctx);
    BN_WORD_free(bn_a);
    BN_WORD_free(bn_b);
    BN_WORD_free(bn_result);
    BN_WORD_free(bn_word_result);

// test mad_lo
    
    cout<<"test mad_lo:"<<endl;
    open_a=BN_new();
    open_b=BN_new();
    open_c=BN_new();
    open_result=BN_new();
    open_mid_value1=BN_new();
    open_mid_value2=BN_new();
    ctx=BN_CTX_new();
    pow_2_w=BN_new();
    BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_rand(open_b,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_rand(open_c,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_rand(pow_2_w,DMAX*(sizeof(BN_ULONG)*8),0,0);
    BN_rand(open_mid_value2,DMAX*(sizeof(BN_ULONG)*8),0,0);
    pow_2_w->top=DMAX;
    pow_2_w->dmax=DMAX;
    pow_2_w->d[0]=0;
    pow_2_w->d[1]=1;
    for(int i=2;i<DMAX;i++){
        pow_2_w->d[i]=0;
    }
    BN_mul(open_mid_value1,open_a,open_b,ctx);
    open_mid_value2->d[0]=open_mid_value1->d[0];
    for(int i=1;i<DMAX;i++){
        open_mid_value2->d[i]=0;
    }
    BN_add(open_result,open_mid_value2,open_c);
    bn_a=BN_WORD_new(DMAX);
    bn_b=BN_WORD_new(DMAX);
    bn_c=BN_WORD_new(DMAX);
    bn_result=BN_WORD_new(DMAX*2);
    mad_lo_u=BN_WORD_new(DMAX);
    mad_lo_v=BN_WORD_new(DMAX);
    transform_result=BN_WORD_openssl_transform(open_a,bn_a,DMAX)+BN_WORD_openssl_transform(open_b,bn_b,DMAX)+BN_WORD_openssl_transform(open_c,bn_c,DMAX)
	    +BN_WORD_openssl_transform(open_result,bn_result,DMAX*2);
    if(transform_result!=0){
        cerr<<"Error: transform failed"<<endl;
        exit(1);
    }
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    cout<<"b:"<<endl;
    BN_WORD_print(bn_b);
    cout<<"c:"<<endl;
    BN_WORD_print(bn_c);
    cout<<"open_result"<<endl;
    BN_WORD_print(bn_result);
    mad_lo_global<<<1,1>>>(bn_a,bn_b,bn_c,mad_lo_u,mad_lo_v);
    hipDeviceSynchronize();
    cout<<"bn_word_result_u"<<endl;
    BN_WORD_print(mad_lo_u);
    cout<<"bn_word_result_v"<<endl;
    BN_WORD_print(mad_lo_v);
    BN_free(open_a);
    BN_free(open_b);
    BN_free(open_c);
    BN_free(open_result);
    BN_free(open_mid_value1);
    BN_free(open_mid_value2);
    BN_CTX_free(ctx);
    BN_WORD_free(bn_a);
    BN_WORD_free(bn_b);
    BN_WORD_free(bn_c);
    BN_WORD_free(mad_lo_u);
    BN_WORD_free(mad_lo_v);


//test mad_hi


    cout<<"test mad_hi:"<<endl;
    open_a=BN_new();
    open_b=BN_new();
    open_c=BN_new();
    open_result=BN_new();
    open_mid_value1=BN_new();
    open_mid_value2=BN_new();
    ctx=BN_CTX_new();
    pow_2_w=BN_new();
    BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8),1,0);
    BN_rand(open_b,DMAX*(sizeof(BN_ULONG)*8),1,0);
    BN_rand(open_c,DMAX*(sizeof(BN_ULONG)*8),1,0);
    BN_rand(pow_2_w,DMAX*(sizeof(BN_ULONG)*8),0,0);
    pow_2_w->top=DMAX;
    pow_2_w->dmax=DMAX;
    pow_2_w->d[0]=0;
    pow_2_w->d[1]=1;
    for(int i=2;i<DMAX;i++) pow_2_w->d[i]=0;
    BN_mul(open_mid_value1,open_a,open_b,ctx);
    BN_mul(open_mid_value2,open_c,pow_2_w,ctx);
    BN_add(open_result,open_mid_value2,open_mid_value1);
    bn_a=BN_WORD_new(DMAX);
    bn_b=BN_WORD_new(DMAX);
    bn_c=BN_WORD_new(DMAX);
    bn_result=BN_WORD_new(DMAX*2);
    mad_hi_u=BN_WORD_new(DMAX);
    mad_hi_v=BN_WORD_new(DMAX);
    transform_result=BN_WORD_openssl_transform(open_a,bn_a,DMAX)+BN_WORD_openssl_transform(open_b,bn_b,DMAX)+BN_WORD_openssl_transform(open_c,bn_c,DMAX)+BN_WORD_openssl_transform(open_result,bn_result,DMAX*2);
    if(transform_result!=0){
        cerr<<"Error: transform failed"<<endl;
        exit(1);
    }
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    cout<<"b:"<<endl;
    BN_WORD_print(bn_b);
    cout<<"c:"<<endl;
    BN_WORD_print(bn_c);
    cout<<"open_result"<<endl;
    BN_WORD_print(bn_result);
    mad_hi_global<<<1,1>>>(bn_a, bn_b, bn_c, mad_hi_u, mad_hi_v);
    hipDeviceSynchronize();
    cout<<"bn_word_result_u"<<endl;
    BN_WORD_print(mad_hi_u);
    cout<<"bn_word_result_v"<<endl;
    BN_WORD_print(mad_hi_v);
    
    BN_free(open_a);
    BN_free(open_b);
    BN_free(open_c);
    BN_free(pow_2_w);
    BN_free(open_result);
    BN_CTX_free(ctx);
    BN_WORD_free(bn_a);
    BN_WORD_free(bn_b);
    BN_WORD_free(bn_c);
    BN_WORD_free(mad_hi_u);
    BN_WORD_free(mad_hi_v);

/*  test mad_hi
   
   
   
    cout<<"test mul_hi:"<<endl;
    open_a=BN_new();
    open_b=BN_new();
    open_c=BN_new();
    open_result=BN_new();
    open_remain=BN_new();
    open_mid_value1=BN_new();
    open_mid_value2=BN_new();
    ctx=BN_CTX_new();
    pow_2_w=BN_new();
    BN_rand(open_a,DMAX*(sizeof(BN_ULONG)*8),1,0);
    BN_rand(open_b,DMAX*(sizeof(BN_ULONG)*8),1,0);
    BN_rand(open_c,DMAX*(sizeof(BN_ULONG)*8),1,0);
    BN_rand(pow_2_w,DMAX*(sizeof(BN_ULONG)*8),0,0);
    pow_2_w->top=DMAX;
    pow_2_w->dmax=DMAX;
    pow_2_w->d[0]=0;
    pow_2_w->d[1]=1;
    for(int i=2;i<DMAX;i++) pow_2_w->d[i]=0;
    BN_mul(open_mid_value1,open_a,open_b,ctx);
    BN_mul(open_mid_value2,open_c,pow_2_w,ctx);
    BN_add(open_result,open_mid_value2,open_mid_value1);
    bn_a=BN_WORD_new(DMAX);
    bn_b=BN_WORD_new(DMAX);
    bn_c=BN_WORD_new(DMAX);
    bn_result=BN_WORD_new(DMAX*2);
    mad_hi_u=BN_WORD_new(DMAX);
    mad_hi_v=BN_WORD_new(DMAX);
    mid_value1=BN_WORD_new(DMAX);
    mid_value2=BN_WORD_new(DMAX);
    mid_value3=BN_WORD_new(DMAX);
    mid_value4=BN_WORD_new(DMAX);
    mul_word_result=BN_WORD_new(DMAX);
    transform_result=BN_WORD_openssl_transform(open_a,bn_a,DMAX)+BN_WORD_openssl_transform(open_b,bn_b,DMAX)+BN_WORD_openssl_transform(open_c,bn_c,DMAX)
            +BN_WORD_openssl_transform(open_result,bn_result,DMAX);
    if(transform_result!=0){
        cerr<<"Error: transform failed"<<endl;
        exit(1);
    }
    cout<<"a:"<<endl;
    BN_WORD_print(bn_a);
    cout<<"b:"<<endl;
    BN_WORD_print(bn_b);
    cout<<"c:"<<endl;
    BN_WORD_print(bn_c);
    cout<<"open_result"<<endl;
    BN_WORD_print(bn_result);
    hipMallocManaged((void**)&(add_return_value),sizeof(int));
    hipMallocManaged((void**)&(mid_return_value),sizeof(int));
    hipMallocManaged((void**)&(mul_return_value),sizeof(int));
    hipMallocManaged((void**)&(mad_hi_return_value),sizeof(int));
    mad_hi(bn_a,bn_b,bn_c,mad_hi_u, mad_hi_v,mul_word_result,mid_value1,mid_value2,mid_value3,mid_value4,mul_return_value,
                add_return_value, mid_return_value, mad_hi_return_value);
    if(*(mad_hi_return_value)!=0){
        cerr<<"Error: add failed"<<endl;
        exit(1);
    }
    cout<<"bn_word_result_u"<<endl;
    BN_WORD_print(mad_hi_u);
    cout<<"bn_word_result_v"<<endl;
    BN_WORD_print(mad_hi_v);
    BN_free(open_a);
    BN_free(open_b);
    BN_free(open_c);
    BN_free(pow_2_w);
    BN_free(open_result);
    BN_CTX_free(ctx);
    BN_WORD_free(bn_a);
    BN_WORD_free(bn_b);
    BN_WORD_free(bn_c);
    BN_WORD_free(bn_result);
    BN_WORD_free(mad_lo_u);
    BN_WORD_free(mad_lo_v);
    hipFree(add_return_value);
    hipFree(mad_hi_return_value);
    hipFree(mad_hi_return_value);
*/
}
